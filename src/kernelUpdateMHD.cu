#include "hip/hip_runtime.h"
// Filename: kernelUpdateMHD.cu
//
// Copyright (c) 2010-2016, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


__global__ void kernelUpdateMHD(hipfftDoubleComplex *vxZ, 
				hipfftDoubleComplex *vyZ,
				hipfftDoubleComplex *vzZ, 
				hipfftDoubleComplex *WxZ, 
				hipfftDoubleComplex *WyZ,
				hipfftDoubleComplex *WzZ, 
				prefactorsFourier *pF){

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i>=ncellsGPU) return;   

  //Find mode
  int kx, ky, kz;
  kz = i / (mxGPU*myGPU);
  ky = (i % (mxGPU*myGPU)) / mxGPU;
  kx = i % mxGPU;
  
  //Construct L
  double L;
  L = -((pF->gradKx[kx].y) * (pF->gradKx[kx].y)) - 
    ((pF->gradKy[ky].y) * (pF->gradKy[ky].y)) -
    ((pF->gradKz[kz].y) * (pF->gradKz[kz].y));

  //Construct GG
  double GG;
  GG = L;

  //Construct denominator
  double denominator = 1 - 0.5 * dtGPU * shearviscosityGPU * L / densfluidGPU;
  double denominatorB = 1 - 0.5 * dtGPU * diffusionGPU * L ;

  //Construct GW
  hipfftDoubleComplex GW, GB;
  GW.x = pF->gradKx[kx].y * vxZ[i].x + pF->gradKy[ky].y * vyZ[i].x + pF->gradKz[kz].y * vzZ[i].x;
  GW.y = pF->gradKx[kx].y * vxZ[i].y + pF->gradKy[ky].y * vyZ[i].y + pF->gradKz[kz].y * vzZ[i].y;

  GB.x = pF->gradKx[kx].y * WxZ[i].x + pF->gradKy[ky].y * WyZ[i].x + pF->gradKz[kz].y * WzZ[i].x;
  GB.y = pF->gradKx[kx].y * WxZ[i].y + pF->gradKy[ky].y * WyZ[i].y + pF->gradKz[kz].y * WzZ[i].y;
  
  if(i==0){
    //vxZ[i].x = WxZ[i].x;
    //vxZ[i].y = WxZ[i].y;
    //vyZ[i].x = WyZ[i].x;
    //vyZ[i].y = WyZ[i].y;
    //vzZ[i].x = WzZ[i].x;
    //vzZ[i].y = WzZ[i].y;
  }
  else{
    vxZ[i].x = (vxZ[i].x + pF->gradKx[kx].y * GW.x / GG) / denominator;
    vxZ[i].y = (vxZ[i].y + pF->gradKx[kx].y * GW.y / GG) / denominator;
    vyZ[i].x = (vyZ[i].x + pF->gradKy[ky].y * GW.x / GG) / denominator;
    vyZ[i].y = (vyZ[i].y + pF->gradKy[ky].y * GW.y / GG) / denominator;
    vzZ[i].x = (vzZ[i].x + pF->gradKz[kz].y * GW.x / GG) / denominator;
    vzZ[i].y = (vzZ[i].y + pF->gradKz[kz].y * GW.y / GG) / denominator;

    WxZ[i].x = (WxZ[i].x + pF->gradKx[kx].y * GB.x / GG) / denominatorB;
    WxZ[i].y = (WxZ[i].y + pF->gradKx[kx].y * GB.y / GG) / denominatorB;
    WyZ[i].x = (WyZ[i].x + pF->gradKy[ky].y * GB.x / GG) / denominatorB;
    WyZ[i].y = (WyZ[i].y + pF->gradKy[ky].y * GB.y / GG) / denominatorB;
    WzZ[i].x = (WzZ[i].x + pF->gradKz[kz].y * GB.x / GG) / denominatorB;
    WzZ[i].y = (WzZ[i].y + pF->gradKz[kz].y * GB.y / GG) / denominatorB;

    // WxZ[i].x = WxZ[i].x / denominatorB;
    // WxZ[i].y = WxZ[i].y / denominatorB;   
    // WyZ[i].x = WyZ[i].x / denominatorB;
    // WyZ[i].y = WyZ[i].y / denominatorB;   
    // WzZ[i].x = WzZ[i].x / denominatorB;
    // WzZ[i].y = WzZ[i].y / denominatorB;   
  }
  
}















__global__ void filterTwoThirds(hipfftDoubleComplex *vxZ,
				hipfftDoubleComplex *vyZ,
				hipfftDoubleComplex *vzZ,
				prefactorsFourier *pF){

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i>=ncellsGPU) return;   

  double pi = 4. * atan(1.);

  // Find mode
  int nx, ny, nz;
  nz = i / (mxGPU*myGPU);
  ny = (i % (mxGPU*myGPU)) / mxGPU;
  nx = i % mxGPU;

  // Find mode in intervale (-pi/L, pi/L)
  double kx = pF->gradKx[nx].y;
  double ky = pF->gradKx[ny].y;
  double kz = pF->gradKx[nz].y;
  double k = kx*kx + ky*ky + kz*kz;  

  // Set k_max to the standard 2/3 rule
  double k_x_max = (2.0/3.0) * pi * invdxGPU; // right value is 2*pi/3.0 * invdxGPU
  double k_y_max = (2.0/3.0) * pi * invdyGPU;
  double k_z_max = (2.0/3.0) * pi * invdzGPU;

  // For k>m_max set modes to zero 
  if((abs(kx) >= k_x_max) || (abs(ky) >= k_y_max) || (abs(kz) >= k_z_max) || (k >= k_x_max*k_y_max)){
    vxZ[i].x = 0;
    vxZ[i].y = 0;
    vyZ[i].x = 0;
    vyZ[i].y = 0;
    vzZ[i].x = 0;
    vzZ[i].y = 0;
  }

  return;
}




__global__ void filterExponential(hipfftDoubleComplex *vxZ,
				  hipfftDoubleComplex *vyZ,
				  hipfftDoubleComplex *vzZ,
				  prefactorsFourier *pF){

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i>=ncellsGPU) return;   

  double pi = 4. * atan(1.);

  // Find mode
  int nx, ny, nz;
  nz = i / (mxGPU*myGPU);
  ny = (i % (mxGPU*myGPU)) / mxGPU;
  nx = i % mxGPU;

  // Find mode in intervale (-pi/L, pi/L)
  double kx = pF->gradKx[nx].y;
  double ky = pF->gradKx[ny].y;
  double kz = pF->gradKx[nz].y;
  double k = kx*kx + ky*ky + kz*kz;  

  // Set k_max to the standard 2/3 rule
  double k_x_max = pi * invdxGPU; // right value is 2*pi/3.0 * invdxGPU
  double k_y_max = pi * invdyGPU;
  double k_z_max = pi * invdzGPU;

  double factor = exp(-36.0 * pow(k/(k_x_max*k_y_max), 36));

  // Scale all modes
  vxZ[i].x *= factor;
  vxZ[i].y *= factor;
  vyZ[i].x *= factor;
  vyZ[i].y *= factor;
  vzZ[i].x *= factor;
  vzZ[i].y *= factor;
  
  return;
}
