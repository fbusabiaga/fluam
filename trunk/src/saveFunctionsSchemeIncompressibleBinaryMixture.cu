#include "hip/hip_runtime.h"
// Filename: saveFunctionsSchemeIncompressibleBinaryMixture.cu
//
// Copyright (c) 2010-2013, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


bool saveFunctionsSchemeIncompressibleBinaryMixture(int index){
  //Initialize save functions
  //cout << "INDEX " << index << endl;
  if(index==0){
    if(!saveSeed()) return 0;
    if(!temperatureFunction(index)) return 0;
    //if(!hydroAnalysisIncompressibleBinaryMixture(0)) return 0;
    if(!saveTime(index)) return 0;
  }
  //Use save functions
  else if(index==1){
    if(!temperatureFunction(index)) return 0;
    if(samplefreq>0) // Call updateHydroGrid
    {
      //if(!hydroAnalysisIncompressibleBinaryMixture(1)) return 0;
    }  
    if((savefreq>0))
    {
      //if((step%savefreq)==0) // Save a snapshot of spectral average data
      //if(!hydroAnalysisIncompressibleBinaryMixture(3)) return 0;
    }
    else if(savefreq<0)
    {
      //if((step%abs(savefreq))==0) // Save a snapshot and do some analysis right now 
      //if(!hydroAnalysisIncompressibleBinaryMixture(4)) return 0;
    }
  }
  //Close save functions
  else if(index==2){
    if(!saveTime(index)) return 0;
    if(!temperatureFunction(index)) return 0;
    //if(!hydroAnalysisIncompressibleBinaryMixture(2)) return 0;
    //if(!saveFluidFinalConfiguration()) return 0; // A. Donev: This takes lots of space so remove it
  }
  else{
    cout << "SAVE FUNCTIONS ERROR, INDEX !=0,1,2 " << endl;
    return 0;
  }
  


  return 1;
}
