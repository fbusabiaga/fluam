#include "hip/hip_runtime.h"
// Filename: createBondedForcesGPU.cu
//
// Copyright (c) 2010-2013, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


__global__ void initBondedForcesVariables(bondedForcesVariables* bFV,
					  int* bondsParticleParticleGPU,
					  int* bondsParticleParticleOffsetGPU,
					  int* bondsIndexParticleParticleGPU,
					  double* r0ParticleParticleGPU,
					  double* kSpringParticleParticleGPU,
					  int* bondsParticleFixedPointGPU,
					  int* bondsParticleFixedPointOffsetGPU,
					  //int* bondsIndexParticleFixedParticleGPU,
					  double* r0ParticleFixedPointGPU,
					  double* kSpringParticleFixedPointGPU,
					  double* rxFixedPointGPU,
					  double* ryFixedPointGPU,
					  double* rzFixedPointGPU){

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i>0) return;   

  bFV->bondsParticleParticleGPU       = bondsParticleParticleGPU;
  bFV->bondsParticleParticleOffsetGPU = bondsParticleParticleOffsetGPU;
  bFV->bondsIndexParticleParticleGPU  = bondsIndexParticleParticleGPU;
  bFV->r0ParticleParticleGPU          = r0ParticleParticleGPU;
  bFV->kSpringParticleParticleGPU     = kSpringParticleParticleGPU;


  bFV->bondsParticleFixedPointGPU       = bondsParticleFixedPointGPU;
  bFV->bondsParticleFixedPointOffsetGPU = bondsParticleFixedPointOffsetGPU;
  //bFV->bondsIndexParticleFixedPointGPU  = bondsIndexParticleFixedPointGPU;
  bFV->r0ParticleFixedPointGPU          = r0ParticleFixedPointGPU;
  bFV->kSpringParticleFixedPointGPU     = kSpringParticleFixedPointGPU;
  bFV->rxFixedPointGPU          = rxFixedPointGPU;
  bFV->ryFixedPointGPU          = ryFixedPointGPU;
  bFV->rzFixedPointGPU          = rzFixedPointGPU;

}




bool createBondedForcesGPU(){

  //Copy constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(bondedForcesGPU),&bondedForces,sizeof(bool));

  /*int aux[np];
  for(int i=0;i<np;i++)
  aux[i]=0;*/

  //Allocate memory
  cutilSafeCall(hipMalloc((void**)&bFV,sizeof(bondedForcesVariables)));
  cutilSafeCall(hipMalloc((void**)&bondsParticleParticleGPU,np*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&bondsParticleParticleOffsetGPU,np*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&bondsIndexParticleParticleGPU,nbondsParticleParticle*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&r0ParticleParticleGPU,nbondsParticleParticle*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&kSpringParticleParticleGPU,nbondsParticleParticle*sizeof(double)));

  //Copy global memory
  cutilSafeCall(hipMemcpy(bondsParticleParticleGPU,bondsParticleParticle,
			   np*sizeof(int),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(bondsParticleParticleOffsetGPU,bondsParticleParticleOffset,
			   np*sizeof(int),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(bondsIndexParticleParticleGPU,bondsIndexParticleParticle,
			   nbondsParticleParticle*sizeof(int),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(r0ParticleParticleGPU,r0ParticleParticle,
			   nbondsParticleParticle*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(kSpringParticleParticleGPU,kSpringParticleParticle,
			   nbondsParticleParticle*sizeof(double),hipMemcpyHostToDevice));





  //Allocate memory
  cutilSafeCall(hipMalloc((void**)&bondsParticleFixedPointGPU,np*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&bondsParticleFixedPointOffsetGPU,np*sizeof(int)));
  //cutilSafeCall(hipMalloc((void**)&bondsIndexParticleFixedPointGPU,nbondsParticleFixedPoint*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&r0ParticleFixedPointGPU,nbondsParticleFixedPoint*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&kSpringParticleFixedPointGPU,nbondsParticleFixedPoint*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&rxFixedPointGPU,nbondsParticleFixedPoint*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&ryFixedPointGPU,nbondsParticleFixedPoint*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&rzFixedPointGPU,nbondsParticleFixedPoint*sizeof(double)));

  //Copy global memory
  cutilSafeCall(hipMemcpy(bondsParticleFixedPointGPU,bondsParticleFixedPoint,
			   np*sizeof(int),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(bondsParticleFixedPointOffsetGPU,bondsParticleFixedPointOffset,
			   np*sizeof(int),hipMemcpyHostToDevice));
  //cutilSafeCall(hipMemcpy(bondsIndexParticleFixedPointGPU,bondsIndexParticleFixedPoint,
  //		   nbondsParticleFixedPoint*sizeof(int),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(r0ParticleFixedPointGPU,r0ParticleFixedPoint,
			   nbondsParticleFixedPoint*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(kSpringParticleFixedPointGPU,kSpringParticleFixedPoint,
			   nbondsParticleFixedPoint*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(rxFixedPointGPU,rxFixedPoint,
			   nbondsParticleFixedPoint*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(ryFixedPointGPU,ryFixedPoint,
			   nbondsParticleFixedPoint*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(rzFixedPointGPU,rzFixedPoint,
			   nbondsParticleFixedPoint*sizeof(double),hipMemcpyHostToDevice));




  /*cutilSafeCall(hipMemcpy(bondsParticleParticleGPU,aux,np*sizeof(int),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(bondsParticleParticleOffsetGPU,aux,
			   np*sizeof(int),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(bondsIndexParticleParticleGPU,aux,
  nbondsParticleParticle*sizeof(int),hipMemcpyHostToDevice));*/




  initBondedForcesVariables<<<1,1>>>(bFV,
				     bondsParticleParticleGPU,
				     bondsParticleParticleOffsetGPU,
				     bondsIndexParticleParticleGPU,
				     r0ParticleParticleGPU,
				     kSpringParticleParticleGPU,
				     bondsParticleFixedPointGPU,
				     bondsParticleFixedPointOffsetGPU,
				     //bondsIndexParticleFixedPointGPU,
				     r0ParticleFixedPointGPU,
				     kSpringParticleFixedPointGPU,
				     rxFixedPointGPU,
				     ryFixedPointGPU,
				     rzFixedPointGPU);




  return 1;
}
