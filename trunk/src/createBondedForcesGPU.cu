#include "hip/hip_runtime.h"
// Filename: createBondedForcesGPU.cu
//
// Copyright (c) 2010-2012, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


__global__ void initBondedForcesVariables(bondedForcesVariables* bFV,
					  int* bondsParticleParticleGPU,
					  int* bondsParticleParticleOffsetGPU,
					  int* bondsIndexParticleParticleGPU,
					  double* r0ParticleParticleGPU,
					  double* kSpringParticleParticleGPU,
					  int* bondsParticleFixedPointGPU,
					  int* bondsParticleFixedPointOffsetGPU,
					  //int* bondsIndexParticleFixedParticleGPU,
					  double* r0ParticleFixedPointGPU,
					  double* kSpringParticleFixedPointGPU,
					  double* rxFixedPointGPU,
					  double* ryFixedPointGPU,
					  double* rzFixedPointGPU){

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i>0) return;   

  bFV->bondsParticleParticleGPU       = bondsParticleParticleGPU;
  bFV->bondsParticleParticleOffsetGPU = bondsParticleParticleOffsetGPU;
  bFV->bondsIndexParticleParticleGPU  = bondsIndexParticleParticleGPU;
  bFV->r0ParticleParticleGPU          = r0ParticleParticleGPU;
  bFV->kSpringParticleParticleGPU     = kSpringParticleParticleGPU;


  bFV->bondsParticleFixedPointGPU       = bondsParticleFixedPointGPU;
  bFV->bondsParticleFixedPointOffsetGPU = bondsParticleFixedPointOffsetGPU;
  //bFV->bondsIndexParticleFixedPointGPU  = bondsIndexParticleFixedPointGPU;
  bFV->r0ParticleFixedPointGPU          = r0ParticleFixedPointGPU;
  bFV->kSpringParticleFixedPointGPU     = kSpringParticleFixedPointGPU;
  bFV->rxFixedPointGPU          = rxFixedPointGPU;
  bFV->ryFixedPointGPU          = ryFixedPointGPU;
  bFV->rzFixedPointGPU          = rzFixedPointGPU;

}




bool createBondedForcesGPU(){

  //Copy constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(bondedForcesGPU),&bondedForces,sizeof(bool));

  /*int aux[np];
  for(int i=0;i<np;i++)
  aux[i]=0;*/

  //Allocate memory
  hipMalloc((void**)&bFV,sizeof(bondedForcesVariables));
  hipMalloc((void**)&bondsParticleParticleGPU,np*sizeof(int));
  hipMalloc((void**)&bondsParticleParticleOffsetGPU,np*sizeof(int));
  hipMalloc((void**)&bondsIndexParticleParticleGPU,nbondsParticleParticle*sizeof(int));
  hipMalloc((void**)&r0ParticleParticleGPU,nbondsParticleParticle*sizeof(double));
  hipMalloc((void**)&kSpringParticleParticleGPU,nbondsParticleParticle*sizeof(double));

  //Copy global memory
  hipMemcpy(bondsParticleParticleGPU,bondsParticleParticle,
			   np*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(bondsParticleParticleOffsetGPU,bondsParticleParticleOffset,
			   np*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(bondsIndexParticleParticleGPU,bondsIndexParticleParticle,
			   nbondsParticleParticle*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(r0ParticleParticleGPU,r0ParticleParticle,
			   nbondsParticleParticle*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(kSpringParticleParticleGPU,kSpringParticleParticle,
			   nbondsParticleParticle*sizeof(double),hipMemcpyHostToDevice);





  //Allocate memory
  hipMalloc((void**)&bondsParticleFixedPointGPU,np*sizeof(int));
  hipMalloc((void**)&bondsParticleFixedPointOffsetGPU,np*sizeof(int));
  //hipMalloc((void**)&bondsIndexParticleFixedPointGPU,nbondsParticleFixedPoint*sizeof(int));
  hipMalloc((void**)&r0ParticleFixedPointGPU,nbondsParticleFixedPoint*sizeof(double));
  hipMalloc((void**)&kSpringParticleFixedPointGPU,nbondsParticleFixedPoint*sizeof(double));
  hipMalloc((void**)&rxFixedPointGPU,nbondsParticleFixedPoint*sizeof(double));
  hipMalloc((void**)&ryFixedPointGPU,nbondsParticleFixedPoint*sizeof(double));
  hipMalloc((void**)&rzFixedPointGPU,nbondsParticleFixedPoint*sizeof(double));

  //Copy global memory
  hipMemcpy(bondsParticleFixedPointGPU,bondsParticleFixedPoint,
			   np*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(bondsParticleFixedPointOffsetGPU,bondsParticleFixedPointOffset,
			   np*sizeof(int),hipMemcpyHostToDevice);
  //hipMemcpy(bondsIndexParticleFixedPointGPU,bondsIndexParticleFixedPoint,
  //		   nbondsParticleFixedPoint*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(r0ParticleFixedPointGPU,r0ParticleFixedPoint,
			   nbondsParticleFixedPoint*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(kSpringParticleFixedPointGPU,kSpringParticleFixedPoint,
			   nbondsParticleFixedPoint*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(rxFixedPointGPU,rxFixedPoint,
			   nbondsParticleFixedPoint*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(ryFixedPointGPU,ryFixedPoint,
			   nbondsParticleFixedPoint*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(rzFixedPointGPU,rzFixedPoint,
			   nbondsParticleFixedPoint*sizeof(double),hipMemcpyHostToDevice);




  /*cutilSafeCall(hipMemcpy(bondsParticleParticleGPU,aux,np*sizeof(int),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(bondsParticleParticleOffsetGPU,aux,
			   np*sizeof(int),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(bondsIndexParticleParticleGPU,aux,
  nbondsParticleParticle*sizeof(int),hipMemcpyHostToDevice));*/




  initBondedForcesVariables<<<1,1>>>(bFV,
				     bondsParticleParticleGPU,
				     bondsParticleParticleOffsetGPU,
				     bondsIndexParticleParticleGPU,
				     r0ParticleParticleGPU,
				     kSpringParticleParticleGPU,
				     bondsParticleFixedPointGPU,
				     bondsParticleFixedPointOffsetGPU,
				     //bondsIndexParticleFixedPointGPU,
				     r0ParticleFixedPointGPU,
				     kSpringParticleFixedPointGPU,
				     rxFixedPointGPU,
				     ryFixedPointGPU,
				     rzFixedPointGPU);




  return 1;
}
