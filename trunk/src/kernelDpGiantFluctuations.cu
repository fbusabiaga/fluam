#include "hip/hip_runtime.h"
// Filename: kernelDpGiantFluctuations.cu
//
// Copyright (c) 2010-2013, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


__global__ void kernelDpGiantFluctuations_1(double* densityGPU, 
					    double* densityGPU2,
					    double* vxGPU, 
					    double* vyGPU,
					    double* vzGPU,
					    double* dmGPU,
					    double* dpxGPU, 
					    double* dpyGPU, 
					    double* dpzGPU,
					    double* cGPU, 
					    double* cGPU2, 
					    double* dcGPU,
					    double* d_rand,
					    int* ghostIndex, 
					    int* realIndex,
					    int substep, 
					    double RK1, 
					    double RK2, 
					    double RK3){
  
  int j = blockDim.x * blockIdx.x + threadIdx.x;
  if(j>=ncellsGPU) return;   
  int i = ghostIndex[j];

  double dm;
  double pressure, pressure3, pressure4, pressure5;
  double vx, vy, vz;

  double density = densityGPU[i];
  pressure = pressure_GPU(density);

  int vecino0, vecino1, vecino2, vecino3, vecino4, vecino5; 

  int vecinopxpy, vecinopxmy, vecinopxpz, vecinopxmz;
  int vecinomxpy, vecinomxpz;
  int vecinopypz, vecinopymz, vecinomypz;
  int vecinopxpx, vecinopypy, vecinopzpz;

  vecino0 = i - mxmytGPU;
  vecino1 = i - mxtGPU;
  vecino2 = i - 1;
  vecino3 = i + 1;
  vecinopxpx = ghostIndex[ realIndex[vecino3] ] + 1;
  vecino4 = i + mxtGPU;
  vecinopypy = ghostIndex[ realIndex[vecino4] ] + mxtGPU;
  vecino5 = i + mxmytGPU;
  vecinopzpz = ghostIndex[ realIndex[vecino5] ] + mxmytGPU;
  
  vecinopxpy = i + 1 + mxtGPU;
  vecinopxmy = i + 1 - mxtGPU;
  vecinopxpz = i + 1 + mxmytGPU;
  vecinopxmz = i + 1 - mxmytGPU;
  vecinomxpy = i - 1 + mxtGPU;
  vecinomxpz = i - 1 + mxmytGPU;
  vecinopypz = i + mxtGPU + mxmytGPU;
  vecinopymz = i + mxtGPU - mxmytGPU;
  vecinomypz = i - mxtGPU + mxmytGPU;


  double vx0, vx1, vx2, vx3, vx4, vx5;
  double vy0, vy1, vy2, vy3, vy4, vy5;
  double vz0, vz1, vz2, vz3, vz4, vz5;

  double vxmxpy,vxmxpz;
  double vypxmy,vymypz;
  double vzpxmz,vzpymz;

  vx = fetch_double(texVxGPU,i);
  vy = fetch_double(texVyGPU,i);
  vz = fetch_double(texVzGPU,i);
  vx0 = fetch_double(texVxGPU,vecino0);
  vx1 = fetch_double(texVxGPU,vecino1);
  vx2 = fetch_double(texVxGPU,vecino2);
  vx3 = fetch_double(texVxGPU,vecino3);
  vx4 = fetch_double(texVxGPU,vecino4);
  vx5 = fetch_double(texVxGPU,vecino5);
  vy0 = fetch_double(texVyGPU,vecino0);
  vy1 = fetch_double(texVyGPU,vecino1);
  vy2 = fetch_double(texVyGPU,vecino2);
  vy3 = fetch_double(texVyGPU,vecino3);
  vy4 = fetch_double(texVyGPU,vecino4);
  vy5 = fetch_double(texVyGPU,vecino5);
  vz0 = fetch_double(texVzGPU,vecino0);
  vz1 = fetch_double(texVzGPU,vecino1);
  vz2 = fetch_double(texVzGPU,vecino2);
  vz3 = fetch_double(texVzGPU,vecino3);
  vz4 = fetch_double(texVzGPU,vecino4);
  vz5 = fetch_double(texVzGPU,vecino5);

  vxmxpy = fetch_double(texVxGPU,vecinomxpy);
  vxmxpz = fetch_double(texVxGPU,vecinomxpz);
  vypxmy = fetch_double(texVyGPU,vecinopxmy);
  vymypz = fetch_double(texVyGPU,vecinomypz);
  vzpxmz = fetch_double(texVzGPU,vecinopxmz);
  vzpymz = fetch_double(texVzGPU,vecinopymz);

  double density0, density1, density2;
  volatile double density3, density4, density5;

  density0 = densityGPU[vecino0];
  density1 = densityGPU[vecino1];
  density2 = densityGPU[vecino2];
  density3 = densityGPU[vecino3];
  pressure3 = pressure_GPU(density3);
  density4 = densityGPU[vecino4];
  pressure4 = pressure_GPU(density4);
  density5 = densityGPU[vecino5];
  pressure5 = pressure_GPU(density5);


  dm = 
    invdxGPU * ((density3 + density) * vx - (density + density2) * vx2) +
    invdyGPU * ((density4 + density) * vy - (density + density1) * vy1) +
    invdzGPU * ((density5 + density) * vz - (density + density0) * vz0);

  dm = -0.5 * dm;

  double sXX, sXY, sXZ;
  double sYX, sYY, sYZ;
  double sZX, sZY, sZZ;

  sXX = pressure3 - pressure;
  sYY = pressure4 - pressure;
  sZZ = pressure5 - pressure;

  sXX += 0.125 * ((density3+density)*vx + (densityGPU[vecinopxpx]+density3)*vx3)*(vx+vx3);
  sXX -= 0.125 * ((density+density2)*vx2 + (density3+density)*vx)*(vx2+vx);

  sYY += 0.125 * ((density4+density)*vy + (densityGPU[vecinopypy]+density4)*vy4)*(vy+vy4);
  sYY -= 0.125 * ((density+density1)*vy1 + (density4+density)*vy)*(vy1+vy);

  sZZ += 0.125 * ((density5+density)*vz + (densityGPU[vecinopzpz]+density5)*vz5)*(vz+vz5);
  sZZ -= 0.125 * ((density+density0)*vz0 + (density5+density)*vz)*(vz0+vz);

  double densitypxpy, densitypxmy, densitypxpz, densitypxmz;
  double densitymxpy, densitymxpz;
  double densitypypz, densitypymz, densitymypz;

  densitypxpy = densityGPU[vecinopxpy];
  densitypxmy = densityGPU[vecinopxmy];
  densitypxpz = densityGPU[vecinopxpz];
  densitypxmz = densityGPU[vecinopxmz];
  densitymxpy = densityGPU[vecinomxpy];
  densitymxpz = densityGPU[vecinomxpz];
  densitypypz = densityGPU[vecinopypz];
  densitypymz = densityGPU[vecinopymz];
  densitymypz = densityGPU[vecinomypz];

  
  sXY  = 0.125 * ((density4 + density)  * vy  + (densitypxpy + density3)    * vy3)    * (vx  + vx4);
  sXY -= 0.125 * ((density  + density1) * vy1 + (density3    + densitypxmy) * vypxmy) * (vx1 + vx);
  sXZ  = 0.125 * ((density5 + density)  * vz  + (densitypxpz + density3)    * vz3)    * (vx  + vx5);
  sXZ -= 0.125 * ((density  + density0) * vz0 + (density3    + densitypxmz) * vzpxmz) * (vx0 + vx);
  sYX  = 0.125 * ((density3 + density)  * vx  + (densitypxpy + density4)    * vx4)    * (vy  + vy3);
  sYX -= 0.125 * ((density  + density2) * vx2 + (density4    + densitymxpy) * vxmxpy) * (vy2 + vy);
  sYZ  = 0.125 * ((density5 + density)  * vz  + (densitypypz + density4)    * vz4)    * (vy  + vy5);
  sYZ -= 0.125 * ((density  + density0) * vz0 + (density4    + densitypymz) * vzpymz) * (vy0 + vy);
  sZX  = 0.125 * ((density3 + density)  * vx  + (densitypxpz + density5)    * vx5)    * (vz  + vz3);
  sZX -= 0.125 * ((density  + density2) * vx2 + (density5    + densitymxpz) * vxmxpz) * (vz2 + vz);
  sZY  = 0.125 * ((density4 + density)  * vy  + (densitypypz + density5)    * vy5)    * (vz  + vz4);
  sZY -= 0.125 * ((density  + density1) * vy1 + (density5    + densitymypz) * vymypz) * (vz1 + vz);

  //STRESS TENSOR CONTRIBUTION
  sXX -= 2. * shearviscosityGPU * invdxGPU * (vx3 - vx) + fact3GPU *
    (invdxGPU * (vx3 - vx) + invdyGPU * (vy3 - vypxmy) + invdzGPU * (vz3 - vzpxmz));
  sXX += 2. * shearviscosityGPU * invdxGPU * (vx - vx2) + fact3GPU *
    (invdxGPU * (vx - vx2) + invdyGPU * (vy - vy1) + invdzGPU * (vz - vz0));
  sYY -= 2. * shearviscosityGPU * invdyGPU * (vy4 - vy) + fact3GPU * 
    (invdxGPU * (vx4 - vxmxpy) + invdyGPU * (vy4 - vy) + invdzGPU * (vz4 - vzpymz));
  sYY += 2. * shearviscosityGPU * invdyGPU * (vy - vy1) + fact3GPU *
    (invdxGPU * (vx - vx2) + invdyGPU * (vy - vy1) + invdzGPU * (vz - vz0));
  sZZ -= 2. * shearviscosityGPU * invdzGPU * (vz5 - vz) + fact3GPU *
    (invdxGPU * (vx5 - vxmxpz) + invdyGPU * (vy5 - vymypz) + invdzGPU * (vz5 -vz));
  sZZ += 2. * shearviscosityGPU * invdzGPU * (vz - vz0) + fact3GPU *
    (invdxGPU * (vx - vx2) + invdyGPU * (vy - vy1) + invdzGPU * (vz - vz0));

  sXY -= shearviscosityGPU * (invdyGPU * (vx4 - vx) + invdxGPU * (vy3 - vy));
  sXY += shearviscosityGPU * (invdyGPU * (vx - vx1) + invdxGPU * (vypxmy - vy1));
  sXZ -= shearviscosityGPU * (invdzGPU * (vx5 - vx) + invdxGPU * (vz3 - vz));
  sXZ += shearviscosityGPU * (invdzGPU * (vx - vx0) + invdxGPU * (vzpxmz - vz0));
  sYX -= shearviscosityGPU * (invdxGPU * (vy3 - vy) + invdyGPU * (vx4 - vx));
  sYX += shearviscosityGPU * (invdxGPU * (vy - vy2) + invdyGPU * (vxmxpy - vx2));
  sYZ -= shearviscosityGPU * (invdzGPU * (vy5 - vy) + invdyGPU * (vz4 - vz));
  sYZ += shearviscosityGPU * (invdzGPU * (vy - vy0) + invdyGPU * (vzpymz - vz0));
  sZX -= shearviscosityGPU * (invdxGPU * (vz3 - vz) + invdzGPU * (vx5 - vx));
  sZX += shearviscosityGPU * (invdxGPU * (vz - vz2) + invdzGPU * (vxmxpz - vx2));
  sZY -= shearviscosityGPU * (invdyGPU * (vz4 - vz) + invdzGPU * (vy5 - vy));
  sZY += shearviscosityGPU * (invdyGPU * (vz - vz1) + invdzGPU * (vymypz - vy1));

  //Noise contribution
  int n0;
  n0 = substep * ncellstGPU * 18;
  double dnoise_sXX, dnoise_sXY, dnoise_sXZ;
  double dnoise_sYY, dnoise_sYZ;
  double dnoise_sZZ;
  double dnoise_tr;
  double fact1, fact2, fact4;
  fact1 = fact1GPU;
  fact2 = fact2GPU;
  fact4 = fact4GPU;
  
  dnoise_tr = d_rand[n0 + vecino3] + d_rand[n0 + vecino3 + 3*ncellstGPU] + d_rand[n0 + vecino3 + 5*ncellstGPU];
  dnoise_sXX = d_rand[n0 + vecino3] - dnoise_tr/3.;
  sXX += fact1 * dnoise_sXX + fact2 * dnoise_tr;
  
  dnoise_tr = d_rand[n0 + vecino4] + d_rand[n0 + vecino4 + 3*ncellstGPU] + d_rand[n0 + vecino4 + 5*ncellstGPU];
  dnoise_sYY = d_rand[n0 + vecino4 + 3*ncellstGPU] - dnoise_tr/3.;
  sYY += fact1 * dnoise_sYY + fact2 * dnoise_tr;
  
  dnoise_tr = d_rand[n0 + vecino5] + d_rand[n0 + vecino5 + 3*ncellstGPU] + d_rand[n0 + vecino5 + 5*ncellstGPU];
  dnoise_sZZ = d_rand[n0 + vecino5 + 5*ncellstGPU] - dnoise_tr/3.;
  sZZ += fact1 * dnoise_sZZ + fact2 * dnoise_tr;
  
  dnoise_sXY = d_rand[n0 + i + ncellstGPU];
  sXY += fact4 * dnoise_sXY;
  sYX += fact4 * dnoise_sXY;
  
  dnoise_sXZ = d_rand[n0 + i + 2*ncellstGPU];
  sXZ += fact4 * dnoise_sXZ;
  sZX += fact4 * dnoise_sXZ;
  
  dnoise_sYZ = d_rand[n0 + i + 4*ncellstGPU];
  sYZ += fact4 * dnoise_sYZ;
  sZY += fact4 * dnoise_sYZ;
  
  dnoise_tr = d_rand[n0 + i] + d_rand[n0 + i + 3*ncellstGPU] + d_rand[n0 + i + 5*ncellstGPU];
  dnoise_sXX = d_rand[n0 + i] - dnoise_tr/3.;
  sXX -= fact1 * dnoise_sXX + fact2 * dnoise_tr;
  
  dnoise_sYY = d_rand[n0 + i + 3*ncellstGPU] - dnoise_tr/3.;
  sYY -= fact1 * dnoise_sYY + fact2 * dnoise_tr;
  
  dnoise_sZZ = d_rand[n0 + i + 5*ncellstGPU] - dnoise_tr/3.;
  sZZ -= fact1 * dnoise_sZZ + fact2 * dnoise_tr;
  
  dnoise_sXY = d_rand[n0 + vecino1 + ncellstGPU];
  sXY -= fact4 * dnoise_sXY;
  
  dnoise_sXZ = d_rand[n0 + vecino0 + 2*ncellstGPU];
  sXZ -= fact4 * dnoise_sXZ;
  
  dnoise_sXY = d_rand[n0 + vecino2 + ncellstGPU];
  sYX -= fact4 * dnoise_sXY;
  
  dnoise_sYZ = d_rand[n0 + vecino0 + 4*ncellstGPU];
  sYZ -= fact4 * dnoise_sYZ;
  
  dnoise_sXZ = d_rand[n0 + vecino2 + 2*ncellstGPU];
  sZX -= fact4 * dnoise_sXZ;
  
  dnoise_sYZ = d_rand[n0 + vecino1 + 4*ncellstGPU];
  sZY -= fact4 * dnoise_sYZ;
  
  
  if(RK3 != 0){
    n0 += ncellstGPU * 9;
    fact1 = RK3 * fact1GPU;
    fact2 = RK3 * fact2GPU;
    fact4 = RK3 * fact4GPU;
    dnoise_tr = d_rand[n0 + vecino3] + d_rand[n0 + vecino3 + 3*ncellstGPU] + d_rand[n0 + vecino3 + 5*ncellstGPU];
    dnoise_sXX = d_rand[n0 + vecino3] - dnoise_tr/3.;
    sXX += fact1 * dnoise_sXX + fact2 * dnoise_tr;
    
    dnoise_tr = d_rand[n0 + vecino4] + d_rand[n0 + vecino4 + 3*ncellstGPU] + d_rand[n0 + vecino4 + 5*ncellstGPU];
    dnoise_sYY = d_rand[n0 + vecino4 + 3*ncellstGPU] - dnoise_tr/3.;
    sYY += fact1 * dnoise_sYY + fact2 * dnoise_tr;
    
    dnoise_tr = d_rand[n0 + vecino5] + d_rand[n0 + vecino5 + 3*ncellstGPU] + d_rand[n0 + vecino5 + 5*ncellstGPU];
    dnoise_sZZ = d_rand[n0 + vecino5 + 5*ncellstGPU] - dnoise_tr/3.;
    sZZ += fact1 * dnoise_sZZ + fact2 * dnoise_tr;
    
    dnoise_sXY = d_rand[n0 + i + ncellstGPU];
    sXY += fact4 * dnoise_sXY;
    sYX += fact4 * dnoise_sXY;
    
    dnoise_sXZ = d_rand[n0 + i + 2*ncellstGPU];
    sXZ += fact4 * dnoise_sXZ;
    sZX += fact4 * dnoise_sXZ;
    
    dnoise_sYZ = d_rand[n0 + i + 4*ncellstGPU];
    sYZ += fact4 * dnoise_sYZ;
    sZY += fact4 * dnoise_sYZ;
    
    dnoise_tr = d_rand[n0 + i] + d_rand[n0 + i + 3*ncellstGPU] + d_rand[n0 + i + 5*ncellstGPU];
    dnoise_sXX = d_rand[n0 + i] - dnoise_tr/3.;
    sXX -= fact1 * dnoise_sXX + fact2 * dnoise_tr;
    
    dnoise_sYY = d_rand[n0 + i + 3*ncellstGPU] - dnoise_tr/3.;
    sYY -= fact1 * dnoise_sYY + fact2 * dnoise_tr;
    
    dnoise_sZZ = d_rand[n0 + i + 5*ncellstGPU] - dnoise_tr/3.;
    sZZ -= fact1 * dnoise_sZZ + fact2 * dnoise_tr;
    
    dnoise_sXY = d_rand[n0 + vecino1 + ncellstGPU];
    sXY -= fact4 * dnoise_sXY;
    
    dnoise_sXZ = d_rand[n0 + vecino0 + 2*ncellstGPU];
    sXZ -= fact4 * dnoise_sXZ;
    
    dnoise_sXY = d_rand[n0 + vecino2 + ncellstGPU];
    sYX -= fact4 * dnoise_sXY;
    
    dnoise_sYZ = d_rand[n0 + vecino0 + 4*ncellstGPU];
    sYZ -= fact4 * dnoise_sYZ;
    
    dnoise_sXZ = d_rand[n0 + vecino2 + 2*ncellstGPU];
    sZX -= fact4 * dnoise_sXZ;
    
    dnoise_sYZ = d_rand[n0 + vecino1 + 4*ncellstGPU];
    sZY -= fact4 * dnoise_sYZ;
  }
  
  
  double px = vxGPU[i] * 0.5 * (densityGPU2[i] + densityGPU2[vecino3]) * RK1;
  double py = vyGPU[i] * 0.5 * (densityGPU2[i] + densityGPU2[vecino4]) * RK1;
  double pz = vzGPU[i] * 0.5 * (densityGPU2[i] + densityGPU2[vecino5]) * RK1;

  px += vx * 0.5 * (density + density3) * RK2;
  py += vy * 0.5 * (density + density4) * RK2;
  pz += vz * 0.5 * (density + density5) * RK2;

  
  
  px += -(invdxGPU * sXX + invdyGPU * sXY + invdzGPU * sXZ)*dtGPU*RK2;
  py += -(invdxGPU * sYX + invdyGPU * sYY + invdzGPU * sYZ)*dtGPU*RK2;
  pz += -(invdxGPU * sZX + invdyGPU * sZY + invdzGPU * sZZ)*dtGPU*RK2;

  dmGPU[i] = RK1 * densityGPU2[i] + RK2 * (density + dm * dtGPU);
  
  dpxGPU[i] = px;
  dpyGPU[i] = py;
  dpzGPU[i] = pz;




  //Concentration
  //Concentration
  double dc;
  //double pressure2, pressure1, pressure0;
  //pressure0 = pressure_GPU(density0);
  //pressure1 = pressure_GPU(density1);
  //pressure2 = pressure_GPU(density2);
  dc = 0;
  
  dc = invdxGPU * ((cGPU[vecino3]*density3 + cGPU[i]*density) * vx
		   -(cGPU[i]*density + cGPU[vecino2]*density2) * vx2) +
    invdyGPU * ((cGPU[vecino4]*density4 + cGPU[i]*density) * vy
		-(cGPU[i]*density + cGPU[vecino1]*density1) * vy1) +
    invdzGPU * ((cGPU[vecino5]*density5 + cGPU[i]*density) * vz
		-(cGPU[i]*density + cGPU[vecino0]*density0) * vz0);
  
  dc -= invdxGPU*invdxGPU*((funcDiffusionBM(vecino3) + funcDiffusionBM(i))*
			   (cGPU[vecino3] - cGPU[i]) -
			   (funcDiffusionBM(i) + funcDiffusionBM(vecino2))*
			   (cGPU[i] - cGPU[vecino2])) +
    invdyGPU*invdyGPU*((funcDiffusionBM(vecino4) + funcDiffusionBM(i))*
		       (cGPU[vecino4] - cGPU[i]) -
		       (funcDiffusionBM(i) + funcDiffusionBM(vecino1))*
		       (cGPU[i] - cGPU[vecino1])) +
    invdzGPU*invdzGPU*((funcDiffusionBM(vecino5) + funcDiffusionBM(i))*
		       (cGPU[vecino5] - cGPU[i]) -
		       (funcDiffusionBM(i) + funcDiffusionBM(vecino0))*
		       (cGPU[i] - cGPU[vecino0]));
  
  dc -= invdyGPU * funcDiffusionBM(i) * ((cGPU[vecino4]+cGPU[i])-(cGPU[i]+cGPU[vecino1])) *
    soretCoefficientGPU * gradTemperatureGPU;
  
  dc = -0.5 * dc;

  /*
  dc += fact5GPU * (
    invdxGPU*invdxGPU*((baroDiffusion(pressure3,cGPU[vecino3]) + baroDiffusion(pressure,cGPU[i]))*
			   (pressure3 - pressure) -
 			   (baroDiffusion(pressure,cGPU[i]) + baroDiffusion(pressure2,cGPU[vecino2]))*
			   (pressure - pressure2)) +
    invdyGPU*invdyGPU*((baroDiffusion(pressure4,cGPU[vecino4]) + baroDiffusion(pressure,cGPU[i]))*
		       (pressure4 - pressure) -
		       (baroDiffusion(pressure,cGPU[i]) + baroDiffusion(pressure1,cGPU[vecino1]))*
		       (pressure - pressure1)) +
    invdzGPU*invdzGPU*((baroDiffusion(pressure5,cGPU[vecino5]) + baroDiffusion(pressure,cGPU[i]))*
		       (pressure5 - pressure) -
		       (baroDiffusion(pressure,cGPU[i]) + baroDiffusion(pressure0,cGPU[vecino0]))*
		       (pressure - pressure0)));
  */
  //Noise terms
  n0 = substep * ncellstGPU * 18;
  double c = 0.5 * (cGPU[vecino3] + cGPU[i]);
  dc += fact5GPU*invdxGPU * sqrt(2*funcDiffusionBM(i)*c*(1-c)*(massSpecies1GPU*(1-c) + massSpecies0GPU*c))*d_rand[n0 + i + 6*ncellstGPU];
  c = 0.5 * (cGPU[i] + cGPU[vecino2]);
  dc -= fact5GPU*invdxGPU * sqrt(2*funcDiffusionBM(i)*c*(1-c)*(massSpecies1GPU*(1-c) + massSpecies0GPU*c))*d_rand[n0 + vecino2 + 6*ncellstGPU];

  c = 0.5 * (cGPU[i] + cGPU[vecino4]);
  dc += fact5GPU*invdyGPU * sqrt(2*funcDiffusionBM(i)*c*(1-c)*(massSpecies1GPU*(1-c) + massSpecies0GPU*c))*d_rand[n0 + i + 7*ncellstGPU];
  c = 0.5 * (cGPU[i] + cGPU[vecino1]);
  dc -= fact5GPU*invdyGPU * sqrt(2*funcDiffusionBM(i)*c*(1-c)*(massSpecies1GPU*(1-c) + massSpecies0GPU*c))*d_rand[n0 + vecino1 + 7*ncellstGPU];

  c = 0.5 * (cGPU[i] + cGPU[vecino5]);
  dc += fact5GPU*invdzGPU * sqrt(2*funcDiffusionBM(i)*c*(1-c)*(massSpecies1GPU*(1-c) + massSpecies0GPU*c))*d_rand[n0 + i + 8*ncellstGPU];
  c = 0.5 * (cGPU[i] + cGPU[vecino0]);
  dc -= fact5GPU*invdzGPU * sqrt(2*funcDiffusionBM(i)*c*(1-c)*(massSpecies1GPU*(1-c) + massSpecies0GPU*c))*d_rand[n0 + vecino0 + 8*ncellstGPU];

  if(RK3 != 0){
    n0 += ncellstGPU * 9;
    c = 0.5 * (cGPU[vecino3] + cGPU[i]);
    dc += fact5GPU*RK3*invdxGPU * sqrt(2*funcDiffusionBM(i)*c*(1-c)*(massSpecies1GPU*(1-c)+massSpecies0GPU*c))*d_rand[n0 + i + 6*ncellstGPU];
    c = 0.5 * (cGPU[i] + cGPU[vecino2]);
    dc -= fact5GPU*RK3*invdxGPU * sqrt(2*funcDiffusionBM(i)*c*(1-c)*(massSpecies1GPU*(1-c)+massSpecies0GPU*c))*d_rand[n0 + vecino2 + 6*ncellstGPU];
    
    c = 0.5 * (cGPU[i] + cGPU[vecino4]);
    dc += fact5GPU*RK3*invdyGPU * sqrt(2*funcDiffusionBM(i)*c*(1-c)*(massSpecies1GPU*(1-c) + massSpecies0GPU*c))*d_rand[n0 + i + 7*ncellstGPU];
    c = 0.5 * (cGPU[i] + cGPU[vecino1]);
    dc -= fact5GPU*RK3*invdyGPU * sqrt(2*funcDiffusionBM(i)*c*(1-c)*(massSpecies1GPU*(1-c) + massSpecies0GPU*c))*d_rand[n0 + vecino1 + 7*ncellstGPU];
    
    c = 0.5 * (cGPU[i] + cGPU[vecino5]);
    dc += fact5GPU*RK3*invdzGPU * sqrt(2*funcDiffusionBM(i)*c*(1-c)*(massSpecies1GPU*(1-c) + massSpecies0GPU*c))*d_rand[n0 + i + 8*ncellstGPU];
    c = 0.5 * (cGPU[i] + cGPU[vecino0]);
    dc -= fact5GPU*RK3*invdzGPU * sqrt(2*funcDiffusionBM(i)*c*(1-c)*(massSpecies1GPU*(1-c) + massSpecies0GPU*c))*d_rand[n0 + vecino0 + 8*ncellstGPU];
  }
  
  
  dcGPU[i] = RK1 * densityGPU2[i] * cGPU2[i] + RK2 * (density*cGPU[i] + dc * dtGPU);


  
  

















}

__global__ void kernelDpGiantFluctuations_2(double* densityGPU,
					    double* vxGPU, 
					    double* vyGPU,
					    double* vzGPU,
					    double* dmGPU,
					    double* dpxGPU, 
					    double* dpyGPU, 
					    double* dpzGPU,
					    double* cGPU, 
					    double* dcGPU,
					    int* ghostIndex, 
					    int* realIndex){
  

  int j = blockDim.x * blockIdx.x + threadIdx.x;
  if(j>=ncellsGPU) return;   
  int i = ghostIndex[j];
  
  int vecino3, vecino4, vecino5;
  vecino3 = ghostIndex[ realIndex[i + 1] ];
  vecino4 = ghostIndex[ realIndex[i + mxtGPU ] ];
  vecino5 = ghostIndex[ realIndex[i + mxmytGPU] ];


  densityGPU[i] = dmGPU[i];


  vxGPU[i] = dpxGPU[i] * 2. / (dmGPU[i] + dmGPU[vecino3]);
  vyGPU[i] = dpyGPU[i] * 2. / (dmGPU[i] + dmGPU[vecino4]);
  vzGPU[i] = dpzGPU[i] * 2. / (dmGPU[i] + dmGPU[vecino5]);

  cGPU[i] = dcGPU[i] / dmGPU[i];

}
