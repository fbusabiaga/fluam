// Filename: copyToGPUGiantFluctuations.cu
//
// Copyright (c) 2010-2012, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


bool copyToGPUGiantFluctuations(){
  hipMemcpyToSymbol(HIP_SYMBOL(cWall0GPU),&cWall0,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(cWall1GPU),&cWall1,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(densityWall0GPU),&densityWall0,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(densityWall1GPU),&densityWall1,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(vxWall0GPU),&vxWall0,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(vxWall1GPU),&vxWall1,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(vyWall0GPU),&vyWall0,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(vyWall1GPU),&vyWall1,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(vzWall0GPU),&vzWall0,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(vzWall1GPU),&vzWall1,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(soretCoefficientGPU),&soretCoefficient,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(gradTemperatureGPU),&gradTemperature,sizeof(double));

  cout << "COPY TO GPU :                   DONE" << endl;

  return 1;
}
