// Filename: gpuToHostStokesLimit.cu
//
// Copyright (c) 2010-2012, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


bool gpuToHostStokesLimit(){

  int auxb[5];
  cutilSafeCall(hipMemcpy(auxb,errorKernel,5*sizeof(int),hipMemcpyDeviceToHost));
  if(auxb[0] == 1){
    for(int i=0;i<5;i++){
      cout << "ERROR IN KERNEL " << i << " " << auxb[i] << endl;
    }
    return 0;
  }

  //cutilSafeCall(hipMemcpy(cDensity,densityGPU,ncellst*sizeof(double),hipMemcpyDeviceToHost));
  //cutilSafeCall(hipMemcpy(cvx,vxGPU,ncellst*sizeof(double),hipMemcpyDeviceToHost));
  //cutilSafeCall(hipMemcpy(cvy,vyGPU,ncellst*sizeof(double),hipMemcpyDeviceToHost));
  //cutilSafeCall(hipMemcpy(cvz,vzGPU,ncellst*sizeof(double),hipMemcpyDeviceToHost));

  if(setparticles){
    cutilSafeCall(hipMemcpy(rxParticle,&rxboundaryGPU[nboundary],np*sizeof(double),hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(ryParticle,&ryboundaryGPU[nboundary],np*sizeof(double),hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(rzParticle,&rzboundaryGPU[nboundary],np*sizeof(double),hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(vxParticle,&vxboundaryGPU[nboundary],np*sizeof(double),hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(vyParticle,&vyboundaryGPU[nboundary],np*sizeof(double),hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(vzParticle,&vzboundaryGPU[nboundary],np*sizeof(double),hipMemcpyDeviceToHost));
  }


  return 1;
}
