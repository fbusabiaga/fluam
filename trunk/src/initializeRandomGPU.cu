#include "hip/hip_runtime.h"
#include "header.h"
#include "headerRandomGPU.cu"
#include <iostream>
using namespace std;

#define   MT_RNG_COUNT 4096
#define          MT_MM 9
#define          MT_NN 19
#define       MT_WMASK 0xFFFFFFFFU
#define       MT_UMASK 0xFFFFFFFEU
#define       MT_LMASK 0x1U
#define      MT_SHIFT0 12
#define      MT_SHIFTB 7
#define      MT_SHIFTC 15
#define      MT_SHIFT1 18

typedef unsigned int uint32_t;
#define UINT32_C(a) ((uint32_t)a)

typedef struct {
    uint32_t aaa;
    int mm,nn,rr,ww;
    uint32_t wmask,umask,lmask;
    int shift0, shift1, shiftB, shiftC;
    uint32_t maskB, maskC;
    int i;
    uint32_t *state;
}mt_struct;

typedef struct{
    unsigned int matrix_a;
    unsigned int mask_b;
    unsigned int mask_c;
    unsigned int seed;
} mt_struct_stripped;



//const int MT_RNG_COUNT = 4096;
static mt_struct MT[MT_RNG_COUNT];
static mt_struct_stripped h_MT[MT_RNG_COUNT];


__device__ static mt_struct_stripped ds_MT[MT_RNG_COUNT];

//#define     DCMT_SEED 4172
//#define MT_RNG_PERIOD 607

__global__ void RandomGPU(double *d_Random, int NPerRng);
__device__ void BoxMuller(double& u1, double& u2);
__global__ void BoxMullerGPU(double *d_Random, int NPerRng);


extern "C" int iDivUp(int a, int b){
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

//floor(a / b)
extern "C" int iDivDown(int a, int b){
    return a / b;
}

//Align a to nearest higher multiple of b
extern "C" int iAlignUp(int a, int b){
    return ((a % b) != 0) ?  (a - a % b + b) : a;
}

//Align a to nearest lower multiple of b
extern "C" int iAlignDown(int a, int b){
    return a - a % b;
}

extern "C" void initMTRef(const char *fname){
    
    FILE *fd = fopen(fname, "rb");
    if(!fd){
        printf("initMTRef(): failed to open %s\n", fname);
        printf("TEST FAILED\n");
        exit(0);
    }

    for (int i = 0; i < MT_RNG_COUNT; i++){
        //Inline structure size for compatibility,
        //since pointer types are 8-byte on 64-bit systems (unused *state variable)
        if( !fread(MT + i, 16 /* sizeof(mt_struct) */ * sizeof(int), 1, fd) ){
            printf("initMTRef(): failed to load %s\n", fname);
            printf("TEST FAILED\n");
            exit(0);
        }
    }

    fclose(fd);
}

//Load twister configurations
void loadMTGPU(const char *fname){
    FILE *fd = fopen(fname, "rb");
    if(!fd){
        printf("initMTGPU(): failed to open %s\n", fname);
        printf("TEST FAILED\n");
        exit(0);
    }
    if( !fread(h_MT, sizeof(h_MT), 1, fd) ){
        printf("initMTGPU(): failed to load %s\n", fname);
        printf("TEST FAILED\n");
        exit(0);
    }
    fclose(fd);
}

//Initialize/seed twister for current GPU context
void seedMTGPU(unsigned int seed){
    int i;
    //Need to be thread-safe
    mt_struct_stripped *MT = (mt_struct_stripped *)malloc(MT_RNG_COUNT * sizeof(mt_struct_stripped));

    for(i = 0; i < MT_RNG_COUNT; i++){
        MT[i]      = h_MT[i];
        MT[i].seed = seed;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(ds_MT), MT, sizeof(h_MT));
    //CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(ds_MT), MT, sizeof(h_MT)) );

    free(MT);
}

__global__ void RandomGPU(
    double *d_Random,
    int NPerRng
){
    const int      tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int THREAD_N = blockDim.x * gridDim.x;

    int iState, iState1, iStateM, iOut;
    unsigned int mti, mti1, mtiM, x;
    unsigned int mt[MT_NN];

    for(int iRng = tid; iRng < MT_RNG_COUNT; iRng += THREAD_N){
        //Load bit-vector Mersenne Twister parameters
        mt_struct_stripped config = ds_MT[iRng];

        //Initialize current state
        mt[0] = config.seed;
        for(iState = 1; iState < MT_NN; iState++)
            mt[iState] = (1812433253U * (mt[iState - 1] ^ (mt[iState - 1] >> 30)) + iState) & MT_WMASK;

        iState = 0;
        mti1 = mt[0];
        for(iOut = 0; iOut < NPerRng; iOut++){
            //iState1 = (iState +     1) % MT_NN
            //iStateM = (iState + MT_MM) % MT_NN
            iState1 = iState + 1;
            iStateM = iState + MT_MM;
            if(iState1 >= MT_NN) iState1 -= MT_NN;
            if(iStateM >= MT_NN) iStateM -= MT_NN;
            mti  = mti1;
            mti1 = mt[iState1];
            mtiM = mt[iStateM];

            x    = (mti & MT_UMASK) | (mti1 & MT_LMASK);
            x    =  mtiM ^ (x >> 1) ^ ((x & 1) ? config.matrix_a : 0);
            mt[iState] = x;
            iState = iState1;

            //Tempering transformation
            x ^= (x >> MT_SHIFT0);
            x ^= (x << MT_SHIFTB) & config.mask_b;
            x ^= (x << MT_SHIFTC) & config.mask_c;
            x ^= (x >> MT_SHIFT1);

            //Convert to (0, 1] double and write to global memory
            d_Random[iRng + iOut * MT_RNG_COUNT] = ((double)x + 1.0) / 4294967296.0;
	    
        }
	//config.seed = mt[iState];
	//ds_MT[iRng] = config;
	ds_MT[iRng].seed = mt[iState];
    }
}



////////////////////////////////////////////////////////////////////////////////
// Transform each of MT_RNG_COUNT lanes of NPerRng uniformly distributed 
// random samples, produced by RandomGPU(), to normally distributed lanes
// using Cartesian form of Box-Muller transformation.
// NPerRng must be even.
////////////////////////////////////////////////////////////////////////////////
#define PI 3.14159265358979
__device__ void BoxMuller(double& u1, double& u2){
    double   r = sqrtf(-2.0 * logf(u1));
    double phi = 2 * PI * u2;
    u1 = r * __cosf(phi);
    u2 = r * __sinf(phi);
}

__global__ void BoxMullerGPU(double *d_Random, int NPerRng){
    const int      tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int THREAD_N = blockDim.x * gridDim.x;

    for(int iRng = tid; iRng < MT_RNG_COUNT; iRng += THREAD_N)
        for(int iOut = 0; iOut < NPerRng; iOut += 2)
            BoxMuller(
                d_Random[iRng + (iOut + 0) * MT_RNG_COUNT],
                d_Random[iRng + (iOut + 1) * MT_RNG_COUNT]
            );
}


void init_random_gpu(int SEED){
  const int PATH_N = 24000000;

  int N_PER_RNG = iAlignUp(iDivUp(PATH_N, MT_RNG_COUNT), 2);
  int RAND_N = MT_RNG_COUNT * N_PER_RNG;

  hipMalloc((void**)&d_rand,RAND_N*sizeof(double));

  //const char *raw_path = cutFindFilePath("MersenneTwister.raw", argv[0]);
  //const char *dat_path = cutFindFilePath("MersenneTwister.dat", argv[0]);
  initMTRef("MersenneTwister.raw");
  loadMTGPU("MersenneTwister.dat");
  seedMTGPU(SEED);

  //RandomGPU<<<32, 128>>>(d_rand, N_PER_RNG);

  //BoxMullerGPU<<<32, 128>>>(d_rand, N_PER_RNG);

  //hipFree(d_rand);
}

void free_random_gpu(){
  hipFree(d_rand);
}





bool initializeRandomGPU(){

  if(thermostat==1){
    init_random_gpu(seed);
    cout << "INITIALIZE RANDOM NUMBER GPU : DONE" << endl;
    return 1;
  }
  else{
    return 1;
  }


}
