// Filename: freeBondedForcesGPU.cu
//
// Copyright (c) 2010-2013, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


bool freeBondedForcesGPU(){

  
  cutilSafeCall(hipFree(bondsParticleParticleGPU));
  cutilSafeCall(hipFree(bondsParticleParticleOffsetGPU));
  cutilSafeCall(hipFree(bondsIndexParticleParticleGPU));
  cutilSafeCall(hipFree(r0ParticleParticleGPU));
  cutilSafeCall(hipFree(kSpringParticleParticleGPU));


  cutilSafeCall(hipFree(bondsParticleFixedPointGPU));
  cutilSafeCall(hipFree(bondsParticleFixedPointOffsetGPU));
  //cutilSafeCall(hipFree(bondsIndexParticleFixedPointGPU));
  cutilSafeCall(hipFree(r0ParticleFixedPointGPU));
  cutilSafeCall(hipFree(kSpringParticleFixedPointGPU));
  cutilSafeCall(hipFree(rxFixedPointGPU));
  cutilSafeCall(hipFree(ryFixedPointGPU));
  cutilSafeCall(hipFree(rzFixedPointGPU));

  cutilSafeCall(hipFree(bFV));
    
  return 1;
}


