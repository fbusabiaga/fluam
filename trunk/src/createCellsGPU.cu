#include "hip/hip_runtime.h"
// Filename: createCellsGPU.cu
//
// Copyright (c) 2010-2012, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.



#define GPUVARIABLES 1


bool createCellsGPU(){
  
  hipMemcpyToSymbol(HIP_SYMBOL(mxGPU),&mx,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(myGPU),&my,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(mzGPU),&mz,sizeof(int));

  hipMemcpyToSymbol(HIP_SYMBOL(mxtGPU),&mxt,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(mytGPU),&myt,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(mztGPU),&mzt,sizeof(int));

  int aux = (mxt) * (myt);
  hipMemcpyToSymbol(HIP_SYMBOL(mxmytGPU),&aux,sizeof(int));


  hipMemcpyToSymbol(HIP_SYMBOL(ncellsGPU),&ncells,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(ncellstGPU),&ncellst,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(lxGPU),&lx,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(lyGPU),&ly,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(lzGPU),&lz,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(dtGPU),&dt,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(volumeGPU),&cVolume,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(shearviscosityGPU),&shearviscosity,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(bulkviscosityGPU),&bulkviscosity,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(temperatureGPU),&temperature,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(pressurea0GPU),&pressurea0,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(pressurea1GPU),&pressurea1,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(pressurea2GPU),&pressurea2,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(thermostatGPU),&thermostat,sizeof(bool));

  hipMemcpyToSymbol(HIP_SYMBOL(densfluidGPU),&densfluid,sizeof(double));

  hipMalloc((void**)&densityGPU,ncellst*sizeof(double));
  hipMalloc((void**)&vxGPU,ncellst*sizeof(double));
  hipMalloc((void**)&vyGPU,ncellst*sizeof(double));
  hipMalloc((void**)&vzGPU,ncellst*sizeof(double));
  hipMalloc((void**)&densityPredictionGPU,ncellst*sizeof(double));
  hipMalloc((void**)&vxPredictionGPU,ncellst*sizeof(double));
  hipMalloc((void**)&vyPredictionGPU,ncellst*sizeof(double));
  hipMalloc((void**)&vzPredictionGPU,ncellst*sizeof(double));

 
  hipMalloc((void**)&dmGPU,ncellst*sizeof(double));
  hipMalloc((void**)&dpxGPU,ncellst*sizeof(double));
  hipMalloc((void**)&dpyGPU,ncellst*sizeof(double));
  hipMalloc((void**)&dpzGPU,ncellst*sizeof(double));

  hipMalloc((void**)&rxcellGPU,ncellst*sizeof(double));
  hipMalloc((void**)&rycellGPU,ncellst*sizeof(double));
  hipMalloc((void**)&rzcellGPU,ncellst*sizeof(double));

  double fact1 = sqrt((4.*temperature*shearviscosity)/(dt*cVolume));
  double fact2 = sqrt((2.*temperature*bulkviscosity)/(3.*dt*cVolume));
  double fact3 = bulkviscosity - 2. * shearviscosity/3.;
  double fact4 = sqrt((2.*temperature*shearviscosity)/(dt*cVolume));
  double fact5 = sqrt(1./(dt*cVolume));

  hipMemcpyToSymbol(HIP_SYMBOL(fact1GPU),&fact1,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(fact2GPU),&fact2,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(fact3GPU),&fact3,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(fact4GPU),&fact4,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(fact5GPU),&fact5,sizeof(double));


  fact1 = lx/double(mx);
  fact2 = ly/double(my);
  fact3 = lz/double(mz);
  hipMemcpyToSymbol(HIP_SYMBOL(dxGPU),&fact1,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(dyGPU),&fact2,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(dzGPU),&fact3,sizeof(double));

  fact1 = double(mx)/lx;
  fact2 = double(my)/ly;
  fact3 = double(mz)/lz;
  hipMemcpyToSymbol(HIP_SYMBOL(invdxGPU),&fact1,sizeof(double));  
  hipMemcpyToSymbol(HIP_SYMBOL(invdyGPU),&fact2,sizeof(double));  
  hipMemcpyToSymbol(HIP_SYMBOL(invdzGPU),&fact3,sizeof(double));  
  fact1 = 1./dt;
  hipMemcpyToSymbol(HIP_SYMBOL(invdtGPU),&fact1,sizeof(double));
  fact1 = 1./lx;
  fact2 = 1./ly;
  fact3 = 1./lz;
  hipMemcpyToSymbol(HIP_SYMBOL(invlxGPU),&fact1,sizeof(double));  
  hipMemcpyToSymbol(HIP_SYMBOL(invlyGPU),&fact2,sizeof(double));  
  hipMemcpyToSymbol(HIP_SYMBOL(invlzGPU),&fact3,sizeof(double));

 
  bool auxbool = 0;
  hipMemcpyToSymbol(HIP_SYMBOL(setparticlesGPU),&auxbool,sizeof(bool));
  hipMemcpyToSymbol(HIP_SYMBOL(setboundaryGPU),&auxbool,sizeof(bool));


  long long auxulonglong = 0;
  hipMalloc((void**)&stepGPU,sizeof(long long));
  hipMemcpy(stepGPU,&auxulonglong,sizeof(long long),hipMemcpyHostToDevice);


  hipMalloc((void**)&vecino0GPU,ncellst*sizeof(int));
  hipMalloc((void**)&vecino1GPU,ncellst*sizeof(int));
  hipMalloc((void**)&vecino2GPU,ncellst*sizeof(int));
  hipMalloc((void**)&vecino3GPU,ncellst*sizeof(int));
  hipMalloc((void**)&vecino4GPU,ncellst*sizeof(int));
  hipMalloc((void**)&vecino5GPU,ncellst*sizeof(int));
  hipMalloc((void**)&vecinopxpyGPU,ncellst*sizeof(int)); 
  hipMalloc((void**)&vecinopxmyGPU,ncellst*sizeof(int)); 
  hipMalloc((void**)&vecinopxpzGPU,ncellst*sizeof(int)); 
  hipMalloc((void**)&vecinopxmzGPU,ncellst*sizeof(int)); 
  hipMalloc((void**)&vecinomxpyGPU,ncellst*sizeof(int)); 
  hipMalloc((void**)&vecinomxmyGPU,ncellst*sizeof(int)); 
  hipMalloc((void**)&vecinomxpzGPU,ncellst*sizeof(int)); 
  hipMalloc((void**)&vecinomxmzGPU,ncellst*sizeof(int)); 
  hipMalloc((void**)&vecinopypzGPU,ncellst*sizeof(int)); 
  hipMalloc((void**)&vecinopymzGPU,ncellst*sizeof(int)); 
  hipMalloc((void**)&vecinomypzGPU,ncellst*sizeof(int)); 
  hipMalloc((void**)&vecinomymzGPU,ncellst*sizeof(int)); 
  hipMalloc((void**)&vecinopxpypzGPU,ncellst*sizeof(int)); 
  hipMalloc((void**)&vecinopxpymzGPU,ncellst*sizeof(int)); 
  hipMalloc((void**)&vecinopxmypzGPU,ncellst*sizeof(int)); 
  hipMalloc((void**)&vecinopxmymzGPU,ncellst*sizeof(int)); 
  hipMalloc((void**)&vecinomxpypzGPU,ncellst*sizeof(int)); 
  hipMalloc((void**)&vecinomxpymzGPU,ncellst*sizeof(int));
  hipMalloc((void**)&vecinomxmypzGPU,ncellst*sizeof(int)); 
  hipMalloc((void**)&vecinomxmymzGPU,ncellst*sizeof(int)); 



  if(particlesWall){
    hipMalloc((void**)&ghostIndexGPU,ncells*sizeof(int));
    hipMalloc((void**)&realIndexGPU,ncellst*sizeof(int));
    hipMalloc((void**)&ghostToPIGPU,(ncellst-ncells)*sizeof(int));
    hipMalloc((void**)&ghostToGhostGPU,(ncellst-ncells)*sizeof(int));

    fact1 = ly + 2*ly/double(my);
    fact2 = 1. / (ly + 2*ly/double(my));
    hipMemcpyToSymbol(HIP_SYMBOL(lyGPU),&fact1,sizeof(double)); 
    hipMemcpyToSymbol(HIP_SYMBOL(invlyGPU),&fact2,sizeof(double));   
  }
  

  
  cout << "CREATE CELLS GPU :              DONE" << endl;

  return 1;
}
