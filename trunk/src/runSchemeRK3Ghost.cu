// Filename: runSchemeRK3Ghost.cu
//
// Copyright (c) 2010-2013, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


bool runSchemeRK3Ghost(){
  int threadsPerBlock = 128;
  if((ncells/threadsPerBlock) < 60) threadsPerBlock = 64;
  if((ncells/threadsPerBlock) < 60) threadsPerBlock = 32;
  int numBlocks = (ncells-1)/threadsPerBlock + 1;

  int nGhost = ncellst - ncells;
  int threadsPerBlockGhost = 128;
  if((nGhost/threadsPerBlockGhost) < 60) threadsPerBlockGhost = 64;
  if((nGhost/threadsPerBlockGhost) < 60) threadsPerBlockGhost = 32;
  int numBlocksGhost = (nGhost-1)/threadsPerBlockGhost + 1;



  //Initialize textures cells
  if(!texturesCellsGhost()) return 0;
  
  //Inilialize ghost index
  if(!initGhostIndexGPU()) return 0;
  
  step = -numstepsRelaxation;


  while(step<numsteps){
    //Provide data to ghost cells
    kernelFeedGhostCellsRK3<<<numBlocksGhost,threadsPerBlockGhost>>>(ghostToPIGPU,
								     ghostToGhostGPU,
								     densityGPU,
								     densityPredictionGPU,
								     vxGPU,
								     vyGPU,
								     vzGPU,
								     vxPredictionGPU,
								     vyPredictionGPU,
								     vzPredictionGPU);
    //First substep RK3
    kernelDpRK3Ghost_1<<<numBlocks,threadsPerBlock>>>(densityGPU,
						      densityGPU,
						      vxGPU,
						      vyGPU,
						      vzGPU,
						      dmGPU,
						      dpxGPU,
						      dpyGPU,
						      dpzGPU,
						      ghostIndexGPU,
						      realIndexGPU,
						      0,1);

    kernelDpRK3Ghost_2<<<numBlocks,threadsPerBlock>>>(densityPredictionGPU,
						      vxPredictionGPU,
						      vyPredictionGPU,
						      vzPredictionGPU,
						      dmGPU,
						      dpxGPU,
						      dpyGPU,
						      dpzGPU,
						      ghostIndexGPU,
						      realIndexGPU);

    cutilSafeCall( hipBindTexture(0,texVxGPU,vxPredictionGPU,ncellst*sizeof(double)));
    cutilSafeCall( hipBindTexture(0,texVyGPU,vyPredictionGPU,ncellst*sizeof(double)));
    cutilSafeCall( hipBindTexture(0,texVzGPU,vzPredictionGPU,ncellst*sizeof(double)));

    //Provide data to ghost cells
    kernelFeedGhostCellsRK3<<<numBlocksGhost,threadsPerBlockGhost>>>(ghostToPIGPU,
								     ghostToGhostGPU,
								     densityGPU,
								     densityPredictionGPU,
								     vxGPU,
								     vyGPU,
								     vzGPU,
								     vxPredictionGPU,
								     vyPredictionGPU,
								     vzPredictionGPU);

    //Second substep RK3
    kernelDpRK3Ghost_1<<<numBlocks,threadsPerBlock>>>(densityPredictionGPU,
						      densityGPU,
						      vxGPU,
						      vyGPU,
						      vzGPU,
						      dmGPU,
						      dpxGPU,
						      dpyGPU,
						      dpzGPU,
						      ghostIndexGPU,
						      realIndexGPU,
						      0.75,0.25);

    kernelDpRK3Ghost_2<<<numBlocks,threadsPerBlock>>>(densityPredictionGPU,
						      vxPredictionGPU,
						      vyPredictionGPU,
						      vzPredictionGPU,
						      dmGPU,
						      dpxGPU,
						      dpyGPU,
						      dpzGPU,
						      ghostIndexGPU,
						      realIndexGPU);

    //Provide data to ghost cells
    kernelFeedGhostCellsRK3<<<numBlocksGhost,threadsPerBlockGhost>>>(ghostToPIGPU,
								     ghostToGhostGPU,
								     densityGPU,
								     densityPredictionGPU,
								     vxGPU,
								     vyGPU,
								     vzGPU,
								     vxPredictionGPU,
								     vyPredictionGPU,
								     vzPredictionGPU);

    //Third substep RK3
    kernelDpRK3Ghost_1<<<numBlocks,threadsPerBlock>>>(densityPredictionGPU,
						      densityGPU,
						      vxGPU,
						      vyGPU,
						      vzGPU,
						      dmGPU,
						      dpxGPU,
						      dpyGPU,
						      dpzGPU,
						      ghostIndexGPU,
						      realIndexGPU,
						      1./3.,2./3.);

    kernelDpRK3Ghost_2<<<numBlocks,threadsPerBlock>>>(densityGPU,
						      vxGPU,
						      vyGPU,
						      vzGPU,
						      dmGPU,
						      dpxGPU,
						      dpyGPU,
						      dpzGPU,
						      ghostIndexGPU,
						      realIndexGPU);

    cutilSafeCall( hipBindTexture(0,texVxGPU,vxGPU,ncellst*sizeof(double)));
    cutilSafeCall( hipBindTexture(0,texVyGPU,vyGPU,ncellst*sizeof(double)));
    cutilSafeCall( hipBindTexture(0,texVzGPU,vzGPU,ncellst*sizeof(double)));
    
    
    step++;
    //cout << step << endl;

    if(!(step%samplefreq)&&(step>0)){
      kernelFeedGhostCellsRK3<<<numBlocksGhost,threadsPerBlockGhost>>>(ghostToPIGPU,
								       ghostToGhostGPU,
								       densityGPU,
								       densityPredictionGPU,
								       vxGPU,
								       vyGPU,
								       vzGPU,
								       vxPredictionGPU,
								       vyPredictionGPU,
								       vzPredictionGPU);

      cout << "RK3 " << step << endl;
      if(!gpuToHostRK3Ghost()) return 0;
      if(!saveFunctionsSchemeRK3Ghost(1)) return 0;
    }
  }
  
    






  return 1;
}
