#include "hip/hip_runtime.h"
// Filename: createCellsIncompressibleBinaryMixtureGPU.cu
//
// Copyright (c) 2010-2012, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


#define GPUVARIABLES 1


bool createCellsIncompressibleBinaryMixtureGPU(){
  //Number of cells to constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(mxGPU),&mx,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(myGPU),&my,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(mzGPU),&mz,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(ncellsGPU),&ncells,sizeof(int));

  hipMemcpyToSymbol(HIP_SYMBOL(mxtGPU),&mxt,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(mytGPU),&myt,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(mztGPU),&mzt,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(ncellstGPU),&ncellst,sizeof(int));

  //Simulation box size to constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(lxGPU),&lx,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(lyGPU),&ly,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(lzGPU),&lz,sizeof(double));

  //Time step to constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(dtGPU),&dt,sizeof(double));

  //Volume cell to constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(volumeGPU),&cVolume,sizeof(double));

  //Viscosity and temperature to constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(shearviscosityGPU),&shearviscosity,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(temperatureGPU),&temperature,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(thermostatGPU),&thermostat,sizeof(bool));

  //Mass diffusion coefficient to constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(diffusionGPU),&diffusion,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(massSpecies0GPU),&massSpecies0,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(massSpecies1GPU),&massSpecies1,sizeof(double));

  //Fluid density to constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(densfluidGPU),&densfluid,sizeof(double));

  double fact1, fact4, fact5;
  //FACT1 DIFFERENT FOR INCOMPRESSIBLE
  fact1 = sqrt((4.*temperature*shearviscosity*dt)/(cVolume*densfluid*densfluid));
  //FACT4 DIFFERENT FOR INCOMPRESSIBLE
  fact4 = sqrt((2.*temperature*shearviscosity*dt)/(cVolume*densfluid*densfluid));
  fact5 = sqrt(1./(dt*cVolume));
  hipMemcpyToSymbol(HIP_SYMBOL(gradTemperatureGPU),&gradTemperature,sizeof(double));
  
  //Prefactor for stochastic force to constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(fact1GPU),&fact1,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(fact4GPU),&fact4,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(fact5GPU),&fact5,sizeof(double));

  //Cell size to constant memory
  fact1 = lx/double(mx);
  double fact2 = ly/double(my);
  double fact3 = lz/double(mz);
  hipMemcpyToSymbol(HIP_SYMBOL(dxGPU),&fact1,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(dyGPU),&fact2,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(dzGPU),&fact3,sizeof(double));

  //Inverse cell size to cosntant memory
  fact1 = double(mx)/lx;
  fact2 = double(my)/ly;
  fact3 = double(mz)/lz;
  hipMemcpyToSymbol(HIP_SYMBOL(invdxGPU),&fact1,sizeof(double));  
  hipMemcpyToSymbol(HIP_SYMBOL(invdyGPU),&fact2,sizeof(double));  
  hipMemcpyToSymbol(HIP_SYMBOL(invdzGPU),&fact3,sizeof(double));  

  //Inverse time step to constant memory
  fact1 = 1./dt;
  hipMemcpyToSymbol(HIP_SYMBOL(invdtGPU),&fact1,sizeof(double));

  //Inverse box size to constant memory
  fact1 = 1./lx;
  fact2 = 1./ly;
  fact3 = 1./lz;
  hipMemcpyToSymbol(HIP_SYMBOL(invlxGPU),&fact1,sizeof(double));  
  hipMemcpyToSymbol(HIP_SYMBOL(invlyGPU),&fact2,sizeof(double));  
  hipMemcpyToSymbol(HIP_SYMBOL(invlzGPU),&fact3,sizeof(double));

  //Some options to constant memory
  bool auxbool = 0;
  hipMemcpyToSymbol(HIP_SYMBOL(setparticlesGPU),&auxbool,sizeof(bool));
  hipMemcpyToSymbol(HIP_SYMBOL(setboundaryGPU),&auxbool,sizeof(bool));
  








  //Step to global memory
  long long auxulonglong = 0;
  hipMalloc((void**)&stepGPU,sizeof(long long));
  hipMemcpy(stepGPU,&auxulonglong,sizeof(long long),hipMemcpyHostToDevice);

  //Fluid velocity and velocity prediction to
  //global memory
  hipMalloc((void**)&vxGPU,ncells*sizeof(double));
  hipMalloc((void**)&vyGPU,ncells*sizeof(double));
  hipMalloc((void**)&vzGPU,ncells*sizeof(double));
  hipMalloc((void**)&vxPredictionGPU,ncells*sizeof(double));
  hipMalloc((void**)&vyPredictionGPU,ncells*sizeof(double));
  hipMalloc((void**)&vzPredictionGPU,ncells*sizeof(double));

  //Concentration to global memory
  hipMalloc((void**)&cGPU,ncells*sizeof(double));
  hipMalloc((void**)&cPredictionGPU,ncells*sizeof(double));

  //Centers cells to global memory
  hipMalloc((void**)&rxcellGPU,ncells*sizeof(double));
  hipMalloc((void**)&rycellGPU,ncells*sizeof(double));
  hipMalloc((void**)&rzcellGPU,ncells*sizeof(double));

  //List of neighbors cells to global memory
  hipMalloc((void**)&vecino0GPU,ncells*sizeof(int));
  hipMalloc((void**)&vecino1GPU,ncells*sizeof(int));
  hipMalloc((void**)&vecino2GPU,ncells*sizeof(int));
  hipMalloc((void**)&vecino3GPU,ncells*sizeof(int));
  hipMalloc((void**)&vecino4GPU,ncells*sizeof(int));
  hipMalloc((void**)&vecino5GPU,ncells*sizeof(int));
  hipMalloc((void**)&vecinopxpyGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinopxmyGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinopxpzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinopxmzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinomxpyGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinomxmyGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinomxpzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinomxmzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinopypzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinopymzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinomypzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinomymzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinopxpypzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinopxpymzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinopxmypzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinopxmymzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinomxpypzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinomxpymzGPU,ncells*sizeof(int));
  hipMalloc((void**)&vecinomxmypzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinomxmymzGPU,ncells*sizeof(int)); 

  //Factors for the update in fourier space to global memory
  hipMalloc((void**)&gradKx,     mx*sizeof(hipfftDoubleComplex));
  hipMalloc((void**)&gradKy,     my*sizeof(hipfftDoubleComplex));
  hipMalloc((void**)&gradKz,     mz*sizeof(hipfftDoubleComplex));
  hipMalloc((void**)&expKx,      mx*sizeof(hipfftDoubleComplex));
  hipMalloc((void**)&expKy,      my*sizeof(hipfftDoubleComplex));  
  hipMalloc((void**)&expKz,      mz*sizeof(hipfftDoubleComplex));

  hipMalloc((void**)&pF,sizeof(prefactorsFourier));

  //Complex velocity field to global memory
  hipMalloc((void**)&vxZ,ncells*sizeof(hipfftDoubleComplex));
  hipMalloc((void**)&vyZ,ncells*sizeof(hipfftDoubleComplex));
  hipMalloc((void**)&vzZ,ncells*sizeof(hipfftDoubleComplex)); 

  //Complex concentration to global memory
  hipMalloc((void**)&cZ,ncells*sizeof(hipfftDoubleComplex));


  cout << "CREATE CELLS GPU :              DONE" << endl;

  return 1;
}
