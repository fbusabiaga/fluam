#include "hip/hip_runtime.h"
// Filename: createCellsIncompressibleBinaryMixtureGPU.cu
//
// Copyright (c) 2010-2013, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


#define GPUVARIABLES 1


bool createCellsIncompressibleBinaryMixtureGPU(){
  //Number of cells to constant memory
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mxGPU),&mx,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(myGPU),&my,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mzGPU),&mz,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ncellsGPU),&ncells,sizeof(int)));

  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mxtGPU),&mxt,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mytGPU),&myt,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mztGPU),&mzt,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ncellstGPU),&ncellst,sizeof(int)));

  //Simulation box size to constant memory
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(lxGPU),&lx,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(lyGPU),&ly,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(lzGPU),&lz,sizeof(double)));

  //Time step to constant memory
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dtGPU),&dt,sizeof(double)));

  //Volume cell to constant memory
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(volumeGPU),&cVolume,sizeof(double)));

  //Viscosity and temperature to constant memory
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(shearviscosityGPU),&shearviscosity,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(temperatureGPU),&temperature,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(thermostatGPU),&thermostat,sizeof(bool)));

  //Mass diffusion coefficient to constant memory
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(diffusionGPU),&diffusion,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(massSpecies0GPU),&massSpecies0,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(massSpecies1GPU),&massSpecies1,sizeof(double)));

  //Fluid density to constant memory
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(densfluidGPU),&densfluid,sizeof(double)));

  double fact1, fact4, fact5;
  //FACT1 DIFFERENT FOR INCOMPRESSIBLE
  fact1 = sqrt((4.*temperature*shearviscosity*dt)/(cVolume*densfluid*densfluid));
  //FACT4 DIFFERENT FOR INCOMPRESSIBLE
  fact4 = sqrt((2.*temperature*shearviscosity*dt)/(cVolume*densfluid*densfluid));
  fact5 = sqrt(1./(dt*cVolume));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gradTemperatureGPU),&gradTemperature,sizeof(double)));
  
  //Prefactor for stochastic force to constant memory
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(fact1GPU),&fact1,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(fact4GPU),&fact4,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(fact5GPU),&fact5,sizeof(double)));

  //Cell size to constant memory
  fact1 = lx/double(mx);
  double fact2 = ly/double(my);
  double fact3 = lz/double(mz);
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dxGPU),&fact1,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dyGPU),&fact2,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dzGPU),&fact3,sizeof(double)));

  //Inverse cell size to cosntant memory
  fact1 = double(mx)/lx;
  fact2 = double(my)/ly;
  fact3 = double(mz)/lz;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invdxGPU),&fact1,sizeof(double)));  
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invdyGPU),&fact2,sizeof(double)));  
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invdzGPU),&fact3,sizeof(double)));  

  //Inverse time step to constant memory
  fact1 = 1./dt;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invdtGPU),&fact1,sizeof(double)));

  //Inverse box size to constant memory
  fact1 = 1./lx;
  fact2 = 1./ly;
  fact3 = 1./lz;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invlxGPU),&fact1,sizeof(double)));  
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invlyGPU),&fact2,sizeof(double)));  
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invlzGPU),&fact3,sizeof(double)));

  //Some options to constant memory
  bool auxbool = 0;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(setparticlesGPU),&auxbool,sizeof(bool)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(setboundaryGPU),&auxbool,sizeof(bool)));
  








  //Step to global memory
  long long auxulonglong = 0;
  cutilSafeCall(hipMalloc((void**)&stepGPU,sizeof(long long)));
  cutilSafeCall(hipMemcpy(stepGPU,&auxulonglong,sizeof(long long),hipMemcpyHostToDevice));

  //Fluid velocity and velocity prediction to
  //global memory
  cutilSafeCall(hipMalloc((void**)&vxGPU,ncells*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vyGPU,ncells*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vzGPU,ncells*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vxPredictionGPU,ncells*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vyPredictionGPU,ncells*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vzPredictionGPU,ncells*sizeof(double)));

  //Concentration to global memory
  cutilSafeCall(hipMalloc((void**)&cGPU,ncells*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&cPredictionGPU,ncells*sizeof(double)));

  //Centers cells to global memory
  cutilSafeCall(hipMalloc((void**)&rxcellGPU,ncells*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&rycellGPU,ncells*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&rzcellGPU,ncells*sizeof(double)));

  //List of neighbors cells to global memory
  cutilSafeCall(hipMalloc((void**)&vecino0GPU,ncells*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&vecino1GPU,ncells*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&vecino2GPU,ncells*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&vecino3GPU,ncells*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&vecino4GPU,ncells*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&vecino5GPU,ncells*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&vecinopxpyGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopxmyGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopxpzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopxmzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomxpyGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomxmyGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomxpzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomxmzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopypzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopymzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomypzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomymzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopxpypzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopxpymzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopxmypzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopxmymzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomxpypzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomxpymzGPU,ncells*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&vecinomxmypzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomxmymzGPU,ncells*sizeof(int))); 

  //Factors for the update in fourier space to global memory
  cutilSafeCall(hipMalloc((void**)&gradKx,     mx*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&gradKy,     my*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&gradKz,     mz*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&expKx,      mx*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&expKy,      my*sizeof(hipfftDoubleComplex)));  
  cutilSafeCall(hipMalloc((void**)&expKz,      mz*sizeof(hipfftDoubleComplex)));

  cutilSafeCall(hipMalloc((void**)&pF,sizeof(prefactorsFourier)));

  //Complex velocity field to global memory
  cutilSafeCall(hipMalloc((void**)&vxZ,ncells*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&vyZ,ncells*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&vzZ,ncells*sizeof(hipfftDoubleComplex))); 

  //Complex concentration to global memory
  cutilSafeCall(hipMalloc((void**)&cZ,ncells*sizeof(hipfftDoubleComplex)));


  cout << "CREATE CELLS GPU :              DONE" << endl;

  return 1;
}
