// Filename: freeCellsBinaryMixtureGPU.cu
//
// Copyright (c) 2010-2013, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


bool freeCellsBinaryMixtureGPU(){
  cutilSafeCall(hipFree(densityGPU));
  cutilSafeCall(hipUnbindTexture(texVxGPU));
  cutilSafeCall(hipUnbindTexture(texVyGPU));
  cutilSafeCall(hipUnbindTexture(texVzGPU));    
  cutilSafeCall(hipFree(vxGPU));
  cutilSafeCall(hipFree(vyGPU));
  cutilSafeCall(hipFree(vzGPU));
  cutilSafeCall(hipFree(densityPredictionGPU));
  cutilSafeCall(hipFree(vxPredictionGPU));
  cutilSafeCall(hipFree(vyPredictionGPU));
  cutilSafeCall(hipFree(vzPredictionGPU));

  cutilSafeCall(hipFree(cGPU));
  cutilSafeCall(hipFree(cPredictionGPU));
  cutilSafeCall(hipFree(dcGPU));

  cutilSafeCall(hipFree(dmGPU));
  cutilSafeCall(hipFree(dpxGPU));
  cutilSafeCall(hipFree(dpyGPU));
  cutilSafeCall(hipFree(dpzGPU));

  cutilSafeCall(hipFree(rxcellGPU));
  cutilSafeCall(hipFree(rycellGPU));
  cutilSafeCall(hipFree(rzcellGPU));

  cutilSafeCall(hipFree(ghostIndexGPU));
  cutilSafeCall(hipFree(realIndexGPU));
  cutilSafeCall(hipFree(ghostToPIGPU));
  cutilSafeCall(hipFree(ghostToGhostGPU));

  cutilSafeCall(hipFree(stepGPU));

  cout << "FREE MEMORY GPU :               DONE" << endl; 


  return 1;
}

