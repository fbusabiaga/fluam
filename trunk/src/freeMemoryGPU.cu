// Filename: freeMemoryGPU.cu
//
// Copyright (c) 2010-2012, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


bool freeMemoryGPU(){
  if(thermostat == 1){
    hipFree(d_rand);
  }
  //hipFree(massGPU);
  hipFree(densityGPU);
  hipUnbindTexture(texVxGPU);
  hipUnbindTexture(texVyGPU);
  hipUnbindTexture(texVzGPU);    
  hipFree(vxGPU);
  hipFree(vyGPU);
  hipFree(vzGPU);
  hipFree(densityPredictionGPU);
  hipFree(vxPredictionGPU);
  hipFree(vyPredictionGPU);
  hipFree(vzPredictionGPU);

  //hipFree(fxGPU);
  //hipFree(fyGPU);
  //hipFree(fzGPU);

  hipFree(dmGPU);
  hipFree(dpxGPU);
  hipFree(dpyGPU);
  hipFree(dpzGPU);

  hipFree(rxcellGPU);
  hipFree(rycellGPU);
  hipFree(rzcellGPU);

  if((setboundary==1) || (setparticles==1)){
    hipUnbindTexture(texrxboundaryGPU);    
    hipUnbindTexture(texryboundaryGPU);    
    hipUnbindTexture(texrzboundaryGPU);
    hipUnbindTexture(texfxboundaryGPU);
    hipUnbindTexture(texfyboundaryGPU);
    hipUnbindTexture(texfzboundaryGPU);
    hipFree(rxboundaryGPU);
    hipFree(ryboundaryGPU);
    hipFree(rzboundaryGPU);
    hipFree(vxboundaryGPU);
    hipFree(vyboundaryGPU);
    hipFree(vzboundaryGPU);
    hipFree(fxboundaryGPU);
    hipFree(fyboundaryGPU);
    hipFree(fzboundaryGPU);
    
    //freeDelta();

    hipUnbindTexture(texCountParticlesInCellX);
    hipUnbindTexture(texCountParticlesInCellY);
    hipUnbindTexture(texCountParticlesInCellZ);
    hipUnbindTexture(texPartInCellX);
    hipUnbindTexture(texPartInCellY);
    hipUnbindTexture(texPartInCellZ);
    hipUnbindTexture(texCountParticlesInCellNonBonded);
    hipUnbindTexture(texPartInCellNonBonded);
  }

  if(setparticles == 1){
    hipFree(countPartInCellNonBonded);
    hipFree(partInCellNonBonded);
    hipFree(neighbor0GPU);
    hipFree(neighbor1GPU);
    hipFree(neighbor2GPU);
    hipFree(neighbor3GPU);
    hipFree(neighbor4GPU);
    hipFree(neighbor5GPU);
    hipFree(neighborpxpyGPU);
    hipFree(neighborpxmyGPU);
    hipFree(neighborpxpzGPU);
    hipFree(neighborpxmzGPU);
    hipFree(neighbormxpyGPU);
    hipFree(neighbormxmyGPU);
    hipFree(neighbormxpzGPU);
    hipFree(neighbormxmzGPU);
    hipFree(neighborpypzGPU);
    hipFree(neighborpymzGPU);
    hipFree(neighbormypzGPU);
    hipFree(neighbormymzGPU);
    hipFree(neighborpxpypzGPU);
    hipFree(neighborpxpymzGPU);
    hipFree(neighborpxmypzGPU);
    hipFree(neighborpxmymzGPU);
    hipFree(neighbormxpypzGPU);
    hipFree(neighbormxpymzGPU);
    hipFree(neighbormxmypzGPU);
    hipFree(neighbormxmymzGPU);
    hipUnbindTexture(texforceNonBonded1);
    hipUnbindTexture(texneighbor0GPU);
    hipUnbindTexture(texneighbor1GPU);
    hipUnbindTexture(texneighbor2GPU);
    hipUnbindTexture(texneighbor3GPU);
    hipUnbindTexture(texneighbor4GPU);
    hipUnbindTexture(texneighbor5GPU);
    hipUnbindTexture(texneighborpxpyGPU);
    hipUnbindTexture(texneighborpxmyGPU);
    hipUnbindTexture(texneighborpxpzGPU);
    hipUnbindTexture(texneighborpxmzGPU);
    hipUnbindTexture(texneighbormxpyGPU);
    hipUnbindTexture(texneighbormxmyGPU);
    hipUnbindTexture(texneighbormxpzGPU);
    hipUnbindTexture(texneighbormxmzGPU);
    hipUnbindTexture(texneighborpypzGPU);
    hipUnbindTexture(texneighborpymzGPU);
    hipUnbindTexture(texneighbormypzGPU);
    hipUnbindTexture(texneighbormymzGPU);
    hipUnbindTexture(texneighborpxpypzGPU);
    hipUnbindTexture(texneighborpxpymzGPU);
    hipUnbindTexture(texneighborpxmypzGPU);
    hipUnbindTexture(texneighborpxmymzGPU);
    hipUnbindTexture(texneighbormxpypzGPU);
    hipUnbindTexture(texneighbormxpymzGPU);
    hipUnbindTexture(texneighbormxmypzGPU);
    hipUnbindTexture(texneighbormxmymzGPU);
    hipFreeArray(forceNonBonded1);
    hipFree(saveForceX);
    hipFree(saveForceY);
    hipFree(saveForceZ);
  }
  
  hipUnbindTexture(texvecino0GPU);
  hipUnbindTexture(texvecino1GPU);
  hipUnbindTexture(texvecino2GPU);
  hipUnbindTexture(texvecino3GPU);
  hipUnbindTexture(texvecino4GPU);
  hipUnbindTexture(texvecino5GPU);
  hipUnbindTexture(texvecinopxpyGPU);
  hipUnbindTexture(texvecinopxmyGPU);
  hipUnbindTexture(texvecinopxpzGPU);
  hipUnbindTexture(texvecinopxmzGPU);
  hipUnbindTexture(texvecinomxpyGPU);
  hipUnbindTexture(texvecinomxmyGPU);
  hipUnbindTexture(texvecinomxpzGPU);
  hipUnbindTexture(texvecinomxmzGPU);
  hipUnbindTexture(texvecinopypzGPU);
  hipUnbindTexture(texvecinopymzGPU);
  hipUnbindTexture(texvecinomypzGPU);
  hipUnbindTexture(texvecinomymzGPU);
  hipUnbindTexture(texvecinopxpypzGPU);
  hipUnbindTexture(texvecinopxpymzGPU);
  hipUnbindTexture(texvecinopxmypzGPU);
  hipUnbindTexture(texvecinopxmymzGPU);
  hipUnbindTexture(texvecinomxpypzGPU);
  hipUnbindTexture(texvecinomxpymzGPU);
  hipUnbindTexture(texvecinomxmypzGPU);
  hipUnbindTexture(texvecinomxmymzGPU);

  hipFree(vecino0GPU);
  hipFree(vecino1GPU);
  hipFree(vecino2GPU);
  hipFree(vecino3GPU);
  hipFree(vecino4GPU);
  hipFree(vecino5GPU);

  hipFree(vecinopxpyGPU);
  hipFree(vecinopxmyGPU);
  hipFree(vecinopxpzGPU);
  hipFree(vecinopxmzGPU);
  hipFree(vecinomxpyGPU);
  hipFree(vecinomxmyGPU);
  hipFree(vecinomxpzGPU);
  hipFree(vecinomxmzGPU);  
  hipFree(vecinopypzGPU);
  hipFree(vecinopymzGPU);
  hipFree(vecinomypzGPU);
  hipFree(vecinomymzGPU);
  hipFree(vecinopxpypzGPU);
  hipFree(vecinopxpymzGPU);
  hipFree(vecinopxmypzGPU);
  hipFree(vecinopxmymzGPU);
  hipFree(vecinomxpypzGPU);
  hipFree(vecinomxpymzGPU);
  hipFree(vecinomxmypzGPU);
  hipFree(vecinomxmymzGPU);

  hipFree(countparticlesincellX);
  hipFree(countparticlesincellY);
  hipFree(countparticlesincellZ);
  hipFree(partincellX);
  hipFree(partincellY);
  hipFree(partincellZ);
  hipFree(errorKernel);
  hipFree(stepGPU);

  if(setCheckVelocity==1){
    hipFree(rxCheckGPU);
    hipFree(ryCheckGPU);
    hipFree(rzCheckGPU);
    hipFree(vxCheckGPU);
    hipFree(vyCheckGPU);
    hipFree(vzCheckGPU);
  }


  if(!freeOtherFluidVariablesGPU()) return 0;

  cout << "FREE MEMORY GPU :               DONE" << endl; 


  return 1;
}

