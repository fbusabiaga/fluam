// Filename: freeMemoryGPU.cu
//
// Copyright (c) 2010-2013, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


bool freeMemoryGPU(){
  if(thermostat == 1){
    hipFree(d_rand);
  }
  //hipFree(massGPU);
  hipFree(densityGPU);
  cutilSafeCall(hipUnbindTexture(texVxGPU));
  cutilSafeCall(hipUnbindTexture(texVyGPU));
  cutilSafeCall(hipUnbindTexture(texVzGPU));    
  hipFree(vxGPU);
  hipFree(vyGPU);
  hipFree(vzGPU);
  hipFree(densityPredictionGPU);
  hipFree(vxPredictionGPU);
  hipFree(vyPredictionGPU);
  hipFree(vzPredictionGPU);

  //hipFree(fxGPU);
  //hipFree(fyGPU);
  //hipFree(fzGPU);

  hipFree(dmGPU);
  hipFree(dpxGPU);
  hipFree(dpyGPU);
  hipFree(dpzGPU);

  hipFree(rxcellGPU);
  hipFree(rycellGPU);
  hipFree(rzcellGPU);

  if((setboundary==1) || (setparticles==1)){
    cutilSafeCall(hipUnbindTexture(texrxboundaryGPU));    
    cutilSafeCall(hipUnbindTexture(texryboundaryGPU));    
    cutilSafeCall(hipUnbindTexture(texrzboundaryGPU));
    cutilSafeCall(hipUnbindTexture(texfxboundaryGPU));
    cutilSafeCall(hipUnbindTexture(texfyboundaryGPU));
    cutilSafeCall(hipUnbindTexture(texfzboundaryGPU));
    hipFree(rxboundaryGPU);
    hipFree(ryboundaryGPU);
    hipFree(rzboundaryGPU);
    cutilSafeCall(hipFree(vxboundaryGPU));
    cutilSafeCall(hipFree(vyboundaryGPU));
    cutilSafeCall(hipFree(vzboundaryGPU));
    cutilSafeCall(hipFree(fxboundaryGPU));
    cutilSafeCall(hipFree(fyboundaryGPU));
    cutilSafeCall(hipFree(fzboundaryGPU));
    
    //freeDelta();

    cutilSafeCall(hipUnbindTexture(texCountParticlesInCellX));
    cutilSafeCall(hipUnbindTexture(texCountParticlesInCellY));
    cutilSafeCall(hipUnbindTexture(texCountParticlesInCellZ));
    cutilSafeCall(hipUnbindTexture(texPartInCellX));
    cutilSafeCall(hipUnbindTexture(texPartInCellY));
    cutilSafeCall(hipUnbindTexture(texPartInCellZ));
    cutilSafeCall(hipUnbindTexture(texCountParticlesInCellNonBonded));
    cutilSafeCall(hipUnbindTexture(texPartInCellNonBonded));
  }

  if(setparticles == 1){
    cutilSafeCall(hipFree(countPartInCellNonBonded));
    cutilSafeCall(hipFree(partInCellNonBonded));
    cutilSafeCall(hipFree(neighbor0GPU));
    cutilSafeCall(hipFree(neighbor1GPU));
    cutilSafeCall(hipFree(neighbor2GPU));
    cutilSafeCall(hipFree(neighbor3GPU));
    cutilSafeCall(hipFree(neighbor4GPU));
    cutilSafeCall(hipFree(neighbor5GPU));
    cutilSafeCall(hipFree(neighborpxpyGPU));
    cutilSafeCall(hipFree(neighborpxmyGPU));
    cutilSafeCall(hipFree(neighborpxpzGPU));
    cutilSafeCall(hipFree(neighborpxmzGPU));
    cutilSafeCall(hipFree(neighbormxpyGPU));
    cutilSafeCall(hipFree(neighbormxmyGPU));
    cutilSafeCall(hipFree(neighbormxpzGPU));
    cutilSafeCall(hipFree(neighbormxmzGPU));
    cutilSafeCall(hipFree(neighborpypzGPU));
    cutilSafeCall(hipFree(neighborpymzGPU));
    cutilSafeCall(hipFree(neighbormypzGPU));
    cutilSafeCall(hipFree(neighbormymzGPU));
    cutilSafeCall(hipFree(neighborpxpypzGPU));
    cutilSafeCall(hipFree(neighborpxpymzGPU));
    cutilSafeCall(hipFree(neighborpxmypzGPU));
    cutilSafeCall(hipFree(neighborpxmymzGPU));
    cutilSafeCall(hipFree(neighbormxpypzGPU));
    cutilSafeCall(hipFree(neighbormxpymzGPU));
    cutilSafeCall(hipFree(neighbormxmypzGPU));
    cutilSafeCall(hipFree(neighbormxmymzGPU));
    cutilSafeCall(hipUnbindTexture(texforceNonBonded1));
    cutilSafeCall(hipUnbindTexture(texneighbor0GPU));
    cutilSafeCall(hipUnbindTexture(texneighbor1GPU));
    cutilSafeCall(hipUnbindTexture(texneighbor2GPU));
    cutilSafeCall(hipUnbindTexture(texneighbor3GPU));
    cutilSafeCall(hipUnbindTexture(texneighbor4GPU));
    cutilSafeCall(hipUnbindTexture(texneighbor5GPU));
    cutilSafeCall(hipUnbindTexture(texneighborpxpyGPU));
    cutilSafeCall(hipUnbindTexture(texneighborpxmyGPU));
    cutilSafeCall(hipUnbindTexture(texneighborpxpzGPU));
    cutilSafeCall(hipUnbindTexture(texneighborpxmzGPU));
    cutilSafeCall(hipUnbindTexture(texneighbormxpyGPU));
    cutilSafeCall(hipUnbindTexture(texneighbormxmyGPU));
    cutilSafeCall(hipUnbindTexture(texneighbormxpzGPU));
    cutilSafeCall(hipUnbindTexture(texneighbormxmzGPU));
    cutilSafeCall(hipUnbindTexture(texneighborpypzGPU));
    cutilSafeCall(hipUnbindTexture(texneighborpymzGPU));
    cutilSafeCall(hipUnbindTexture(texneighbormypzGPU));
    cutilSafeCall(hipUnbindTexture(texneighbormymzGPU));
    cutilSafeCall(hipUnbindTexture(texneighborpxpypzGPU));
    cutilSafeCall(hipUnbindTexture(texneighborpxpymzGPU));
    cutilSafeCall(hipUnbindTexture(texneighborpxmypzGPU));
    cutilSafeCall(hipUnbindTexture(texneighborpxmymzGPU));
    cutilSafeCall(hipUnbindTexture(texneighbormxpypzGPU));
    cutilSafeCall(hipUnbindTexture(texneighbormxpymzGPU));
    cutilSafeCall(hipUnbindTexture(texneighbormxmypzGPU));
    cutilSafeCall(hipUnbindTexture(texneighbormxmymzGPU));
    cutilSafeCall(hipFreeArray(forceNonBonded1));
    cutilSafeCall(hipFree(saveForceX));
    cutilSafeCall(hipFree(saveForceY));
    cutilSafeCall(hipFree(saveForceZ));
  }
  
  cutilSafeCall(hipUnbindTexture(texvecino0GPU));
  cutilSafeCall(hipUnbindTexture(texvecino1GPU));
  cutilSafeCall(hipUnbindTexture(texvecino2GPU));
  cutilSafeCall(hipUnbindTexture(texvecino3GPU));
  cutilSafeCall(hipUnbindTexture(texvecino4GPU));
  cutilSafeCall(hipUnbindTexture(texvecino5GPU));
  cutilSafeCall(hipUnbindTexture(texvecinopxpyGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopxmyGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopxpzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopxmzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomxpyGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomxmyGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomxpzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomxmzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopypzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopymzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomypzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomymzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopxpypzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopxpymzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopxmypzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopxmymzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomxpypzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomxpymzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomxmypzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomxmymzGPU));

  hipFree(vecino0GPU);
  hipFree(vecino1GPU);
  hipFree(vecino2GPU);
  hipFree(vecino3GPU);
  hipFree(vecino4GPU);
  hipFree(vecino5GPU);

  hipFree(vecinopxpyGPU);
  hipFree(vecinopxmyGPU);
  hipFree(vecinopxpzGPU);
  hipFree(vecinopxmzGPU);
  hipFree(vecinomxpyGPU);
  hipFree(vecinomxmyGPU);
  hipFree(vecinomxpzGPU);
  hipFree(vecinomxmzGPU);  
  hipFree(vecinopypzGPU);
  hipFree(vecinopymzGPU);
  hipFree(vecinomypzGPU);
  hipFree(vecinomymzGPU);
  hipFree(vecinopxpypzGPU);
  hipFree(vecinopxpymzGPU);
  hipFree(vecinopxmypzGPU);
  hipFree(vecinopxmymzGPU);
  hipFree(vecinomxpypzGPU);
  hipFree(vecinomxpymzGPU);
  hipFree(vecinomxmypzGPU);
  hipFree(vecinomxmymzGPU);

  hipFree(countparticlesincellX);
  hipFree(countparticlesincellY);
  hipFree(countparticlesincellZ);
  hipFree(partincellX);
  hipFree(partincellY);
  hipFree(partincellZ);
  cutilSafeCall(hipFree(errorKernel));
  cutilSafeCall(hipFree(stepGPU));

  if(setCheckVelocity==1){
    cutilSafeCall(hipFree(rxCheckGPU));
    cutilSafeCall(hipFree(ryCheckGPU));
    cutilSafeCall(hipFree(rzCheckGPU));
    cutilSafeCall(hipFree(vxCheckGPU));
    cutilSafeCall(hipFree(vyCheckGPU));
    cutilSafeCall(hipFree(vzCheckGPU));
  }


  if(!freeOtherFluidVariablesGPU()) return 0;

  cout << "FREE MEMORY GPU :               DONE" << endl; 


  return 1;
}

