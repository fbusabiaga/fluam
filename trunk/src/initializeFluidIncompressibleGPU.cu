// Filename: initializeFluidIncompressibleGPU.cu
//
// Copyright (c) 2010-2012, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


bool initializeFluidIncompressibleGPU(){
  
  hipMemcpy(vxGPU,cvx,ncells*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(vyGPU,cvy,ncells*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(vzGPU,cvz,ncells*sizeof(double),hipMemcpyHostToDevice);

  hipMemcpy(rxcellGPU,crx,ncells*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(rycellGPU,cry,ncells*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(rzcellGPU,crz,ncells*sizeof(double),hipMemcpyHostToDevice);

  if(incompressibleBinaryMixture || incompressibleBinaryMixtureMidPoint)
    hipMemcpy(cGPU,c,ncells*sizeof(double),hipMemcpyHostToDevice);

  
  cout << "INITIALIZE FLUID GPU :          DONE" << endl;

  return 1;
}
