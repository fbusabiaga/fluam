// Filename: runSchemeIncompressibleBoundary.cu
//
// Copyright (c) 2010-2013, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


bool runSchemeIncompressibleBoundary(){
  int threadsPerBlock = 128;
  if((ncells/threadsPerBlock) < 60) threadsPerBlock = 64;
  if((ncells/threadsPerBlock) < 60) threadsPerBlock = 32;
  int numBlocks = (ncells-1)/threadsPerBlock + 1;

  int threadsPerBlockBoundary = 128;
  if((nboundary/threadsPerBlockBoundary) < 60) threadsPerBlockBoundary = 64;
  if((nboundary/threadsPerBlockBoundary) < 60) threadsPerBlockBoundary = 32;
  int numBlocksBoundary = (nboundary-1)/threadsPerBlockBoundary + 1;

  int threadsPerBlockPartAndBoundary = 128;
  if(((np+nboundary)/threadsPerBlockPartAndBoundary) < 60) threadsPerBlockPartAndBoundary = 64;
  if(((np+nboundary)/threadsPerBlockPartAndBoundary) < 60) threadsPerBlockPartAndBoundary = 32;
  int numBlocksPartAndBoundary = (np+nboundary-1)/threadsPerBlockPartAndBoundary + 1;

  int threadsPerBlockParticles = 128;
  if((np/threadsPerBlockParticles) < 60) threadsPerBlockParticles = 64;
  if((np/threadsPerBlockParticles) < 60) threadsPerBlockParticles = 32;
  int numBlocksParticles = (np-1)/threadsPerBlockParticles + 1;

  int threadsPerBlockNeighbors, numBlocksNeighbors;
  if(ncells>numNeighbors){
    threadsPerBlockNeighbors = 128;
    if((ncells/threadsPerBlockNeighbors) < 60) threadsPerBlockNeighbors = 64;
    if((ncells/threadsPerBlockNeighbors) < 60) threadsPerBlockNeighbors = 32;
    numBlocksNeighbors = (ncells-1)/threadsPerBlockNeighbors + 1;
  }
  else{
    threadsPerBlockNeighbors = 128;
    if((numNeighbors/threadsPerBlockNeighbors) < 60) threadsPerBlockNeighbors = 64;
    if((numNeighbors/threadsPerBlockNeighbors) < 60) threadsPerBlockNeighbors = 32;
    numBlocksNeighbors = (numNeighbors-1)/threadsPerBlockNeighbors + 1;
  }

  step = -numstepsRelaxation;

  //initialize random numbers
  size_t numberRandom = 6 * ncells;
  if(!initializeRandomNumbersGPU(numberRandom,seed)) return 0;

  //Initialize textures cells
  if(!texturesCells()) return 0;  

  initializeVecinos<<<numBlocks,threadsPerBlock>>>(vecino1GPU,vecino2GPU,vecino3GPU,vecino4GPU,
						   vecinopxpyGPU,vecinopxmyGPU,vecinopxpzGPU,vecinopxmzGPU,
						   vecinomxpyGPU,vecinomxmyGPU,vecinomxpzGPU,vecinomxmzGPU,
						   vecinopypzGPU,vecinopymzGPU,vecinomypzGPU,vecinomymzGPU,
						   vecinopxpypzGPU,vecinopxpymzGPU,vecinopxmypzGPU,
						   vecinopxmymzGPU,
						   vecinomxpypzGPU,vecinomxpymzGPU,vecinomxmypzGPU,
						   vecinomxmymzGPU);
  initializeVecinos2<<<numBlocks,threadsPerBlock>>>(vecino0GPU,vecino1GPU,vecino2GPU,
						    vecino3GPU,vecino4GPU,vecino5GPU);


  //Initialize plan
  hipfftHandle FFT;
  hipfftPlan3d(&FFT,mz,my,mx,HIPFFT_Z2Z);

  //Initialize factors for update in fourier space
  int threadsPerBlockdim, numBlocksdim;
  if((mx>=my)&&(mx>=mz)){
    threadsPerBlockdim = 128;
    numBlocksdim = (mx-1)/threadsPerBlockdim + 1;
  }
  else if((my>=mz)){
    threadsPerBlockdim = 128;
    numBlocksdim = (my-1)/threadsPerBlockdim + 1;
  }
  else{
    threadsPerBlockdim = 128;
    numBlocksdim = (mz-1)/threadsPerBlockdim + 1;
  }
  initializePrefactorFourierSpace_1<<<1,1>>>(gradKx,gradKy,gradKz,expKx,expKy,expKz,pF);
  initializePrefactorFourierSpace_2<<<numBlocksdim,threadsPerBlockdim>>>(pF);









  // A. Donev: Project the initial velocity to make sure it is div-free
  //---------------------------------------------------------
  //Copy velocities to complex variable
  doubleToDoubleComplex<<<numBlocks,threadsPerBlock>>>(vxGPU,vyGPU,vzGPU,vxZ,vyZ,vzZ);

  //Take velocities to fourier space
  hipfftExecZ2Z(FFT,vxZ,vxZ,HIPFFT_FORWARD);//W
  hipfftExecZ2Z(FFT,vyZ,vyZ,HIPFFT_FORWARD);//W
  hipfftExecZ2Z(FFT,vzZ,vzZ,HIPFFT_FORWARD);//W
  kernelShift<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,pF,-1);

  //Project into divergence free space
  projectionDivergenceFree<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,pF);

  //Take velocities to real space
  kernelShift<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,pF,1);
  hipfftExecZ2Z(FFT,vxZ,vxZ,HIPFFT_BACKWARD);
  hipfftExecZ2Z(FFT,vyZ,vyZ,HIPFFT_BACKWARD);
  hipfftExecZ2Z(FFT,vzZ,vzZ,HIPFFT_BACKWARD);

  //Copy velocities to real variables
  doubleComplexToDoubleNormalized<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,vxGPU,vyGPU,vzGPU);
  //---------------------------------------------------------










  while(step<numsteps){
    //Generate random numbers
    generateRandomNumbers(numberRandom);
    //First substep
    kernelConstructW_1<<<numBlocks,threadsPerBlock>>>(vxGPU,vyGPU,vzGPU,vxZ,vyZ,vzZ,dRand);//W
    hipfftExecZ2Z(FFT,vxZ,vxZ,HIPFFT_FORWARD);//W
    hipfftExecZ2Z(FFT,vyZ,vyZ,HIPFFT_FORWARD);//W
    hipfftExecZ2Z(FFT,vzZ,vzZ,HIPFFT_FORWARD);//W
    kernelShift<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,pF,-1);//W
    kernelUpdateVIncompressible<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,vxZ,vyZ,vzZ,pF);//W
    kernelShift<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,pF,1);
    hipfftExecZ2Z(FFT,vxZ,vxZ,HIPFFT_BACKWARD);
    hipfftExecZ2Z(FFT,vyZ,vyZ,HIPFFT_BACKWARD);
    hipfftExecZ2Z(FFT,vzZ,vzZ,HIPFFT_BACKWARD);
    calculateVelocityPrediction<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,
							       vxPredictionGPU,vyPredictionGPU,
							       vzPredictionGPU);     
    //Second substep
    kernelConstructW_2<<<numBlocks,threadsPerBlock>>>(vxPredictionGPU,vyPredictionGPU,vzPredictionGPU,
						      vxZ,vyZ,vzZ,dRand);//W
    hipfftExecZ2Z(FFT,vxZ,vxZ,HIPFFT_FORWARD);//W
    hipfftExecZ2Z(FFT,vyZ,vyZ,HIPFFT_FORWARD);//W
    hipfftExecZ2Z(FFT,vzZ,vzZ,HIPFFT_FORWARD);//W
    kernelShift<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,pF,-1);//W
    kernelUpdateVIncompressible<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,vxZ,vyZ,vzZ,pF);//W
    kernelShift<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,pF,1);
    hipfftExecZ2Z(FFT,vxZ,vxZ,HIPFFT_BACKWARD);
    hipfftExecZ2Z(FFT,vyZ,vyZ,HIPFFT_BACKWARD);
    hipfftExecZ2Z(FFT,vzZ,vzZ,HIPFFT_BACKWARD);
    doubleComplexToDoubleNormalized<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,
								   vxGPU,vyGPU,
								   vzGPU);
    //Boundaries and particles part start
    boundaryParticlesFunctionIncompressible(numBlocksBoundary,threadsPerBlockBoundary,
					    numBlocksNeighbors,threadsPerBlockNeighbors,
					    numBlocksPartAndBoundary,threadsPerBlockPartAndBoundary,
					    numBlocksParticles,threadsPerBlockParticles,
					    numBlocks,threadsPerBlock);
    //Apply incompressibility again
    doubleToDoubleComplex<<<numBlocks,threadsPerBlock>>>(vxGPU,vyGPU,vzGPU,vxZ,vyZ,vzZ);
    hipfftExecZ2Z(FFT,vxZ,vxZ,HIPFFT_FORWARD);
    hipfftExecZ2Z(FFT,vyZ,vyZ,HIPFFT_FORWARD);
    hipfftExecZ2Z(FFT,vzZ,vzZ,HIPFFT_FORWARD);
    kernelShift<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,pF,-1);
    projectionDivergenceFree<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,pF);
    kernelShift<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,pF,1);
    hipfftExecZ2Z(FFT,vxZ,vxZ,HIPFFT_BACKWARD);
    hipfftExecZ2Z(FFT,vyZ,vyZ,HIPFFT_BACKWARD);
    hipfftExecZ2Z(FFT,vzZ,vzZ,HIPFFT_BACKWARD);
    doubleComplexToDoubleNormalized<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,
								   vxGPU,vyGPU,
								   vzGPU);

    step++;
    if(!(step%samplefreq)&&(step>0)){
      cout << "INCOMPRESSIBLE BOUNDARY  " << step << endl;
      if(!gpuToHostIncompressibleBoundary()) return 0;
      if(!saveFunctionsSchemeIncompressibleBoundary(1,step)) return 0;
      //if(!saveFunctionsSchemeIncompressible(1)) return 0;
    }
  }



  //Free FFT
  hipfftDestroy(FFT);
  freeRandomNumbersGPU();

  

  return 1;
}
