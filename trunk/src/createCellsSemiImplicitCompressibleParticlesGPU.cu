#include "hip/hip_runtime.h"
// Filename: createCellsGPU.cu
//
// Copyright (c) 2010-2013, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.



#define GPUVARIABLES 1


bool createCellsSemiImplicitCompressibleParticlesGPU(){



  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mxGPU),&mx,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(myGPU),&my,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mzGPU),&mz,sizeof(int)));

  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mxtGPU),&mxt,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mytGPU),&myt,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mztGPU),&mzt,sizeof(int)));

  int aux = (mxt) * (myt);
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mxmytGPU),&aux,sizeof(int)));


  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ncellsGPU),&ncells,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ncellstGPU),&ncellst,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(lxGPU),&lx,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(lyGPU),&ly,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(lzGPU),&lz,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dtGPU),&dt,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(volumeGPU),&cVolume,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(shearviscosityGPU),&shearviscosity,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(bulkviscosityGPU),&bulkviscosity,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(temperatureGPU),&temperature,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(pressurea0GPU),&pressurea0,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(pressurea1GPU),&pressurea1,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(pressurea2GPU),&pressurea2,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(thermostatGPU),&thermostat,sizeof(bool)));

  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(densfluidGPU),&densfluid,sizeof(double)));



  cutilSafeCall(hipMalloc((void**)&densityGPU,ncellst*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vxGPU,ncellst*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vyGPU,ncellst*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vzGPU,ncellst*sizeof(double)));

  cutilSafeCall(hipMalloc((void**)&vxPredictionGPU,ncellst*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vyPredictionGPU,ncellst*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vzPredictionGPU,ncellst*sizeof(double)));

 

  cutilSafeCall(hipMalloc((void**)&dpxGPU,ncellst*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&dpyGPU,ncellst*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&dpzGPU,ncellst*sizeof(double)));

  cutilSafeCall(hipMalloc((void**)&rxcellGPU,ncellst*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&rycellGPU,ncellst*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&rzcellGPU,ncellst*sizeof(double)));

  double fact1 = sqrt((4.*temperature*shearviscosity)/(dt*cVolume));
  double fact2 = sqrt((2.*temperature*bulkviscosity)/(3.*dt*cVolume));
  double fact3 = bulkviscosity - 2. * shearviscosity/3.;
  double fact4 = sqrt((2.*temperature*shearviscosity)/(dt*cVolume));
  double fact5 = sqrt(1./(dt*cVolume));

  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(fact1GPU),&fact1,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(fact2GPU),&fact2,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(fact3GPU),&fact3,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(fact4GPU),&fact4,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(fact5GPU),&fact5,sizeof(double)));


  fact1 = lx/double(mx);
  fact2 = ly/double(my);
  fact3 = lz/double(mz);
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dxGPU),&fact1,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dyGPU),&fact2,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dzGPU),&fact3,sizeof(double)));

  fact1 = double(mx)/lx;
  fact2 = double(my)/ly;
  fact3 = double(mz)/lz;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invdxGPU),&fact1,sizeof(double)));  
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invdyGPU),&fact2,sizeof(double)));  
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invdzGPU),&fact3,sizeof(double)));  
  fact1 = 1./dt;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invdtGPU),&fact1,sizeof(double)));
  fact1 = 1./lx;
  fact2 = 1./ly;
  fact3 = 1./lz;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invlxGPU),&fact1,sizeof(double)));  
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invlyGPU),&fact2,sizeof(double)));  
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invlzGPU),&fact3,sizeof(double)));

 
  bool auxbool = 0;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(setparticlesGPU),&auxbool,sizeof(bool)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(setboundaryGPU),&auxbool,sizeof(bool)));


  long long auxulonglong = 0;
  cutilSafeCall(hipMalloc((void**)&stepGPU,sizeof(long long)));
  cutilSafeCall(hipMemcpy(stepGPU,&auxulonglong,sizeof(long long),hipMemcpyHostToDevice));


  cutilSafeCall(hipMalloc((void**)&vecino0GPU,ncellst*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&vecino1GPU,ncellst*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&vecino2GPU,ncellst*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&vecino3GPU,ncellst*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&vecino4GPU,ncellst*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&vecino5GPU,ncellst*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&vecinopxpyGPU,ncellst*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopxmyGPU,ncellst*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopxpzGPU,ncellst*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopxmzGPU,ncellst*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomxpyGPU,ncellst*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomxmyGPU,ncellst*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomxpzGPU,ncellst*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomxmzGPU,ncellst*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopypzGPU,ncellst*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopymzGPU,ncellst*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomypzGPU,ncellst*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomymzGPU,ncellst*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopxpypzGPU,ncellst*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopxpymzGPU,ncellst*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopxmypzGPU,ncellst*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopxmymzGPU,ncellst*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomxpypzGPU,ncellst*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomxpymzGPU,ncellst*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&vecinomxmypzGPU,ncellst*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomxmymzGPU,ncellst*sizeof(int))); 






  //Factors for the update in fourier space
  cutilSafeCall(hipMalloc((void**)&gradKx,     mx*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&gradKy,     my*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&gradKz,     mz*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&expKx,      mx*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&expKy,      my*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&expKz,      mz*sizeof(hipfftDoubleComplex)));

  cutilSafeCall(hipMalloc((void**)&pF,sizeof(prefactorsFourier)));

  cutilSafeCall(hipMalloc((void**)&vxZ,ncells*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&vyZ,ncells*sizeof(hipfftDoubleComplex)));




  double auxD[ncellst];
  for(int i=0;i<ncellst;i++) auxD[i] = 0;
  hipMemcpy(dpxGPU,auxD,ncellst*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(dpyGPU,auxD,ncellst*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(dpzGPU,auxD,ncellst*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(vxPredictionGPU,auxD,ncellst*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(vyPredictionGPU,auxD,ncellst*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(vzPredictionGPU,auxD,ncellst*sizeof(double),hipMemcpyHostToDevice);

  hipfftDoubleComplex auxC[ncellst];
  for(int i=0;i<ncellst;i++){
    auxC[i].x = 1;
    auxC[i].y = 0;
  }
  hipMemcpy(vxZ,auxC,ncellst*sizeof(hipfftDoubleComplex),hipMemcpyHostToDevice);
  hipMemcpy(vyZ,auxC,ncellst*sizeof(hipfftDoubleComplex),hipMemcpyHostToDevice);


  
  cout << "CREATE CELLS GPU :              DONE" << endl;

  return 1;
}




