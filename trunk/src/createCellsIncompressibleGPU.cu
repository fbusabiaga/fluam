#include "hip/hip_runtime.h"
// Filename: createCellsIncompressibleGPU.cu
//
// Copyright (c) 2010-2012, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


#define GPUVARIABLES 1


bool createCellsIncompressibleGPU(){
  hipMemcpyToSymbol(HIP_SYMBOL(mxGPU),&mx,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(myGPU),&my,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(mzGPU),&mz,sizeof(int));

  hipMemcpyToSymbol(HIP_SYMBOL(mxtGPU),&mxt,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(mytGPU),&myt,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(mztGPU),&mzt,sizeof(int));



  hipMemcpyToSymbol(HIP_SYMBOL(ncellsGPU),&ncells,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(ncellstGPU),&ncellst,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(lxGPU),&lx,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(lyGPU),&ly,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(lzGPU),&lz,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(dtGPU),&dt,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(volumeGPU),&cVolume,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(shearviscosityGPU),&shearviscosity,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(temperatureGPU),&temperature,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(thermostatGPU),&thermostat,sizeof(bool));

  hipMemcpyToSymbol(HIP_SYMBOL(densfluidGPU),&densfluid,sizeof(double));

  hipMalloc((void**)&vxGPU,ncells*sizeof(double));
  hipMalloc((void**)&vyGPU,ncells*sizeof(double));
  hipMalloc((void**)&vzGPU,ncells*sizeof(double));
  hipMalloc((void**)&vxPredictionGPU,ncells*sizeof(double));
  hipMalloc((void**)&vyPredictionGPU,ncells*sizeof(double));
  hipMalloc((void**)&vzPredictionGPU,ncells*sizeof(double));

 
  hipMalloc((void**)&rxcellGPU,ncells*sizeof(double));
  hipMalloc((void**)&rycellGPU,ncells*sizeof(double));
  hipMalloc((void**)&rzcellGPU,ncells*sizeof(double));

  //FACT1 DIFFERENT FOR INCOMPRESSIBLE
  double fact1 = sqrt((4.*temperature*shearviscosity*dt)/(cVolume*densfluid*densfluid));
  //FACT4 DIFFERENT FOR INCOMPRESSIBLE
  double fact4 = sqrt((2.*temperature*shearviscosity*dt)/(cVolume*densfluid*densfluid));
  double fact5 = sqrt(1./(dt*cVolume));

  hipMemcpyToSymbol(HIP_SYMBOL(fact1GPU),&fact1,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(fact4GPU),&fact4,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(fact5GPU),&fact5,sizeof(double));


  fact1 = lx/double(mx);
  double fact2 = ly/double(my);
  double fact3 = lz/double(mz);
  hipMemcpyToSymbol(HIP_SYMBOL(dxGPU),&fact1,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(dyGPU),&fact2,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(dzGPU),&fact3,sizeof(double));

  fact1 = double(mx)/lx;
  fact2 = double(my)/ly;
  fact3 = double(mz)/lz;
  hipMemcpyToSymbol(HIP_SYMBOL(invdxGPU),&fact1,sizeof(double));  
  hipMemcpyToSymbol(HIP_SYMBOL(invdyGPU),&fact2,sizeof(double));  
  hipMemcpyToSymbol(HIP_SYMBOL(invdzGPU),&fact3,sizeof(double));  
  fact1 = 1./dt;
  hipMemcpyToSymbol(HIP_SYMBOL(invdtGPU),&fact1,sizeof(double));
  fact1 = 1./lx;
  fact2 = 1./ly;
  fact3 = 1./lz;
  hipMemcpyToSymbol(HIP_SYMBOL(invlxGPU),&fact1,sizeof(double));  
  hipMemcpyToSymbol(HIP_SYMBOL(invlyGPU),&fact2,sizeof(double));  
  hipMemcpyToSymbol(HIP_SYMBOL(invlzGPU),&fact3,sizeof(double));

 
  bool auxbool = 0;
  hipMemcpyToSymbol(HIP_SYMBOL(setparticlesGPU),&auxbool,sizeof(bool));
  hipMemcpyToSymbol(HIP_SYMBOL(setboundaryGPU),&auxbool,sizeof(bool));


  long long auxulonglong = 0;
  hipMalloc((void**)&stepGPU,sizeof(long long));
  hipMemcpy(stepGPU,&auxulonglong,sizeof(long long),hipMemcpyHostToDevice);


  hipMalloc((void**)&vecino0GPU,ncells*sizeof(int));
  hipMalloc((void**)&vecino1GPU,ncells*sizeof(int));
  hipMalloc((void**)&vecino2GPU,ncells*sizeof(int));
  hipMalloc((void**)&vecino3GPU,ncells*sizeof(int));
  hipMalloc((void**)&vecino4GPU,ncells*sizeof(int));
  hipMalloc((void**)&vecino5GPU,ncells*sizeof(int));
  hipMalloc((void**)&vecinopxpyGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinopxmyGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinopxpzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinopxmzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinomxpyGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinomxmyGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinomxpzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinomxmzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinopypzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinopymzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinomypzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinomymzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinopxpypzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinopxpymzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinopxmypzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinopxmymzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinomxpypzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinomxpymzGPU,ncells*sizeof(int));
  hipMalloc((void**)&vecinomxmypzGPU,ncells*sizeof(int)); 
  hipMalloc((void**)&vecinomxmymzGPU,ncells*sizeof(int)); 


  //Factors for the update in fourier space
  hipMalloc((void**)&gradKx,     mx*sizeof(hipfftDoubleComplex));
  hipMalloc((void**)&gradKy,     my*sizeof(hipfftDoubleComplex));
  hipMalloc((void**)&gradKz,     mz*sizeof(hipfftDoubleComplex));
  hipMalloc((void**)&expKx,      mx*sizeof(hipfftDoubleComplex));
  hipMalloc((void**)&expKy,      my*sizeof(hipfftDoubleComplex));
  hipMalloc((void**)&expKz,      mz*sizeof(hipfftDoubleComplex));

  hipMalloc((void**)&pF,sizeof(prefactorsFourier));

  //hipMalloc((void**)&WxZ,ncells*sizeof(hipfftDoubleComplex));
  //hipMalloc((void**)&WyZ,ncells*sizeof(hipfftDoubleComplex));
  //hipMalloc((void**)&WzZ,ncells*sizeof(hipfftDoubleComplex));
  hipMalloc((void**)&vxZ,ncells*sizeof(hipfftDoubleComplex));
  hipMalloc((void**)&vyZ,ncells*sizeof(hipfftDoubleComplex));
  hipMalloc((void**)&vzZ,ncells*sizeof(hipfftDoubleComplex)); 

  if(quasiNeutrallyBuoyant){
    hipMalloc((void**)&advXGPU,ncells*sizeof(double));
    hipMalloc((void**)&advYGPU,ncells*sizeof(double));
    hipMalloc((void**)&advZGPU,ncells*sizeof(double));
  }

  cout << "CREATE CELLS GPU :              DONE" << endl;

  return 1;
}
