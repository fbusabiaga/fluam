// Filename: gpuToHostIncompressibleBoundary.cu
//
// Copyright (c) 2010-2013, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


bool gpuToHostIncompressibleBoundary(){

  int auxb[5];
  cutilSafeCall(hipMemcpy(auxb,errorKernel,5*sizeof(int),hipMemcpyDeviceToHost));
  if(auxb[0] == 1){
    for(int i=0;i<5;i++){
      cout << "ERROR IN KERNEL " << i << " " << auxb[i] << endl;
    }
    return 0;
  }

  cutilSafeCall(hipMemcpy(cvx,vxGPU,ncells*sizeof(double),hipMemcpyDeviceToHost));
  cutilSafeCall(hipMemcpy(cvy,vyGPU,ncells*sizeof(double),hipMemcpyDeviceToHost));
  cutilSafeCall(hipMemcpy(cvz,vzGPU,ncells*sizeof(double),hipMemcpyDeviceToHost));

  if(setparticles){
    cutilSafeCall(hipMemcpy(rxParticle,&rxboundaryGPU[nboundary],np*sizeof(double),hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(ryParticle,&ryboundaryGPU[nboundary],np*sizeof(double),hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(rzParticle,&rzboundaryGPU[nboundary],np*sizeof(double),hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(vxParticle,&vxboundaryGPU[nboundary],np*sizeof(double),hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(vyParticle,&vyboundaryGPU[nboundary],np*sizeof(double),hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(vzParticle,&vzboundaryGPU[nboundary],np*sizeof(double),hipMemcpyDeviceToHost));
  }

  return 1;
}
