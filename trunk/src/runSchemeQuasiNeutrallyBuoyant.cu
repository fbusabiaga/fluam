// Filename: runSchemeQuasiNeutrallyBuoyant.cu
//
// Copyright (c) 2010-2013, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


bool runSchemeQuasiNeutrallyBuoyant(){
  int threadsPerBlock = 128;
  if((ncells/threadsPerBlock) < 60) threadsPerBlock = 64;
  if((ncells/threadsPerBlock) < 60) threadsPerBlock = 32;
  int numBlocks = (ncells-1)/threadsPerBlock + 1;

  int threadsPerBlockParticles = 128;
  if((np/threadsPerBlockParticles) < 60) threadsPerBlockParticles = 64;
  if((np/threadsPerBlockParticles) < 60) threadsPerBlockParticles = 32;
  int numBlocksParticles = (np-1)/threadsPerBlockParticles + 1;

  int threadsPerBlockNeighbors, numBlocksNeighbors;
  if(ncells>numNeighbors){
    threadsPerBlockNeighbors = 128;
    if((ncells/threadsPerBlockNeighbors) < 60) threadsPerBlockNeighbors = 64;
    if((ncells/threadsPerBlockNeighbors) < 60) threadsPerBlockNeighbors = 32;
    numBlocksNeighbors = (ncells-1)/threadsPerBlockNeighbors + 1;
  }
  else{
    threadsPerBlockNeighbors = 128;
    if((numNeighbors/threadsPerBlockNeighbors) < 60) threadsPerBlockNeighbors = 64;
    if((numNeighbors/threadsPerBlockNeighbors) < 60) threadsPerBlockNeighbors = 32;
    numBlocksNeighbors = (numNeighbors-1)/threadsPerBlockNeighbors + 1;
  }

  step = -numstepsRelaxation;

  //initialize random numbers
  size_t numberRandom = 6 * ncells;
  if(!initializeRandomNumbersGPU(numberRandom,seed)) return 0;

  //Initialize textures cells
  if(!texturesCells()) return 0;  

  initializeVecinos<<<numBlocks,threadsPerBlock>>>(vecino1GPU,
						   vecino2GPU,
						   vecino3GPU,
						   vecino4GPU,
						   vecinopxpyGPU,
						   vecinopxmyGPU,
						   vecinopxpzGPU,
						   vecinopxmzGPU,
						   vecinomxpyGPU,
						   vecinomxmyGPU,
						   vecinomxpzGPU,
						   vecinomxmzGPU,
						   vecinopypzGPU,
						   vecinopymzGPU,
						   vecinomypzGPU,
						   vecinomymzGPU,
						   vecinopxpypzGPU,
						   vecinopxpymzGPU,
						   vecinopxmypzGPU,
						   vecinopxmymzGPU,
						   vecinomxpypzGPU,
						   vecinomxpymzGPU,
						   vecinomxmypzGPU,
						   vecinomxmymzGPU);
  
  initializeVecinos2<<<numBlocks,threadsPerBlock>>>(vecino0GPU,
						    vecino1GPU,
						    vecino2GPU,
						    vecino3GPU,
						    vecino4GPU,
						    vecino5GPU);


  //Initialize plan
  hipfftHandle FFT;
  hipfftPlan3d(&FFT,mz,my,mx,HIPFFT_Z2Z);

  //Initialize factors for fourier space update
  int threadsPerBlockdim, numBlocksdim;
  if((mx>=my)&&(mx>=mz)){
    threadsPerBlockdim = 128;
    numBlocksdim = (mx-1)/threadsPerBlockdim + 1;
  }
  else if((my>=mz)){
    threadsPerBlockdim = 128;
    numBlocksdim = (my-1)/threadsPerBlockdim + 1;
  }
  else{
    threadsPerBlockdim = 128;
    numBlocksdim = (mz-1)/threadsPerBlockdim + 1;
  }
  initializePrefactorFourierSpace_1<<<1,1>>>(gradKx,
					     gradKy,
					     gradKz,
					     expKx,
					     expKy,
					     expKz,pF);
  
  initializePrefactorFourierSpace_2<<<numBlocksdim,threadsPerBlockdim>>>(pF);









  // A. Donev: Project the initial velocity to make sure it is div-free
  //---------------------------------------------------------
  //Copy velocities to complex variable
  doubleToDoubleComplex<<<numBlocks,threadsPerBlock>>>(vxGPU,vyGPU,vzGPU,vxZ,vyZ,vzZ);

  //Take velocities to fourier space
  hipfftExecZ2Z(FFT,vxZ,vxZ,HIPFFT_FORWARD);//W
  hipfftExecZ2Z(FFT,vyZ,vyZ,HIPFFT_FORWARD);//W
  hipfftExecZ2Z(FFT,vzZ,vzZ,HIPFFT_FORWARD);//W
  kernelShift<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,pF,-1);

  //Project into divergence free space
  projectionDivergenceFree<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,pF);

  //Take velocities to real space
  kernelShift<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,pF,1);
  hipfftExecZ2Z(FFT,vxZ,vxZ,HIPFFT_BACKWARD);
  hipfftExecZ2Z(FFT,vyZ,vyZ,HIPFFT_BACKWARD);
  hipfftExecZ2Z(FFT,vzZ,vzZ,HIPFFT_BACKWARD);

  //Copy velocities to real variables
  doubleComplexToDoubleNormalized<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,vxGPU,vyGPU,vzGPU);
  //---------------------------------------------------------










  //First step. We use mid-point rule for the advection in the
  //first step, after that we continue with the 
  //Adams-Bashforth rule
  if(!firstStepQuasiNeutrallyBuoyant(numBlocksNeighbors,
				     threadsPerBlockNeighbors,
				     numBlocksParticles,
				     threadsPerBlockParticles,
				     numBlocks,
				     threadsPerBlock,
				     numberRandom,
				     FFT,
				     step)) return 0;

  while(step<numsteps){
    
    //
    //
    //
    //
    //
    //
    //Generate random numbers
    generateRandomNumbers(numberRandom);
    




    
    //
    //
    //
    //
    //
    //
    //STEP 1: UPDATE PARTICLE POSITIONS TO  q^{n+1/2}
    //Clear neighbor lists
    countToZero<<<numBlocksNeighbors,threadsPerBlockNeighbors>>>(pc);
    
    //Update particle positions q^{n+1/2} = q^n + dt * J^n * v^n
    //saved in rxboundaryPredictionGPU
    findNeighborParticlesQuasiNeutrallyBuoyant_1<<<numBlocksParticles,threadsPerBlockParticles>>>
      (pc, 
       errorKernel,
       rxcellGPU,
       rycellGPU,
       rzcellGPU,
       rxboundaryGPU,  //q^{n}
       ryboundaryGPU, 
       rzboundaryGPU,
       rxboundaryPredictionGPU, //q^{n+1/2}
       ryboundaryPredictionGPU, 
       rzboundaryPredictionGPU,
       vxGPU, //v^n
       vyGPU, 
       vzGPU);

    //Load textures with particles position q^{n+1/2}
    cutilSafeCall( hipBindTexture(0,texrxboundaryGPU,rxboundaryPredictionGPU,(nboundary+np)*sizeof(double)));
    cutilSafeCall( hipBindTexture(0,texryboundaryGPU,ryboundaryPredictionGPU,(nboundary+np)*sizeof(double)));
    cutilSafeCall( hipBindTexture(0,texrzboundaryGPU,rzboundaryPredictionGPU,(nboundary+np)*sizeof(double)));






    //
    //
    //
    //
    //
    //
    //STEP 2: CALCULATE FORCES AND SPREAD THEM TO THE FLUID S^{n+1/2} * F^{n+1/2}
    //Fill "countparticlesincellX" lists
    //and spread particle force F 
    kernelSpreadParticlesForce<<<numBlocksParticles,threadsPerBlockParticles>>>(rxcellGPU,
										rycellGPU,
										rzcellGPU,
										fxboundaryGPU,
										fyboundaryGPU,
										fzboundaryGPU,
										pc,
										errorKernel,
										bFV);    






    //
    //
    //
    //
    //
    //
    //STEP 3: SOLVE UNPERTURBED FLUID MOMENTUM
    //Construct vector W
    // W = v^n + 0.5*dt*nu*L*v^n + Advection(v^n) + (dt/rho)*f^n_{noise} + dt*SF/rho + dt*(m_e/rho)*div*Suu
    //and save advection
    kernelConstructWQuasiNeutrallyBuoyantTEST5_3<<<numBlocks,threadsPerBlock>>>(vxPredictionGPU,
										vyPredictionGPU,
										vzPredictionGPU,
										vxZ,//W
										vyZ,//W
										vzZ,//W
										dRand,
										fxboundaryGPU,
										fyboundaryGPU,
										fzboundaryGPU,
										advXGPU,
										advYGPU,
										advZGPU);
									      
    
    //Calculate velocity prediction with incompressibility "\tilde{v}^{n+1}"
    //Go to fourier space
    hipfftExecZ2Z(FFT,vxZ,vxZ,HIPFFT_FORWARD);//W
    hipfftExecZ2Z(FFT,vyZ,vyZ,HIPFFT_FORWARD);//W
    hipfftExecZ2Z(FFT,vzZ,vzZ,HIPFFT_FORWARD);//W
    //Apply shift for the staggered grid
    kernelShift<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,pF,-1);//W
    //Update fluid velocity
    kernelUpdateVIncompressible<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,vxZ,vyZ,vzZ,pF);//W
    //Apply shift for the staggered grid
    kernelShift<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,pF,1);
    //Come back to real space
    hipfftExecZ2Z(FFT,vxZ,vxZ,HIPFFT_BACKWARD);
    hipfftExecZ2Z(FFT,vyZ,vyZ,HIPFFT_BACKWARD);
    hipfftExecZ2Z(FFT,vzZ,vzZ,HIPFFT_BACKWARD);

    //Store velocity prediction "\tilde{v}^{n+1}" on vxPredictionGPU
    predictionVQuasiNeutrallyBuoyant<<<numBlocks,threadsPerBlock>>>(vxZ,
								    vyZ,
								    vzZ,
								    vxGPU,
								    vyGPU,
								    vzGPU,
								    vxPredictionGPU,
								    vyPredictionGPU,
								    vzPredictionGPU);
    
    //Load textures with velocity prediction "\tilde{v}^{n+1}"
    cutilSafeCall( hipBindTexture(0,texVxGPU,vxPredictionGPU,ncells*sizeof(double)));
    cutilSafeCall( hipBindTexture(0,texVyGPU,vyPredictionGPU,ncells*sizeof(double)));
    cutilSafeCall( hipBindTexture(0,texVzGPU,vzPredictionGPU,ncells*sizeof(double)));






    //
    //
    //
    //
    //
    //
    //STEP 4: IF EXCESS OF MASS != 0 CALCULATE 
    //\delta u^{n+1/2} = (J^{n+1/2} - J^{n}) * v^n + 0.5*nu*dt * J^{n-1/2} * L * \Delta v^{n-1/2}
    if(mass != 0){
      //Calculate \delta u^{n+1/2} and saved in vxboundaryPredictionGPU
      kernelCalculateDeltau<<<numBlocksParticles,threadsPerBlockParticles>>>(rxcellGPU,
									     rycellGPU,
									     rzcellGPU,
									     vxGPU,
									     vyGPU,
									     vzGPU,
									     rxboundaryGPU,
									     ryboundaryGPU,
									     rzboundaryGPU,
									     vxboundaryPredictionGPU,
									     vyboundaryPredictionGPU,
									     vzboundaryPredictionGPU);
    }






    //
    //
    //
    //
    //
    //
    //STEP 5: IF EXCESS OF MASS != 0 CALCULATE
    //\Delta p
    if(mass != 0 ){
      //Calculate \Delta p and saved in vxboundaryGPU
      kernelCalculateDeltap<<<numBlocksParticles,threadsPerBlockParticles>>>(rxcellGPU,
									     rycellGPU,
									     rzcellGPU,
									     vxboundaryGPU,
									     vyboundaryGPU,
									     vzboundaryGPU,
									     vxboundaryPredictionGPU,
									     vyboundaryPredictionGPU,
									     vzboundaryPredictionGPU);
    }






    //
    //
    //
    //
    //
    //
    //STEP 6: IF EXCESS OF MASS != 0 CALCULATE VELOCITY CORRECTION
    //\Delta \tilde{ v }
    if(mass != 0){
      //First, spread \Delta p, prefactor * S*{\Delta p}
      kernelSpreadDeltap<<<numBlocksParticles,threadsPerBlockParticles>>>(rxcellGPU,
									  rycellGPU,
									  rzcellGPU,
									  vxboundaryGPU,
									  vyboundaryGPU,
									  vzboundaryGPU,
									  fxboundaryGPU,
									  fyboundaryGPU,
									  fzboundaryGPU);

      //Second, add all terms prefactor*S*{\Delta p} and store in vxZ.x
      kernelCorrectionVQuasiNeutrallyBuoyant_2<<<numBlocks,threadsPerBlock>>>(vxZ,
									      vyZ,
									      vzZ,
									      fxboundaryGPU,
									      fyboundaryGPU,
									      fzboundaryGPU);

      //Third apply incompressibility to calculate \Delta \tilde{ v }
      hipfftExecZ2Z(FFT,vxZ,vxZ,HIPFFT_FORWARD);
      hipfftExecZ2Z(FFT,vyZ,vyZ,HIPFFT_FORWARD);
      hipfftExecZ2Z(FFT,vzZ,vzZ,HIPFFT_FORWARD);
      kernelShift<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,pF,-1);
      projectionDivergenceFree<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,pF);
      kernelShift<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,pF,1);
      hipfftExecZ2Z(FFT,vxZ,vxZ,HIPFFT_BACKWARD);
      hipfftExecZ2Z(FFT,vyZ,vyZ,HIPFFT_BACKWARD);
      hipfftExecZ2Z(FFT,vzZ,vzZ,HIPFFT_BACKWARD);
      //Store \Delta \tilde{ v } in vxZ.y
      saveDeltaTildev<<<numBlocks,threadsPerBlock>>>(vxZ,
						     vyZ,
						     vzZ);
    }
    





    //
    //
    //
    //
    //
    //
    //STEP 7: IF EXCESS OF MASS != 0 CALCULATE VELOCITY CORRECTION
    //\Delta v
    if(mass != 0){
      //First, spread S*(\Delta p - m_e*J*\Delta \tilde{ v })
      kernelSpreadDeltapMinusJTildev<<<numBlocksParticles,threadsPerBlockParticles>>>(rxcellGPU,
										      rycellGPU,
										      rzcellGPU,
										      vxZ,
										      vyZ,
										      vzZ,
										      vxboundaryGPU,
										      vyboundaryGPU,
										      vzboundaryGPU,
										      fxboundaryGPU,
										      fyboundaryGPU,
										      fzboundaryGPU);
      //The calculation of \Delta v has to wait, first we will
      //update the particle velocity
    }







    //
    //
    //
    //
    //
    //
    //STEP 8: UPDATE PARTICLE VELOCITY
    if(mass == 0){
      updateParticleVelocityme0<<<numBlocksParticles,threadsPerBlockParticles>>>(rxcellGPU,
										 rycellGPU,
										 rzcellGPU,
										 vxboundaryGPU,
										 vyboundaryGPU,
										 vzboundaryGPU);
    }
    else{
      updateParticleVelocityme<<<numBlocksParticles,threadsPerBlockParticles>>>(rxcellGPU,
										rycellGPU,
										rzcellGPU,
										vxZ,
										vyZ,
										vzZ,
										vxboundaryGPU,
										vyboundaryGPU,
										vzboundaryGPU,
										vxboundaryPredictionGPU,
										vyboundaryPredictionGPU,
										vzboundaryPredictionGPU);
    }





    //
    //
    //
    //
    //
    //
    //STEP 9: IF EXCESS OF MASS != 0 CALCULATE VELOCITY CORRECTION
    //\Delta v
    if(mass != 0){
      //We finish here what we started in STEP 7.
      
      //Add all terms S*({\Delta p} - me*J*\tilde{v}) and store in vxZ.x
      kernelCorrectionVQuasiNeutrallyBuoyant_2<<<numBlocks,threadsPerBlock>>>(vxZ,
									      vyZ,
									      vzZ,
									      fxboundaryGPU,
									      fyboundaryGPU,
									      fzboundaryGPU);

      //Third apply incompressibility to calculate \Delta \tilde{ v }
      hipfftExecZ2Z(FFT,vxZ,vxZ,HIPFFT_FORWARD);
      hipfftExecZ2Z(FFT,vyZ,vyZ,HIPFFT_FORWARD);
      hipfftExecZ2Z(FFT,vzZ,vzZ,HIPFFT_FORWARD);
      kernelShift<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,pF,-1);
      //projectionDivergenceFree<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,pF);
      kernelUpdateVIncompressible<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,vxZ,vyZ,vzZ,pF);//W 
      kernelShift<<<numBlocks,threadsPerBlock>>>(vxZ,vyZ,vzZ,pF,1);
      hipfftExecZ2Z(FFT,vxZ,vxZ,HIPFFT_BACKWARD);
      hipfftExecZ2Z(FFT,vyZ,vyZ,HIPFFT_BACKWARD);
      hipfftExecZ2Z(FFT,vzZ,vzZ,HIPFFT_BACKWARD);

      //The result \Delta v is store in vxZ.x without normalization
    }


    
    
    
    //
    //
    //
    //
    //
    //
    //STEP 10: UPDATE PARTICLE POSITION
    //q^{n+1} = q^n + 0.5*dt * J^{n+1/2} * (v^n + v^{n+1})
    //Calculate v^{n+0.5}
    //Store it in vxGPU
    if(mass != 0){
      calculateVelocityAtHalfTimeStep<<<numBlocks,threadsPerBlock>>>(vxGPU,
								     vyGPU,
								     vzGPU,
								     vxPredictionGPU,
								     vyPredictionGPU,
								     vzPredictionGPU,
								     vxZ,
								     vyZ,
								     vzZ);
    }
    else{
      calculateVelocityAtHalfTimeStepme0<<<numBlocks,threadsPerBlock>>>(vxGPU,
									vyGPU,
									vzGPU,
									vxPredictionGPU,
									vyPredictionGPU,
									vzPredictionGPU);
    }
    
    //Update particle position q^{n+1} = q^n + dt * J^{n+1/2} v^{n+1/2}
    findNeighborParticlesQuasiNeutrallyBuoyantTEST4_2<<<numBlocksParticles,threadsPerBlockParticles>>>
      (pc, 
       errorKernel,
       rxcellGPU,
       rycellGPU,
       rzcellGPU,
       rxboundaryGPU, //q^{n} and q^{n+1}
       ryboundaryGPU,
       rzboundaryGPU,
       rxboundaryPredictionGPU, //q^{n+1/2}
       ryboundaryPredictionGPU, 
       rzboundaryPredictionGPU,
       vxGPU, // v^{n+1/2}
       vyGPU,
       vzGPU);





    //
    //
    //
    //
    //
    //
    //STEP 11: IF EXCESS OF MASS !=0 CALCULATE LAGGING TERM FOR THE NEXT STEP
    //0.5*nu*dt * J^{n-1/2} * \Delta v
    if(mass != 0){
      //Calculate 0.5*dt*nu*L*\Delta v and store it
      //in vxGPU
      laplacianDeltaV<<<numBlocks,threadsPerBlock>>>(vxZ,
						     vyZ,
						     vzZ,
						     vxGPU,
						     vyGPU,
						     vzGPU);
      
      //Calculate 0.5*dt*nu*J*L*\Delta v and store it
      //in vxboundaryPredictionGPU
      interpolateLaplacianDeltaV<<<numBlocksParticles,threadsPerBlockParticles>>>(rxcellGPU,
										  rycellGPU,
										  rzcellGPU,
										  vxGPU,
										  vyGPU,
										  vzGPU,
										  rxboundaryPredictionGPU,
										  ryboundaryPredictionGPU,
										  rzboundaryPredictionGPU,
										  vxboundaryPredictionGPU,
										  vyboundaryPredictionGPU,
										  vzboundaryPredictionGPU);
    }



    //Load textures with particles position q^{n}
    cutilSafeCall( hipBindTexture(0,texrxboundaryGPU,rxboundaryGPU,(nboundary+np)*sizeof(double)));
    cutilSafeCall( hipBindTexture(0,texryboundaryGPU,ryboundaryGPU,(nboundary+np)*sizeof(double)));
    cutilSafeCall( hipBindTexture(0,texrzboundaryGPU,rzboundaryGPU,(nboundary+np)*sizeof(double)));



    //
    //
    //
    //
    //
    //
    //STEP 12: UPDATE FLUID VELOCITY
    //Update fluid velocity
    if(mass == 0){
      updateFluidme0<<<numBlocks,threadsPerBlock>>>(vxGPU,
						    vyGPU,
						    vzGPU,
						    vxPredictionGPU,
						    vyPredictionGPU,
						    vzPredictionGPU);
    }
    else{
      updateFluidQuasiNeutrallyBuoyantSemiImplicit<<<numBlocks,threadsPerBlock>>>(vxGPU,
										  vyGPU,
										  vzGPU,
										  vxPredictionGPU,
										  vyPredictionGPU,
										  vzPredictionGPU,
										  vxZ,
										  vyZ,
										  vzZ);
    }

    //Load textures with velocity prediction "\tilde{v}^{n+1}"
    cutilSafeCall( hipBindTexture(0,texVxGPU,vxGPU,ncells*sizeof(double)));
    cutilSafeCall( hipBindTexture(0,texVyGPU,vyGPU,ncells*sizeof(double)));
    cutilSafeCall( hipBindTexture(0,texVzGPU,vzGPU,ncells*sizeof(double)));








								    
    step++;
    if(!(step%samplefreq)&&(step>0)){
      cout << "INCOMPRESSIBLE BOUNDARY  ;) " << step << endl;
      if(!gpuToHostIncompressibleBoundaryRK2(numBlocksParticles,threadsPerBlockParticles)) return 0;
      if(!saveFunctionsSchemeIncompressibleBoundary(1,step)) return 0;
    }
  }



  //Free FFT
  hipfftDestroy(FFT);
  freeRandomNumbersGPU();

  

  return 1;
}
