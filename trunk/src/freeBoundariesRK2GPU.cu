// Filename: freeBoundariesRK2GPU.cu
//
// Copyright (c) 2010-2012, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


bool freeBoundariesRK2GPU(){

  hipUnbindTexture(texrxboundaryGPU);    
  hipUnbindTexture(texryboundaryGPU);    
  hipUnbindTexture(texrzboundaryGPU);
  //hipUnbindTexture(texfxboundaryGPU);
  //hipUnbindTexture(texfyboundaryGPU);
  //hipUnbindTexture(texfzboundaryGPU);

  hipFree(rxboundaryGPU);
  hipFree(ryboundaryGPU);
  hipFree(rzboundaryGPU);
  hipFree(rxboundaryPredictionGPU);
  hipFree(ryboundaryPredictionGPU);
  hipFree(rzboundaryPredictionGPU);
  hipFree(vxboundaryGPU);
  hipFree(vyboundaryGPU);
  hipFree(vzboundaryGPU);
  hipFree(vxboundaryPredictionGPU);
  hipFree(vyboundaryPredictionGPU);
  hipFree(vzboundaryPredictionGPU);
  hipFree(fxboundaryGPU);
  hipFree(fyboundaryGPU);
  hipFree(fzboundaryGPU);

  hipUnbindTexture(texCountParticlesInCellX);
  hipUnbindTexture(texCountParticlesInCellY);
  hipUnbindTexture(texCountParticlesInCellZ);
  hipUnbindTexture(texPartInCellX);
  hipUnbindTexture(texPartInCellY);
  hipUnbindTexture(texPartInCellZ);
  hipUnbindTexture(texCountParticlesInCellNonBonded);
  hipUnbindTexture(texPartInCellNonBonded);



  if(setparticles){
    hipUnbindTexture(texCountParticlesInCellNonBonded);
    hipFree(countPartInCellNonBonded);

    hipUnbindTexture(texPartInCellNonBonded);
    hipFree(partInCellNonBonded);

    hipUnbindTexture(texneighbor0GPU);
    hipUnbindTexture(texneighbor1GPU);
    hipUnbindTexture(texneighbor2GPU);
    hipUnbindTexture(texneighbor3GPU);
    hipUnbindTexture(texneighbor4GPU);
    hipUnbindTexture(texneighbor5GPU);
    hipUnbindTexture(texneighborpxpyGPU);
    hipUnbindTexture(texneighborpxmyGPU);
    hipUnbindTexture(texneighborpxpzGPU);
    hipUnbindTexture(texneighborpxmzGPU);
    hipUnbindTexture(texneighbormxpyGPU);
    hipUnbindTexture(texneighbormxmyGPU);
    hipUnbindTexture(texneighbormxpzGPU);
    hipUnbindTexture(texneighbormxmzGPU);
    hipUnbindTexture(texneighborpypzGPU);
    hipUnbindTexture(texneighborpymzGPU);
    hipUnbindTexture(texneighbormypzGPU);
    hipUnbindTexture(texneighbormymzGPU);
    hipUnbindTexture(texneighborpxpypzGPU);
    hipUnbindTexture(texneighborpxpymzGPU);
    hipUnbindTexture(texneighborpxmypzGPU);
    hipUnbindTexture(texneighborpxmymzGPU);
    hipUnbindTexture(texneighbormxpypzGPU);
    hipUnbindTexture(texneighbormxpymzGPU);
    hipUnbindTexture(texneighbormxmypzGPU);
    hipUnbindTexture(texneighbormxmymzGPU);
    hipFree(neighbor0GPU);
    hipFree(neighbor1GPU);
    hipFree(neighbor2GPU);
    hipFree(neighbor3GPU);
    hipFree(neighbor4GPU);
    hipFree(neighbor5GPU);
    hipFree(neighborpxpyGPU);
    hipFree(neighborpxmyGPU);
    hipFree(neighborpxpzGPU);
    hipFree(neighborpxmzGPU);
    hipFree(neighbormxpyGPU);
    hipFree(neighbormxmyGPU);
    hipFree(neighbormxpzGPU);
    hipFree(neighbormxmzGPU);
    hipFree(neighborpypzGPU);
    hipFree(neighborpymzGPU);
    hipFree(neighbormypzGPU);
    hipFree(neighbormymzGPU);
    hipFree(neighborpxpypzGPU);
    hipFree(neighborpxpymzGPU);
    hipFree(neighborpxmypzGPU);
    hipFree(neighborpxmymzGPU);
    hipFree(neighbormxpypzGPU);
    hipFree(neighbormxpymzGPU);
    hipFree(neighbormxmypzGPU);
    hipFree(neighbormxmymzGPU);
  }

  freeErrorArray();
  hipFree(pc);
  freeDelta();

  if(setparticles){
    hipUnbindTexture(texforceNonBonded1);
    hipFreeArray(forceNonBonded1);
  }

  //No-slip Test
  //hipFree(saveForceX);
  //hipFree(saveForceY);
  //hipFree(saveForceZ);

  cout << "FREE BOUNDARIES GPU :           DONE" << endl; 

  return 1;
}
