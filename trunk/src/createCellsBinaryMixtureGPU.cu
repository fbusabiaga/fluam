#include "hip/hip_runtime.h"
// Filename: createCellsBinaryMixtureGPU.cu
//
// Copyright (c) 2010-2012, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


//#define GPUVARIABLES 1


bool createCellsBinaryMixtureGPU(){
  hipMemcpyToSymbol(HIP_SYMBOL(mxGPU),&mx,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(myGPU),&my,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(mzGPU),&mz,sizeof(int));
  int aux;
  aux = mx+2;
  hipMemcpyToSymbol(HIP_SYMBOL(mxtGPU),&aux,sizeof(int));
  aux = my+2;
  hipMemcpyToSymbol(HIP_SYMBOL(mytGPU),&aux,sizeof(int));
  aux = mz+2;
  hipMemcpyToSymbol(HIP_SYMBOL(mztGPU),&aux,sizeof(int));
  aux = (mx+2) * (my+2);
  hipMemcpyToSymbol(HIP_SYMBOL(mxmytGPU),&aux,sizeof(int));


  hipMemcpyToSymbol(HIP_SYMBOL(ncellsGPU),&ncells,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(ncellstGPU),&ncellst,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(lxGPU),&lx,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(lyGPU),&ly,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(lzGPU),&lz,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(dtGPU),&dt,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(volumeGPU),&cVolume,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(shearviscosityGPU),&shearviscosity,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(bulkviscosityGPU),&bulkviscosity,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(temperatureGPU),&temperature,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(pressurea0GPU),&pressurea0,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(pressurea1GPU),&pressurea1,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(pressurea2GPU),&pressurea2,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(thermostatGPU),&thermostat,sizeof(bool));

  hipMalloc((void**)&densityGPU,ncellst*sizeof(double));
  hipMalloc((void**)&vxGPU,ncellst*sizeof(double));
  hipMalloc((void**)&vyGPU,ncellst*sizeof(double));
  hipMalloc((void**)&vzGPU,ncellst*sizeof(double));
  hipMalloc((void**)&densityPredictionGPU,ncellst*sizeof(double));
  hipMalloc((void**)&vxPredictionGPU,ncellst*sizeof(double));
  hipMalloc((void**)&vyPredictionGPU,ncellst*sizeof(double));
  hipMalloc((void**)&vzPredictionGPU,ncellst*sizeof(double));

  hipMalloc((void**)&cGPU,ncellst*sizeof(double));
  hipMalloc((void**)&cPredictionGPU,ncellst*sizeof(double));
  hipMalloc((void**)&dcGPU,ncellst*sizeof(double));

  hipMemcpyToSymbol(HIP_SYMBOL(diffusionGPU),&diffusion,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(massSpecies0GPU),&massSpecies0,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(massSpecies1GPU),&massSpecies1,sizeof(double));


  hipMalloc((void**)&dmGPU,ncellst*sizeof(double));
  hipMalloc((void**)&dpxGPU,ncellst*sizeof(double));
  hipMalloc((void**)&dpyGPU,ncellst*sizeof(double));
  hipMalloc((void**)&dpzGPU,ncellst*sizeof(double));

  hipMalloc((void**)&rxcellGPU,ncellst*sizeof(double));
  hipMalloc((void**)&rycellGPU,ncellst*sizeof(double));
  hipMalloc((void**)&rzcellGPU,ncellst*sizeof(double));

  double fact1 = sqrt((4.*temperature*shearviscosity)/(dt*cVolume));
  double fact2 = sqrt((2.*temperature*bulkviscosity)/(3.*dt*cVolume));
  double fact3 = bulkviscosity - 2. * shearviscosity/3.;
  double fact4 = sqrt((2.*temperature*shearviscosity)/(dt*cVolume));
  double fact5 = sqrt(1./(dt*cVolume));

  hipMemcpyToSymbol(HIP_SYMBOL(fact1GPU),&fact1,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(fact2GPU),&fact2,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(fact3GPU),&fact3,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(fact4GPU),&fact4,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(fact5GPU),&fact5,sizeof(double));


  fact1 = lx/double(mx);
  fact2 = ly/double(my);
  fact3 = lz/double(mz);
  hipMemcpyToSymbol(HIP_SYMBOL(dxGPU),&fact1,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(dyGPU),&fact2,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(dzGPU),&fact3,sizeof(double));

  fact1 = double(mx)/lx;
  fact2 = double(my)/ly;
  fact3 = double(mz)/lz;
  hipMemcpyToSymbol(HIP_SYMBOL(invdxGPU),&fact1,sizeof(double));  
  hipMemcpyToSymbol(HIP_SYMBOL(invdyGPU),&fact2,sizeof(double));  
  hipMemcpyToSymbol(HIP_SYMBOL(invdzGPU),&fact3,sizeof(double));  
  fact1 = 1./dt;
  hipMemcpyToSymbol(HIP_SYMBOL(invdtGPU),&fact1,sizeof(double));
  fact1 = 1./lx;
  fact2 = 1./ly;
  fact3 = 1./lz;
  hipMemcpyToSymbol(HIP_SYMBOL(invlxGPU),&fact1,sizeof(double));  
  hipMemcpyToSymbol(HIP_SYMBOL(invlyGPU),&fact2,sizeof(double));  
  hipMemcpyToSymbol(HIP_SYMBOL(invlzGPU),&fact3,sizeof(double));

  //
  hipMalloc((void**)&ghostIndexGPU,ncells*sizeof(int));
  hipMalloc((void**)&realIndexGPU,ncellst*sizeof(int));
  hipMalloc((void**)&ghostToPIGPU,(ncellst-ncells)*sizeof(int));
  hipMalloc((void**)&ghostToGhostGPU,(ncellst-ncells)*sizeof(int));

  bool auxbool = 0;
  hipMemcpyToSymbol(HIP_SYMBOL(setparticlesGPU),&auxbool,sizeof(bool));
  hipMemcpyToSymbol(HIP_SYMBOL(setboundaryGPU),&auxbool,sizeof(bool));


  long long auxulonglong = 0;
  hipMalloc((void**)&stepGPU,sizeof(long long));
  hipMemcpy(stepGPU,&auxulonglong,sizeof(long long),hipMemcpyHostToDevice);




  

  
  cout << "CREATE CELLS GPU :              DONE" << endl;

  return 1;
}
