#include "hip/hip_runtime.h"
// Filename: createCellsBinaryMixtureGPU.cu
//
// Copyright (c) 2010-2013, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


//#define GPUVARIABLES 1


bool createCellsBinaryMixtureGPU(){
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mxGPU),&mx,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(myGPU),&my,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mzGPU),&mz,sizeof(int)));
  int aux;
  aux = mx+2;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mxtGPU),&aux,sizeof(int)));
  aux = my+2;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mytGPU),&aux,sizeof(int)));
  aux = mz+2;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mztGPU),&aux,sizeof(int)));
  aux = (mx+2) * (my+2);
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mxmytGPU),&aux,sizeof(int)));


  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ncellsGPU),&ncells,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ncellstGPU),&ncellst,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(lxGPU),&lx,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(lyGPU),&ly,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(lzGPU),&lz,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dtGPU),&dt,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(volumeGPU),&cVolume,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(shearviscosityGPU),&shearviscosity,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(bulkviscosityGPU),&bulkviscosity,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(temperatureGPU),&temperature,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(pressurea0GPU),&pressurea0,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(pressurea1GPU),&pressurea1,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(pressurea2GPU),&pressurea2,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(thermostatGPU),&thermostat,sizeof(bool)));

  cutilSafeCall(hipMalloc((void**)&densityGPU,ncellst*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vxGPU,ncellst*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vyGPU,ncellst*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vzGPU,ncellst*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&densityPredictionGPU,ncellst*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vxPredictionGPU,ncellst*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vyPredictionGPU,ncellst*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vzPredictionGPU,ncellst*sizeof(double)));

  cutilSafeCall(hipMalloc((void**)&cGPU,ncellst*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&cPredictionGPU,ncellst*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&dcGPU,ncellst*sizeof(double)));

  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(diffusionGPU),&diffusion,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(massSpecies0GPU),&massSpecies0,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(massSpecies1GPU),&massSpecies1,sizeof(double)));


  cutilSafeCall(hipMalloc((void**)&dmGPU,ncellst*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&dpxGPU,ncellst*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&dpyGPU,ncellst*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&dpzGPU,ncellst*sizeof(double)));

  cutilSafeCall(hipMalloc((void**)&rxcellGPU,ncellst*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&rycellGPU,ncellst*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&rzcellGPU,ncellst*sizeof(double)));

  double fact1 = sqrt((4.*temperature*shearviscosity)/(dt*cVolume));
  double fact2 = sqrt((2.*temperature*bulkviscosity)/(3.*dt*cVolume));
  double fact3 = bulkviscosity - 2. * shearviscosity/3.;
  double fact4 = sqrt((2.*temperature*shearviscosity)/(dt*cVolume));
  double fact5 = sqrt(1./(dt*cVolume));

  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(fact1GPU),&fact1,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(fact2GPU),&fact2,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(fact3GPU),&fact3,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(fact4GPU),&fact4,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(fact5GPU),&fact5,sizeof(double)));


  fact1 = lx/double(mx);
  fact2 = ly/double(my);
  fact3 = lz/double(mz);
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dxGPU),&fact1,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dyGPU),&fact2,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dzGPU),&fact3,sizeof(double)));

  fact1 = double(mx)/lx;
  fact2 = double(my)/ly;
  fact3 = double(mz)/lz;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invdxGPU),&fact1,sizeof(double)));  
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invdyGPU),&fact2,sizeof(double)));  
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invdzGPU),&fact3,sizeof(double)));  
  fact1 = 1./dt;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invdtGPU),&fact1,sizeof(double)));
  fact1 = 1./lx;
  fact2 = 1./ly;
  fact3 = 1./lz;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invlxGPU),&fact1,sizeof(double)));  
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invlyGPU),&fact2,sizeof(double)));  
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invlzGPU),&fact3,sizeof(double)));

  //
  cutilSafeCall(hipMalloc((void**)&ghostIndexGPU,ncells*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&realIndexGPU,ncellst*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&ghostToPIGPU,(ncellst-ncells)*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&ghostToGhostGPU,(ncellst-ncells)*sizeof(int)));

  bool auxbool = 0;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(setparticlesGPU),&auxbool,sizeof(bool)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(setboundaryGPU),&auxbool,sizeof(bool)));


  long long auxulonglong = 0;
  cutilSafeCall(hipMalloc((void**)&stepGPU,sizeof(long long)));
  cutilSafeCall(hipMemcpy(stepGPU,&auxulonglong,sizeof(long long),hipMemcpyHostToDevice));




  

  
  cout << "CREATE CELLS GPU :              DONE" << endl;

  return 1;
}
