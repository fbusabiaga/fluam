// Filename: createBoundariesRK2GPU.cu
//
// Copyright (c) 2010-2013, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


bool createBoundariesRK2GPU(){

  if(setparticles==0) np=0;
  
  hipMemcpyToSymbol(HIP_SYMBOL(nboundaryGPU),&nboundary,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(npGPU),&np,sizeof(int));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(maxNumberPartInCellGPU),&maxNumberPartInCell,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(maxNumberPartInCellNonBondedGPU),&maxNumberPartInCellNonBonded,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(particlesWallGPU),&particlesWall,sizeof(bool)));
  hipMemcpyToSymbol(HIP_SYMBOL(computeNonBondedForcesGPU),&computeNonBondedForces,sizeof(bool));

  //Create boundaries and particles variables
  cutilSafeCall(hipMalloc((void**)&rxboundaryGPU,(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&ryboundaryGPU,(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&rzboundaryGPU,(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&rxboundaryPredictionGPU,(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&ryboundaryPredictionGPU,(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&rzboundaryPredictionGPU,(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vxboundaryGPU,(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vyboundaryGPU,(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vzboundaryGPU,(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vxboundaryPredictionGPU,(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vyboundaryPredictionGPU,(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vzboundaryPredictionGPU,(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&fxboundaryGPU,27*(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&fyboundaryGPU,27*(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&fzboundaryGPU,27*(nboundary+np)*sizeof(double)));

  //Initialize boundaries variables
  cutilSafeCall(hipMemcpy(rxboundaryGPU,rxboundary,nboundary*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(ryboundaryGPU,ryboundary,nboundary*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(rzboundaryGPU,rzboundary,nboundary*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(vxboundaryGPU,vxboundary,nboundary*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(vyboundaryGPU,vyboundary,nboundary*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(vzboundaryGPU,vzboundary,nboundary*sizeof(double),hipMemcpyHostToDevice));

  //Initialize particles variables
  cutilSafeCall(hipMemcpy(&rxboundaryGPU[nboundary],rxParticle,np*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(&ryboundaryGPU[nboundary],ryParticle,np*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(&rzboundaryGPU[nboundary],rzParticle,np*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(&vxboundaryGPU[nboundary],vxParticle,np*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(&vyboundaryGPU[nboundary],vyParticle,np*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(&vzboundaryGPU[nboundary],vzParticle,np*sizeof(double),hipMemcpyHostToDevice));
  
  //Copy some constants
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(volumeboundaryconstGPU),&volumeboundaryconst,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(massParticleGPU),&mass,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(volumeParticleGPU),&volumeParticle,sizeof(double)));
 
              
  cutilSafeCall(hipMalloc((void**)&countparticlesincellX,ncells*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&countparticlesincellY,ncells*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&countparticlesincellZ,ncells*sizeof(int)));
  int aux[ncells];
  for(int i=0;i<ncells;i++) aux[i] = 0;
  hipMemcpy(countparticlesincellX,aux,ncells*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(countparticlesincellY,aux,ncells*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(countparticlesincellZ,aux,ncells*sizeof(int),hipMemcpyHostToDevice);

  hipMalloc((void**)&partincellX,maxNumberPartInCell*ncells*sizeof(int));
  hipMalloc((void**)&partincellY,maxNumberPartInCell*ncells*sizeof(int));
  hipMalloc((void**)&partincellZ,maxNumberPartInCell*ncells*sizeof(int));
  

  //texrxboundaryGPU
  texrxboundaryGPU.normalized = false;
  texrxboundaryGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texrxboundaryGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
  cutilSafeCall( hipBindTexture(0,texrxboundaryGPU,rxboundaryGPU,(nboundary+np)*sizeof(double)));
  //texryboundaryGPU
  texryboundaryGPU.normalized = false;
  texryboundaryGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texryboundaryGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
  cutilSafeCall( hipBindTexture(0,texryboundaryGPU,ryboundaryGPU,(nboundary+np)*sizeof(double)));
  //texrzboundaryGPU
  texrzboundaryGPU.normalized = false;
  texrzboundaryGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texrzboundaryGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
  cutilSafeCall( hipBindTexture(0,texrzboundaryGPU,rzboundaryGPU,(nboundary+np)*sizeof(double)));
  //texCountParticlesInCellX;
  texCountParticlesInCellX.normalized = false;
  texCountParticlesInCellX.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texCountParticlesInCellX.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
  cutilSafeCall( hipBindTexture(0,texCountParticlesInCellX,countparticlesincellX,ncells*sizeof(int)));
  //texCountParticlesInCellY;
  texCountParticlesInCellY.normalized = false;
  texCountParticlesInCellY.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texCountParticlesInCellY.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
  cutilSafeCall( hipBindTexture(0,texCountParticlesInCellY,countparticlesincellY,ncells*sizeof(int)));
  //texCountParticlesInCellZ;
  texCountParticlesInCellZ.normalized = false;
  texCountParticlesInCellZ.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texCountParticlesInCellZ.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
  cutilSafeCall( hipBindTexture(0,texCountParticlesInCellZ,countparticlesincellZ,ncells*sizeof(int)));
  //texPartInCellX;
  texPartInCellX.normalized = false;
  texPartInCellX.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texPartInCellX.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
  cutilSafeCall( hipBindTexture(0,texPartInCellX,partincellX,maxNumberPartInCell*ncells*sizeof(int)));
  //texPartInCellY;
  texPartInCellY.normalized = false;
  texPartInCellY.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texPartInCellY.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
  cutilSafeCall( hipBindTexture(0,texPartInCellY,partincellY,maxNumberPartInCell*ncells*sizeof(int)));
  //texPartInCellZ;
  texPartInCellZ.normalized = false;
  texPartInCellZ.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texPartInCellZ.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
  cutilSafeCall( hipBindTexture(0,texPartInCellZ,partincellZ,maxNumberPartInCell*ncells*sizeof(int)));

  if(setparticles){
    int mxPart = int(lx/cutoff);
    if(mxPart < 3) mxPart = 3;
    int myPart = int(ly/cutoff);
    if(myPart < 3) myPart = 3;
    int mzPart = int(lz/cutoff);
    if(mzPart < 3) mzPart = 3;
    numNeighbors = mxPart * myPart * mzPart;

    cutilSafeCall(hipMalloc((void**)&countPartInCellNonBonded,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&partInCellNonBonded,
			     maxNumberPartInCellNonBonded*numNeighbors*sizeof(int)));

    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mxNeighborsGPU),&mxPart,sizeof(int)));
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(myNeighborsGPU),&myPart,sizeof(int)));
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mzNeighborsGPU),&mzPart,sizeof(int)));
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mNeighborsGPU),&numNeighbors,sizeof(int)));
    
    cutilSafeCall(hipMalloc((void**)&neighbor0GPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbor1GPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbor2GPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbor3GPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbor4GPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbor5GPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighborpxpyGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighborpxmyGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighborpxpzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighborpxmzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbormxpyGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbormxmyGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbormxpzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbormxmzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighborpypzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighborpymzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbormypzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbormymzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighborpxpypzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighborpxpymzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighborpxmypzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighborpxmymzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbormxpypzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbormxpymzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbormxmypzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbormxmymzGPU,numNeighbors*sizeof(int)));
    
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(cutoffGPU),&cutoff,sizeof(double)));
    double invcutoff = 1./cutoff;
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invcutoffGPU),&invcutoff,sizeof(double)));
    invcutoff = 1./(cutoff * cutoff);
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invcutoff2GPU),&invcutoff,sizeof(double)));
    
    
    //texCountParticlesInCellNonBonded;
    texCountParticlesInCellNonBonded.normalized = false;
    texCountParticlesInCellNonBonded.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texCountParticlesInCellNonBonded.filterMode = 
      hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texCountParticlesInCellNonBonded,
				   countPartInCellNonBonded,numNeighbors*sizeof(int)));
    //texPartInCellNonBonded;
    texPartInCellNonBonded.normalized = false;
    texPartInCellNonBonded.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texPartInCellNonBonded.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texPartInCellNonBonded,partInCellNonBonded,
				   maxNumberPartInCellNonBonded*numNeighbors*sizeof(int)));
    //texneighbor0GPU
    texneighbor0GPU.normalized = false;
    texneighbor0GPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbor0GPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighbor0GPU,neighbor0GPU,numNeighbors*sizeof(int)));
    //texneighbor1GPU
    texneighbor1GPU.normalized = false;
    texneighbor1GPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbor1GPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighbor1GPU,neighbor1GPU,numNeighbors*sizeof(int)));
    //texneighbor2GPU
    texneighbor2GPU.normalized = false;
    texneighbor2GPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbor2GPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighbor2GPU,neighbor2GPU,numNeighbors*sizeof(int)));
    //texneighbor3GPU
    texneighbor3GPU.normalized = false;
    texneighbor3GPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbor3GPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighbor3GPU,neighbor3GPU,numNeighbors*sizeof(int)));
    //texneighbor4GPU
    texneighbor4GPU.normalized = false;
    texneighbor4GPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbor4GPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighbor4GPU,neighbor4GPU,numNeighbors*sizeof(int)));
    //texneighbor5GPU
    texneighbor5GPU.normalized = false;
    texneighbor5GPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbor5GPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighbor5GPU,neighbor5GPU,numNeighbors*sizeof(int)));
    //texneighborpxpyGPU
    texneighborpxpyGPU.normalized = false;
    texneighborpxpyGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighborpxpyGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighborpxpyGPU,neighborpxpyGPU,numNeighbors*sizeof(int)));
    //texneighborpxmyGPU
    texneighborpxmyGPU.normalized = false;
    texneighborpxmyGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighborpxmyGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighborpxmyGPU,neighborpxmyGPU,numNeighbors*sizeof(int)));
    //texneighborpxpzGPU
    texneighborpxpzGPU.normalized = false;
    texneighborpxpzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighborpxpzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighborpxpzGPU,neighborpxpzGPU,numNeighbors*sizeof(int)));
    //texneighborpxmzGPU
    texneighborpxmzGPU.normalized = false;
    texneighborpxmzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighborpxmzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighborpxmzGPU,neighborpxmzGPU,numNeighbors*sizeof(int)));
    //texneighbormxpyGPU
    texneighbormxpyGPU.normalized = false;
    texneighbormxpyGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbormxpyGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighbormxpyGPU,neighbormxpyGPU,numNeighbors*sizeof(int)));
    //texneighbormxmyGPU
    texneighbormxmyGPU.normalized = false;
    texneighbormxmyGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbormxmyGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighbormxmyGPU,neighbormxmyGPU,numNeighbors*sizeof(int)));
    //texneighbormxpzGPU
    texneighbormxpzGPU.normalized = false;
    texneighbormxpzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbormxpzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighbormxpzGPU,neighbormxpzGPU,numNeighbors*sizeof(int)));
    //texneighbormxmzGPU
    texneighbormxmzGPU.normalized = false;
    texneighbormxmzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbormxmzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighbormxmzGPU,neighbormxmzGPU,numNeighbors*sizeof(int)));
    //texneighborpypzGPU
    texneighborpypzGPU.normalized = false;
    texneighborpypzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighborpypzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighborpypzGPU,neighborpypzGPU,numNeighbors*sizeof(int)));
    //texneighborpymzGPU
    texneighborpymzGPU.normalized = false;
    texneighborpymzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighborpymzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighborpymzGPU,neighborpymzGPU,numNeighbors*sizeof(int)));
    //texneighbormypzGPU
    texneighbormypzGPU.normalized = false;
    texneighbormypzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbormypzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighbormypzGPU,neighbormypzGPU,numNeighbors*sizeof(int)));
    //texneighbormymzGPU
    texneighbormymzGPU.normalized = false;
    texneighbormymzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbormymzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighbormymzGPU,neighbormymzGPU,numNeighbors*sizeof(int)));
    //texneighborpxpypzGPU
    texneighborpxpypzGPU.normalized = false;
    texneighborpxpypzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighborpxpypzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighborpxpypzGPU,neighborpxpypzGPU,numNeighbors*sizeof(int)));
    //texneighborpxpymzGPU
    texneighborpxpymzGPU.normalized = false;
    texneighborpxpymzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighborpxpymzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighborpxpymzGPU,neighborpxpymzGPU,numNeighbors*sizeof(int)));
    //texneighborpxmypzGPU
    texneighborpxmypzGPU.normalized = false;
    texneighborpxmypzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighborpxmypzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighborpxmypzGPU,neighborpxmypzGPU,numNeighbors*sizeof(int)));
    //texneighborpxmymzGPU
    texneighborpxmymzGPU.normalized = false;
    texneighborpxmymzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighborpxmymzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighborpxmymzGPU,neighborpxmymzGPU,numNeighbors*sizeof(int)));
    //texneighbormxpypzGPU
    texneighbormxpypzGPU.normalized = false;
    texneighbormxpypzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbormxpypzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighbormxpypzGPU,neighbormxpypzGPU,numNeighbors*sizeof(int)));
    //texneighbormymzGPU
    texneighbormxpymzGPU.normalized = false;
    texneighbormxpymzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbormxpymzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighbormxpymzGPU,neighbormxpymzGPU,numNeighbors*sizeof(int)));
    //texneighbormxmypzGPU
    texneighbormxmypzGPU.normalized = false;
    texneighbormxmypzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbormxmypzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighbormxmypzGPU,neighbormxmypzGPU,numNeighbors*sizeof(int)));
    //texneighbormxmymzGPU
    texneighbormxmymzGPU.normalized = false;
    texneighbormxmymzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbormxmymzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    cutilSafeCall( hipBindTexture(0,texneighbormxmymzGPU,neighbormxmymzGPU,numNeighbors*sizeof(int)));
    
    //init_force_non_bonded();

    int block = (numNeighbors-1)/128 + 1;

    initializeNeighbors<<<block,128>>>(neighbor1GPU,neighbor2GPU,neighbor3GPU,neighbor4GPU,
				       neighborpxpyGPU,neighborpxmyGPU,neighborpxpzGPU,neighborpxmzGPU,
				       neighbormxpyGPU,neighbormxmyGPU,neighbormxpzGPU,neighbormxmzGPU,
				       neighborpypzGPU,neighborpymzGPU,neighbormypzGPU,neighbormymzGPU,
				       neighborpxpypzGPU,neighborpxpymzGPU,
				       neighborpxmypzGPU,neighborpxmymzGPU,
				       neighbormxpypzGPU,neighbormxpymzGPU,
				       neighbormxmypzGPU,neighbormxmymzGPU);
    initializeNeighbors2<<<block,128>>>(neighbor0GPU,neighbor1GPU,neighbor2GPU,
					neighbor3GPU,neighbor4GPU,neighbor5GPU);

    initForcesNonBonded();

  }


  initDelta();
  allocateErrorArray();

  cutilSafeCall(hipMalloc((void**)&pc,sizeof(particlesincell)));
  
  initParticlesInCell<<<1,1>>>(partincellX,partincellY,partincellZ,
			       countparticlesincellX,countparticlesincellY,countparticlesincellZ,
			       countPartInCellNonBonded,partInCellNonBonded,pc);

  //No-slip Test
  //cutilSafeCall(hipMalloc((void**)&saveForceX,np*sizeof(double)));
  //cutilSafeCall(hipMalloc((void**)&saveForceY,np*sizeof(double)));
  //cutilSafeCall(hipMalloc((void**)&saveForceZ,np*sizeof(double)));
  

  double auxDouble[27*(nboundary+np)];
  for(int i=0;i<27*(nboundary+np);i++) auxDouble[i] = 0;
  cutilSafeCall(hipMemcpy(fxboundaryGPU,auxDouble,27*(nboundary+np)*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(fyboundaryGPU,auxDouble,27*(nboundary+np)*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(fzboundaryGPU,auxDouble,27*(nboundary+np)*sizeof(double),hipMemcpyHostToDevice));


  //Copy constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(bondedForcesGPU),&bondedForces,sizeof(bool));

  
  cout << "CREATE BOUNDARIES GPU :         DONE" << endl; 

  return 1;
}
