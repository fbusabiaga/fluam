// Filename: createBoundariesRK2GPU.cu
//
// Copyright (c) 2010-2012, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


bool createBoundariesRK2GPU(){

  if(setparticles==0) np=0;
  
  hipMemcpyToSymbol(HIP_SYMBOL(nboundaryGPU),&nboundary,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(npGPU),&np,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(maxNumberPartInCellGPU),&maxNumberPartInCell,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(maxNumberPartInCellNonBondedGPU),&maxNumberPartInCellNonBonded,sizeof(int));

  //Create boundaries and particles variables
  hipMalloc((void**)&rxboundaryGPU,(nboundary+np)*sizeof(double));
  hipMalloc((void**)&ryboundaryGPU,(nboundary+np)*sizeof(double));
  hipMalloc((void**)&rzboundaryGPU,(nboundary+np)*sizeof(double));
  hipMalloc((void**)&rxboundaryPredictionGPU,(nboundary+np)*sizeof(double));
  hipMalloc((void**)&ryboundaryPredictionGPU,(nboundary+np)*sizeof(double));
  hipMalloc((void**)&rzboundaryPredictionGPU,(nboundary+np)*sizeof(double));
  hipMalloc((void**)&vxboundaryGPU,(nboundary+np)*sizeof(double));
  hipMalloc((void**)&vyboundaryGPU,(nboundary+np)*sizeof(double));
  hipMalloc((void**)&vzboundaryGPU,(nboundary+np)*sizeof(double));
  hipMalloc((void**)&vxboundaryPredictionGPU,(nboundary+np)*sizeof(double));
  hipMalloc((void**)&vyboundaryPredictionGPU,(nboundary+np)*sizeof(double));
  hipMalloc((void**)&vzboundaryPredictionGPU,(nboundary+np)*sizeof(double));
  hipMalloc((void**)&fxboundaryGPU,27*(nboundary+np)*sizeof(double));
  hipMalloc((void**)&fyboundaryGPU,27*(nboundary+np)*sizeof(double));
  hipMalloc((void**)&fzboundaryGPU,27*(nboundary+np)*sizeof(double));

  //Initialize boundaries variables
  hipMemcpy(rxboundaryGPU,rxboundary,nboundary*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(ryboundaryGPU,ryboundary,nboundary*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(rzboundaryGPU,rzboundary,nboundary*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(vxboundaryGPU,vxboundary,nboundary*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(vyboundaryGPU,vyboundary,nboundary*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(vzboundaryGPU,vzboundary,nboundary*sizeof(double),hipMemcpyHostToDevice);

  //Initialize particles variables
  hipMemcpy(&rxboundaryGPU[nboundary],rxParticle,np*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(&ryboundaryGPU[nboundary],ryParticle,np*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(&rzboundaryGPU[nboundary],rzParticle,np*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(&vxboundaryGPU[nboundary],vxParticle,np*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(&vyboundaryGPU[nboundary],vyParticle,np*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(&vzboundaryGPU[nboundary],vzParticle,np*sizeof(double),hipMemcpyHostToDevice);
  
  //Copy some constants
  hipMemcpyToSymbol(HIP_SYMBOL(volumeboundaryconstGPU),&volumeboundaryconst,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(massParticleGPU),&mass,sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(volumeParticleGPU),&volumeParticle,sizeof(double));
  
  
  hipMalloc((void**)&countparticlesincellX,ncells*sizeof(int));
  hipMalloc((void**)&countparticlesincellY,ncells*sizeof(int));
  hipMalloc((void**)&countparticlesincellZ,ncells*sizeof(int));
  int aux[ncells];
  for(int i=0;i<ncells;i++) aux[i] = 0;
  hipMemcpy(countparticlesincellX,aux,ncells*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(countparticlesincellY,aux,ncells*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(countparticlesincellZ,aux,ncells*sizeof(int),hipMemcpyHostToDevice);

  hipMalloc((void**)&partincellX,maxNumberPartInCell*ncells*sizeof(int));
  hipMalloc((void**)&partincellY,maxNumberPartInCell*ncells*sizeof(int));
  hipMalloc((void**)&partincellZ,maxNumberPartInCell*ncells*sizeof(int));
  

  //texrxboundaryGPU
  texrxboundaryGPU.normalized = false;
  texrxboundaryGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texrxboundaryGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
  hipBindTexture(0,texrxboundaryGPU,rxboundaryGPU,(nboundary+np)*sizeof(double));
  //texryboundaryGPU
  texryboundaryGPU.normalized = false;
  texryboundaryGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texryboundaryGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
  hipBindTexture(0,texryboundaryGPU,ryboundaryGPU,(nboundary+np)*sizeof(double));
  //texrzboundaryGPU
  texrzboundaryGPU.normalized = false;
  texrzboundaryGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texrzboundaryGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
  hipBindTexture(0,texrzboundaryGPU,rzboundaryGPU,(nboundary+np)*sizeof(double));
  //texCountParticlesInCellX;
  texCountParticlesInCellX.normalized = false;
  texCountParticlesInCellX.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texCountParticlesInCellX.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
  hipBindTexture(0,texCountParticlesInCellX,countparticlesincellX,ncells*sizeof(int));
  //texCountParticlesInCellY;
  texCountParticlesInCellY.normalized = false;
  texCountParticlesInCellY.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texCountParticlesInCellY.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
  hipBindTexture(0,texCountParticlesInCellY,countparticlesincellY,ncells*sizeof(int));
  //texCountParticlesInCellZ;
  texCountParticlesInCellZ.normalized = false;
  texCountParticlesInCellZ.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texCountParticlesInCellZ.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
  hipBindTexture(0,texCountParticlesInCellZ,countparticlesincellZ,ncells*sizeof(int));
  //texPartInCellX;
  texPartInCellX.normalized = false;
  texPartInCellX.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texPartInCellX.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
  hipBindTexture(0,texPartInCellX,partincellX,maxNumberPartInCell*ncells*sizeof(int));
  //texPartInCellY;
  texPartInCellY.normalized = false;
  texPartInCellY.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texPartInCellY.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
  hipBindTexture(0,texPartInCellY,partincellY,maxNumberPartInCell*ncells*sizeof(int));
  //texPartInCellZ;
  texPartInCellZ.normalized = false;
  texPartInCellZ.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texPartInCellZ.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
  hipBindTexture(0,texPartInCellZ,partincellZ,maxNumberPartInCell*ncells*sizeof(int));

  if(setparticles){
    int mxPart = int(lx/cutoff);
    if(mxPart < 3) mxPart = 3;
    int myPart = int(ly/cutoff);
    if(myPart < 3) myPart = 3;
    int mzPart = int(lz/cutoff);
    if(mzPart < 3) mzPart = 3;
    numNeighbors = mxPart * myPart * mzPart;

    hipMalloc((void**)&countPartInCellNonBonded,numNeighbors*sizeof(int));
    hipMalloc((void**)&partInCellNonBonded,
	       maxNumberPartInCellNonBonded*numNeighbors*sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(mxNeighborsGPU),&mxPart,sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(myNeighborsGPU),&myPart,sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(mzNeighborsGPU),&mzPart,sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(mNeighborsGPU),&numNeighbors,sizeof(int));
    
    hipMalloc((void**)&neighbor0GPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighbor1GPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighbor2GPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighbor3GPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighbor4GPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighbor5GPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighborpxpyGPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighborpxmyGPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighborpxpzGPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighborpxmzGPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighbormxpyGPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighbormxmyGPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighbormxpzGPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighbormxmzGPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighborpypzGPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighborpymzGPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighbormypzGPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighbormymzGPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighborpxpypzGPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighborpxpymzGPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighborpxmypzGPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighborpxmymzGPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighbormxpypzGPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighbormxpymzGPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighbormxmypzGPU,numNeighbors*sizeof(int));
    hipMalloc((void**)&neighbormxmymzGPU,numNeighbors*sizeof(int));
    
    hipMemcpyToSymbol(HIP_SYMBOL(cutoffGPU),&cutoff,sizeof(double));
    double invcutoff = 1./cutoff;
    hipMemcpyToSymbol(HIP_SYMBOL(invcutoffGPU),&invcutoff,sizeof(double));
    invcutoff = 1./(cutoff * cutoff);
    hipMemcpyToSymbol(HIP_SYMBOL(invcutoff2GPU),&invcutoff,sizeof(double));
    
    
    //texCountParticlesInCellNonBonded;
    texCountParticlesInCellNonBonded.normalized = false;
    texCountParticlesInCellNonBonded.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texCountParticlesInCellNonBonded.filterMode = 
      hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texCountParticlesInCellNonBonded,
		    countPartInCellNonBonded,numNeighbors*sizeof(int));
    //texPartInCellNonBonded;
    texPartInCellNonBonded.normalized = false;
    texPartInCellNonBonded.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texPartInCellNonBonded.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texPartInCellNonBonded,partInCellNonBonded,
		    maxNumberPartInCellNonBonded*numNeighbors*sizeof(int));
    //texneighbor0GPU
    texneighbor0GPU.normalized = false;
    texneighbor0GPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbor0GPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighbor0GPU,neighbor0GPU,numNeighbors*sizeof(int));
    //texneighbor1GPU
    texneighbor1GPU.normalized = false;
    texneighbor1GPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbor1GPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighbor1GPU,neighbor1GPU,numNeighbors*sizeof(int));
    //texneighbor2GPU
    texneighbor2GPU.normalized = false;
    texneighbor2GPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbor2GPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighbor2GPU,neighbor2GPU,numNeighbors*sizeof(int));
    //texneighbor3GPU
    texneighbor3GPU.normalized = false;
    texneighbor3GPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbor3GPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighbor3GPU,neighbor3GPU,numNeighbors*sizeof(int));
    //texneighbor4GPU
    texneighbor4GPU.normalized = false;
    texneighbor4GPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbor4GPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighbor4GPU,neighbor4GPU,numNeighbors*sizeof(int));
    //texneighbor5GPU
    texneighbor5GPU.normalized = false;
    texneighbor5GPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbor5GPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighbor5GPU,neighbor5GPU,numNeighbors*sizeof(int));
    //texneighborpxpyGPU
    texneighborpxpyGPU.normalized = false;
    texneighborpxpyGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighborpxpyGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighborpxpyGPU,neighborpxpyGPU,numNeighbors*sizeof(int));
    //texneighborpxmyGPU
    texneighborpxmyGPU.normalized = false;
    texneighborpxmyGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighborpxmyGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighborpxmyGPU,neighborpxmyGPU,numNeighbors*sizeof(int));
    //texneighborpxpzGPU
    texneighborpxpzGPU.normalized = false;
    texneighborpxpzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighborpxpzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighborpxpzGPU,neighborpxpzGPU,numNeighbors*sizeof(int));
    //texneighborpxmzGPU
    texneighborpxmzGPU.normalized = false;
    texneighborpxmzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighborpxmzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighborpxmzGPU,neighborpxmzGPU,numNeighbors*sizeof(int));
    //texneighbormxpyGPU
    texneighbormxpyGPU.normalized = false;
    texneighbormxpyGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbormxpyGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighbormxpyGPU,neighbormxpyGPU,numNeighbors*sizeof(int));
    //texneighbormxmyGPU
    texneighbormxmyGPU.normalized = false;
    texneighbormxmyGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbormxmyGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighbormxmyGPU,neighbormxmyGPU,numNeighbors*sizeof(int));
    //texneighbormxpzGPU
    texneighbormxpzGPU.normalized = false;
    texneighbormxpzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbormxpzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighbormxpzGPU,neighbormxpzGPU,numNeighbors*sizeof(int));
    //texneighbormxmzGPU
    texneighbormxmzGPU.normalized = false;
    texneighbormxmzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbormxmzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighbormxmzGPU,neighbormxmzGPU,numNeighbors*sizeof(int));
    //texneighborpypzGPU
    texneighborpypzGPU.normalized = false;
    texneighborpypzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighborpypzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighborpypzGPU,neighborpypzGPU,numNeighbors*sizeof(int));
    //texneighborpymzGPU
    texneighborpymzGPU.normalized = false;
    texneighborpymzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighborpymzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighborpymzGPU,neighborpymzGPU,numNeighbors*sizeof(int));
    //texneighbormypzGPU
    texneighbormypzGPU.normalized = false;
    texneighbormypzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbormypzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighbormypzGPU,neighbormypzGPU,numNeighbors*sizeof(int));
    //texneighbormymzGPU
    texneighbormymzGPU.normalized = false;
    texneighbormymzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbormymzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighbormymzGPU,neighbormymzGPU,numNeighbors*sizeof(int));
    //texneighborpxpypzGPU
    texneighborpxpypzGPU.normalized = false;
    texneighborpxpypzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighborpxpypzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighborpxpypzGPU,neighborpxpypzGPU,numNeighbors*sizeof(int));
    //texneighborpxpymzGPU
    texneighborpxpymzGPU.normalized = false;
    texneighborpxpymzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighborpxpymzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighborpxpymzGPU,neighborpxpymzGPU,numNeighbors*sizeof(int));
    //texneighborpxmypzGPU
    texneighborpxmypzGPU.normalized = false;
    texneighborpxmypzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighborpxmypzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighborpxmypzGPU,neighborpxmypzGPU,numNeighbors*sizeof(int));
    //texneighborpxmymzGPU
    texneighborpxmymzGPU.normalized = false;
    texneighborpxmymzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighborpxmymzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighborpxmymzGPU,neighborpxmymzGPU,numNeighbors*sizeof(int));
    //texneighbormxpypzGPU
    texneighbormxpypzGPU.normalized = false;
    texneighbormxpypzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbormxpypzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighbormxpypzGPU,neighbormxpypzGPU,numNeighbors*sizeof(int));
    //texneighbormymzGPU
    texneighbormxpymzGPU.normalized = false;
    texneighbormxpymzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbormxpymzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighbormxpymzGPU,neighbormxpymzGPU,numNeighbors*sizeof(int));
    //texneighbormxmypzGPU
    texneighbormxmypzGPU.normalized = false;
    texneighbormxmypzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbormxmypzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighbormxmypzGPU,neighbormxmypzGPU,numNeighbors*sizeof(int));
    //texneighbormxmymzGPU
    texneighbormxmymzGPU.normalized = false;
    texneighbormxmymzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
    texneighbormxmymzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
    hipBindTexture(0,texneighbormxmymzGPU,neighbormxmymzGPU,numNeighbors*sizeof(int));
    
    //init_force_non_bonded();

    int block = (numNeighbors-1)/128 + 1;

    initializeNeighbors<<<block,128>>>(neighbor1GPU,neighbor2GPU,neighbor3GPU,neighbor4GPU,
				       neighborpxpyGPU,neighborpxmyGPU,neighborpxpzGPU,neighborpxmzGPU,
				       neighbormxpyGPU,neighbormxmyGPU,neighbormxpzGPU,neighbormxmzGPU,
				       neighborpypzGPU,neighborpymzGPU,neighbormypzGPU,neighbormymzGPU,
				       neighborpxpypzGPU,neighborpxpymzGPU,
				       neighborpxmypzGPU,neighborpxmymzGPU,
				       neighbormxpypzGPU,neighbormxpymzGPU,
				       neighbormxmypzGPU,neighbormxmymzGPU);
    initializeNeighbors2<<<block,128>>>(neighbor0GPU,neighbor1GPU,neighbor2GPU,
					neighbor3GPU,neighbor4GPU,neighbor5GPU);

    initForcesNonBonded();

  }


  initDelta();
  allocateErrorArray();

  hipMalloc((void**)&pc,sizeof(particlesincell));
  
  initParticlesInCell<<<1,1>>>(partincellX,partincellY,partincellZ,
			       countparticlesincellX,countparticlesincellY,countparticlesincellZ,
			       countPartInCellNonBonded,partInCellNonBonded,pc);

  //No-slip Test
  //cutilSafeCall(hipMalloc((void**)&saveForceX,np*sizeof(double)));
  //cutilSafeCall(hipMalloc((void**)&saveForceY,np*sizeof(double)));
  //cutilSafeCall(hipMalloc((void**)&saveForceZ,np*sizeof(double)));
  

  
  cout << "CREATE BOUNDARIES GPU :         DONE" << endl; 

  return 1;
}
