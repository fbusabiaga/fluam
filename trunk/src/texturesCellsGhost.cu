// Filename: texturesCellsGhost.cu
//
// Copyright (c) 2010-2013, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


bool texturesCellsGhost(){

  //texVxGPU;
  texVxGPU.normalized = false;
  texVxGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texVxGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
  cutilSafeCall( hipBindTexture(0,texVxGPU,vxGPU,ncellst*sizeof(double)));
  //texVyGPU;
  texVyGPU.normalized = false;
  texVyGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texVyGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
  cutilSafeCall( hipBindTexture(0,texVyGPU,vyGPU,ncellst*sizeof(double)));
  //texVzGPU;
  texVzGPU.normalized = false;
  texVzGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texVzGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
  cutilSafeCall( hipBindTexture(0,texVzGPU,vzGPU,ncellst*sizeof(double)));



  cout << "TEXTURES CELLS :                DONE " << endl;

  return 1;

}
