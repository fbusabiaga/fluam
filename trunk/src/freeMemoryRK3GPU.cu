// Filename: freeMemoryRK3GPU.cu
//
// Copyright (c) 2010-2013, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


bool freeMemoryRK3GPU(){
  hipFree(densityGPU);
  cutilSafeCall(hipUnbindTexture(texVxGPU));
  cutilSafeCall(hipUnbindTexture(texVyGPU));
  cutilSafeCall(hipUnbindTexture(texVzGPU));    
  hipFree(vxGPU);
  hipFree(vyGPU);
  hipFree(vzGPU);
  hipFree(densityPredictionGPU);
  hipFree(vxPredictionGPU);
  hipFree(vyPredictionGPU);
  hipFree(vzPredictionGPU);

  hipFree(dmGPU);
  hipFree(dpxGPU);
  hipFree(dpyGPU);
  hipFree(dpzGPU);

  hipFree(rxcellGPU);
  hipFree(rycellGPU);
  hipFree(rzcellGPU);

  
  cutilSafeCall(hipUnbindTexture(texvecino0GPU));
  cutilSafeCall(hipUnbindTexture(texvecino1GPU));
  cutilSafeCall(hipUnbindTexture(texvecino2GPU));
  cutilSafeCall(hipUnbindTexture(texvecino3GPU));
  cutilSafeCall(hipUnbindTexture(texvecino4GPU));
  cutilSafeCall(hipUnbindTexture(texvecino5GPU));
  cutilSafeCall(hipUnbindTexture(texvecinopxpyGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopxmyGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopxpzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopxmzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomxpyGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomxmyGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomxpzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomxmzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopypzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopymzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomypzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomymzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopxpypzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopxpymzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopxmypzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopxmymzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomxpypzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomxpymzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomxmypzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomxmymzGPU));

  hipFree(vecino0GPU);
  hipFree(vecino1GPU);
  hipFree(vecino2GPU);
  hipFree(vecino3GPU);
  hipFree(vecino4GPU);
  hipFree(vecino5GPU);

  hipFree(vecinopxpyGPU);
  hipFree(vecinopxmyGPU);
  hipFree(vecinopxpzGPU);
  hipFree(vecinopxmzGPU);
  hipFree(vecinomxpyGPU);
  hipFree(vecinomxmyGPU);
  hipFree(vecinomxpzGPU);
  hipFree(vecinomxmzGPU);  
  hipFree(vecinopypzGPU);
  hipFree(vecinopymzGPU);
  hipFree(vecinomypzGPU);
  hipFree(vecinomymzGPU);
  hipFree(vecinopxpypzGPU);
  hipFree(vecinopxpymzGPU);
  hipFree(vecinopxmypzGPU);
  hipFree(vecinopxmymzGPU);
  hipFree(vecinomxpypzGPU);
  hipFree(vecinomxpymzGPU);
  hipFree(vecinomxmypzGPU);
  hipFree(vecinomxmymzGPU);
  hipFree(stepGPU);

  cout << "FREE MEMORY GPU :               DONE" << endl; 


  return 1;
}

