// Filename: freeMemoryRK3GPU.cu
//
// Copyright (c) 2010-2012, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


bool freeMemoryRK3GPU(){
  hipFree(densityGPU);
  hipUnbindTexture(texVxGPU);
  hipUnbindTexture(texVyGPU);
  hipUnbindTexture(texVzGPU);    
  hipFree(vxGPU);
  hipFree(vyGPU);
  hipFree(vzGPU);
  hipFree(densityPredictionGPU);
  hipFree(vxPredictionGPU);
  hipFree(vyPredictionGPU);
  hipFree(vzPredictionGPU);

  hipFree(dmGPU);
  hipFree(dpxGPU);
  hipFree(dpyGPU);
  hipFree(dpzGPU);

  hipFree(rxcellGPU);
  hipFree(rycellGPU);
  hipFree(rzcellGPU);

  
  hipUnbindTexture(texvecino0GPU);
  hipUnbindTexture(texvecino1GPU);
  hipUnbindTexture(texvecino2GPU);
  hipUnbindTexture(texvecino3GPU);
  hipUnbindTexture(texvecino4GPU);
  hipUnbindTexture(texvecino5GPU);
  hipUnbindTexture(texvecinopxpyGPU);
  hipUnbindTexture(texvecinopxmyGPU);
  hipUnbindTexture(texvecinopxpzGPU);
  hipUnbindTexture(texvecinopxmzGPU);
  hipUnbindTexture(texvecinomxpyGPU);
  hipUnbindTexture(texvecinomxmyGPU);
  hipUnbindTexture(texvecinomxpzGPU);
  hipUnbindTexture(texvecinomxmzGPU);
  hipUnbindTexture(texvecinopypzGPU);
  hipUnbindTexture(texvecinopymzGPU);
  hipUnbindTexture(texvecinomypzGPU);
  hipUnbindTexture(texvecinomymzGPU);
  hipUnbindTexture(texvecinopxpypzGPU);
  hipUnbindTexture(texvecinopxpymzGPU);
  hipUnbindTexture(texvecinopxmypzGPU);
  hipUnbindTexture(texvecinopxmymzGPU);
  hipUnbindTexture(texvecinomxpypzGPU);
  hipUnbindTexture(texvecinomxpymzGPU);
  hipUnbindTexture(texvecinomxmypzGPU);
  hipUnbindTexture(texvecinomxmymzGPU);

  hipFree(vecino0GPU);
  hipFree(vecino1GPU);
  hipFree(vecino2GPU);
  hipFree(vecino3GPU);
  hipFree(vecino4GPU);
  hipFree(vecino5GPU);

  hipFree(vecinopxpyGPU);
  hipFree(vecinopxmyGPU);
  hipFree(vecinopxpzGPU);
  hipFree(vecinopxmzGPU);
  hipFree(vecinomxpyGPU);
  hipFree(vecinomxmyGPU);
  hipFree(vecinomxpzGPU);
  hipFree(vecinomxmzGPU);  
  hipFree(vecinopypzGPU);
  hipFree(vecinopymzGPU);
  hipFree(vecinomypzGPU);
  hipFree(vecinomymzGPU);
  hipFree(vecinopxpypzGPU);
  hipFree(vecinopxpymzGPU);
  hipFree(vecinopxmypzGPU);
  hipFree(vecinopxmymzGPU);
  hipFree(vecinomxpypzGPU);
  hipFree(vecinomxpymzGPU);
  hipFree(vecinomxmypzGPU);
  hipFree(vecinomxmymzGPU);
  hipFree(stepGPU);

  cout << "FREE MEMORY GPU :               DONE" << endl; 


  return 1;
}

