// Filename: freeBoundariesRK2GPU.cu
//
// Copyright (c) 2010-2013, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


bool freeBoundariesRK2GPU(){

  cutilSafeCall(hipUnbindTexture(texrxboundaryGPU));    
  cutilSafeCall(hipUnbindTexture(texryboundaryGPU));    
  cutilSafeCall(hipUnbindTexture(texrzboundaryGPU));
  //cutilSafeCall(hipUnbindTexture(texfxboundaryGPU));
  //cutilSafeCall(hipUnbindTexture(texfyboundaryGPU));
  //cutilSafeCall(hipUnbindTexture(texfzboundaryGPU));

  cutilSafeCall(hipFree(rxboundaryGPU));
  cutilSafeCall(hipFree(ryboundaryGPU));
  cutilSafeCall(hipFree(rzboundaryGPU));
  cutilSafeCall(hipFree(rxboundaryPredictionGPU));
  cutilSafeCall(hipFree(ryboundaryPredictionGPU));
  cutilSafeCall(hipFree(rzboundaryPredictionGPU));
  cutilSafeCall(hipFree(vxboundaryGPU));
  cutilSafeCall(hipFree(vyboundaryGPU));
  cutilSafeCall(hipFree(vzboundaryGPU));
  cutilSafeCall(hipFree(vxboundaryPredictionGPU));
  cutilSafeCall(hipFree(vyboundaryPredictionGPU));
  cutilSafeCall(hipFree(vzboundaryPredictionGPU));
  cutilSafeCall(hipFree(fxboundaryGPU));
  cutilSafeCall(hipFree(fyboundaryGPU));
  cutilSafeCall(hipFree(fzboundaryGPU));

  cutilSafeCall(hipUnbindTexture(texCountParticlesInCellX));
  cutilSafeCall(hipUnbindTexture(texCountParticlesInCellY));
  cutilSafeCall(hipUnbindTexture(texCountParticlesInCellZ));
  cutilSafeCall(hipUnbindTexture(texPartInCellX));
  cutilSafeCall(hipUnbindTexture(texPartInCellY));
  cutilSafeCall(hipUnbindTexture(texPartInCellZ));
  cutilSafeCall(hipUnbindTexture(texCountParticlesInCellNonBonded));
  cutilSafeCall(hipUnbindTexture(texPartInCellNonBonded));



  if(setparticles){
    cutilSafeCall(hipUnbindTexture(texCountParticlesInCellNonBonded));
    cutilSafeCall(hipFree(countPartInCellNonBonded));

    cutilSafeCall(hipUnbindTexture(texPartInCellNonBonded));
    cutilSafeCall(hipFree(partInCellNonBonded));

    cutilSafeCall(hipUnbindTexture(texneighbor0GPU));
    cutilSafeCall(hipUnbindTexture(texneighbor1GPU));
    cutilSafeCall(hipUnbindTexture(texneighbor2GPU));
    cutilSafeCall(hipUnbindTexture(texneighbor3GPU));
    cutilSafeCall(hipUnbindTexture(texneighbor4GPU));
    cutilSafeCall(hipUnbindTexture(texneighbor5GPU));
    cutilSafeCall(hipUnbindTexture(texneighborpxpyGPU));
    cutilSafeCall(hipUnbindTexture(texneighborpxmyGPU));
    cutilSafeCall(hipUnbindTexture(texneighborpxpzGPU));
    cutilSafeCall(hipUnbindTexture(texneighborpxmzGPU));
    cutilSafeCall(hipUnbindTexture(texneighbormxpyGPU));
    cutilSafeCall(hipUnbindTexture(texneighbormxmyGPU));
    cutilSafeCall(hipUnbindTexture(texneighbormxpzGPU));
    cutilSafeCall(hipUnbindTexture(texneighbormxmzGPU));
    cutilSafeCall(hipUnbindTexture(texneighborpypzGPU));
    cutilSafeCall(hipUnbindTexture(texneighborpymzGPU));
    cutilSafeCall(hipUnbindTexture(texneighbormypzGPU));
    cutilSafeCall(hipUnbindTexture(texneighbormymzGPU));
    cutilSafeCall(hipUnbindTexture(texneighborpxpypzGPU));
    cutilSafeCall(hipUnbindTexture(texneighborpxpymzGPU));
    cutilSafeCall(hipUnbindTexture(texneighborpxmypzGPU));
    cutilSafeCall(hipUnbindTexture(texneighborpxmymzGPU));
    cutilSafeCall(hipUnbindTexture(texneighbormxpypzGPU));
    cutilSafeCall(hipUnbindTexture(texneighbormxpymzGPU));
    cutilSafeCall(hipUnbindTexture(texneighbormxmypzGPU));
    cutilSafeCall(hipUnbindTexture(texneighbormxmymzGPU));
    cutilSafeCall(hipFree(neighbor0GPU));
    cutilSafeCall(hipFree(neighbor1GPU));
    cutilSafeCall(hipFree(neighbor2GPU));
    cutilSafeCall(hipFree(neighbor3GPU));
    cutilSafeCall(hipFree(neighbor4GPU));
    cutilSafeCall(hipFree(neighbor5GPU));
    cutilSafeCall(hipFree(neighborpxpyGPU));
    cutilSafeCall(hipFree(neighborpxmyGPU));
    cutilSafeCall(hipFree(neighborpxpzGPU));
    cutilSafeCall(hipFree(neighborpxmzGPU));
    cutilSafeCall(hipFree(neighbormxpyGPU));
    cutilSafeCall(hipFree(neighbormxmyGPU));
    cutilSafeCall(hipFree(neighbormxpzGPU));
    cutilSafeCall(hipFree(neighbormxmzGPU));
    cutilSafeCall(hipFree(neighborpypzGPU));
    cutilSafeCall(hipFree(neighborpymzGPU));
    cutilSafeCall(hipFree(neighbormypzGPU));
    cutilSafeCall(hipFree(neighbormymzGPU));
    cutilSafeCall(hipFree(neighborpxpypzGPU));
    cutilSafeCall(hipFree(neighborpxpymzGPU));
    cutilSafeCall(hipFree(neighborpxmypzGPU));
    cutilSafeCall(hipFree(neighborpxmymzGPU));
    cutilSafeCall(hipFree(neighbormxpypzGPU));
    cutilSafeCall(hipFree(neighbormxpymzGPU));
    cutilSafeCall(hipFree(neighbormxmypzGPU));
    cutilSafeCall(hipFree(neighbormxmymzGPU));
  }

  freeErrorArray();
  cutilSafeCall(hipFree(pc));
  freeDelta();

  if(setparticles){
    cutilSafeCall(hipUnbindTexture(texforceNonBonded1));
    cutilSafeCall(hipFreeArray(forceNonBonded1));
  }

  //No-slip Test
  //cutilSafeCall(hipFree(saveForceX));
  //cutilSafeCall(hipFree(saveForceY));
  //cutilSafeCall(hipFree(saveForceZ));

  cout << "FREE BOUNDARIES GPU :           DONE" << endl; 

  return 1;
}
