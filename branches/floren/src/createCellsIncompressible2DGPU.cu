#include "hip/hip_runtime.h"
// Filename: createCellsIncompressibleGPU.cu
//
// Copyright (c) 2010-2013, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


#define GPUVARIABLES 1


bool createCellsIncompressible2DGPU(){
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mxGPU),&mx,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(myGPU),&my,sizeof(int)));

  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mxtGPU),&mxt,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mytGPU),&myt,sizeof(int)));


  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ncellsGPU),&ncells,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ncellstGPU),&ncellst,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(lxGPU),&lx,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(lyGPU),&ly,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dtGPU),&dt,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(volumeGPU),&cVolume,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(shearviscosityGPU),&shearviscosity,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(temperatureGPU),&temperature,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(thermostatGPU),&thermostat,sizeof(bool)));

  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(densfluidGPU),&densfluid,sizeof(double)));

  cutilSafeCall(hipMalloc((void**)&vxGPU,ncells*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vyGPU,ncells*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vxPredictionGPU,ncells*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vyPredictionGPU,ncells*sizeof(double)));

 
  cutilSafeCall(hipMalloc((void**)&rxcellGPU,ncells*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&rycellGPU,ncells*sizeof(double)));

  //FACT1 DIFFERENT FOR INCOMPRESSIBLE
  double fact1 = sqrt((4.*temperature*shearviscosity*dt)/(cVolume*densfluid*densfluid));
  //FACT4 DIFFERENT FOR INCOMPRESSIBLE
  double fact4 = sqrt((2.*temperature*shearviscosity*dt)/(cVolume*densfluid*densfluid));
  double fact5 = sqrt(1./(dt*cVolume));

  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(fact1GPU),&fact1,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(fact4GPU),&fact4,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(fact5GPU),&fact5,sizeof(double)));


  fact1 = lx/double(mx);
  double fact2 = ly/double(my);
  double fact3 = lz/double(mz);
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dxGPU),&fact1,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dyGPU),&fact2,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dzGPU),&fact3,sizeof(double)));

  fact1 = double(mx)/lx;
  fact2 = double(my)/ly;
  fact3 = double(mz)/lz;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invdxGPU),&fact1,sizeof(double)));  
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invdyGPU),&fact2,sizeof(double)));  
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invdzGPU),&fact3,sizeof(double)));  
  fact1 = 1./dt;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invdtGPU),&fact1,sizeof(double)));
  fact1 = 1./lx;
  fact2 = 1./ly;
  fact3 = 1./lz;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invlxGPU),&fact1,sizeof(double)));  
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invlyGPU),&fact2,sizeof(double)));  
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invlzGPU),&fact3,sizeof(double)));

 
  bool auxbool = 0;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(setparticlesGPU),&auxbool,sizeof(bool)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(setboundaryGPU),&auxbool,sizeof(bool)));


  long long auxulonglong = 0;
  cutilSafeCall(hipMalloc((void**)&stepGPU,sizeof(long long)));
  cutilSafeCall(hipMemcpy(stepGPU,&auxulonglong,sizeof(long long),hipMemcpyHostToDevice));


  cutilSafeCall(hipMalloc((void**)&vecino0GPU,ncells*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&vecino1GPU,ncells*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&vecino2GPU,ncells*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&vecino3GPU,ncells*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&vecino4GPU,ncells*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&vecino5GPU,ncells*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&vecinopxpyGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopxmyGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopxpzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopxmzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomxpyGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomxmyGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomxpzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomxmzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopypzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopymzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomypzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomymzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopxpypzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopxpymzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopxmypzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinopxmymzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomxpypzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomxpymzGPU,ncells*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&vecinomxmypzGPU,ncells*sizeof(int))); 
  cutilSafeCall(hipMalloc((void**)&vecinomxmymzGPU,ncells*sizeof(int))); 


  //Factors for the update in fourier space
  cutilSafeCall(hipMalloc((void**)&gradKx,     mx*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&gradKy,     my*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&gradKz,     mz*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&expKx,      mx*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&expKy,      my*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&expKz,      mz*sizeof(hipfftDoubleComplex)));

  cutilSafeCall(hipMalloc((void**)&pF,sizeof(prefactorsFourier)));

  //cutilSafeCall(hipMalloc((void**)&WxZ,ncells*sizeof(hipfftDoubleComplex)));
  //cutilSafeCall(hipMalloc((void**)&WyZ,ncells*sizeof(hipfftDoubleComplex)));
  //cutilSafeCall(hipMalloc((void**)&WzZ,ncells*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&vxZ,ncells*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&vyZ,ncells*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&vzZ,ncells*sizeof(hipfftDoubleComplex))); 

  if(quasiNeutrallyBuoyant || quasiNeutrallyBuoyant2D){
    cutilSafeCall(hipMalloc((void**)&advXGPU,ncells*sizeof(double)));
    cutilSafeCall(hipMalloc((void**)&advYGPU,ncells*sizeof(double)));
    cutilSafeCall(hipMalloc((void**)&advZGPU,ncells*sizeof(double)));
  }

  
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(pressurea0GPU),&pressurea0,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(pressurea1GPU),&pressurea1,sizeof(double)));
  //cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(pressurea2GPU),&pressurea2,sizeof(double)));


  cout << "CREATE CELLS GPU :              DONE" << endl;

  return 1;
}
