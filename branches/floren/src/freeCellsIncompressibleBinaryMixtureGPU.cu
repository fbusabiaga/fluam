// Filename: freeCellsIncompressibleBinaryMixtureGPU.cu
//
// Copyright (c) 2010-2013, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


bool freeCellsIncompressibleBinaryMixtureGPU(){
  cutilSafeCall(hipUnbindTexture(texVxGPU));
  cutilSafeCall(hipUnbindTexture(texVyGPU));
  cutilSafeCall(hipUnbindTexture(texVzGPU));    
  cutilSafeCall(hipFree(vxGPU));
  cutilSafeCall(hipFree(vyGPU));
  cutilSafeCall(hipFree(vzGPU));
  cutilSafeCall(hipFree(vxPredictionGPU));
  cutilSafeCall(hipFree(vyPredictionGPU));
  cutilSafeCall(hipFree(vzPredictionGPU));
  
  cutilSafeCall(hipFree(cGPU));
  cutilSafeCall(hipFree(cPredictionGPU));

  cutilSafeCall(hipFree(rxcellGPU));
  cutilSafeCall(hipFree(rycellGPU));
  cutilSafeCall(hipFree(rzcellGPU));

  cutilSafeCall(hipUnbindTexture(texvecino0GPU));
  cutilSafeCall(hipUnbindTexture(texvecino1GPU));
  cutilSafeCall(hipUnbindTexture(texvecino2GPU));
  cutilSafeCall(hipUnbindTexture(texvecino3GPU));
  cutilSafeCall(hipUnbindTexture(texvecino4GPU));
  cutilSafeCall(hipUnbindTexture(texvecino5GPU));
  cutilSafeCall(hipUnbindTexture(texvecinopxpyGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopxmyGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopxpzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopxmzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomxpyGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomxmyGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomxpzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomxmzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopypzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopymzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomypzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomymzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopxpypzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopxpymzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopxmypzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinopxmymzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomxpypzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomxpymzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomxmypzGPU));
  cutilSafeCall(hipUnbindTexture(texvecinomxmymzGPU));

  cutilSafeCall(hipFree(vecino0GPU));
  cutilSafeCall(hipFree(vecino1GPU));
  cutilSafeCall(hipFree(vecino2GPU));
  cutilSafeCall(hipFree(vecino3GPU));
  cutilSafeCall(hipFree(vecino4GPU));
  cutilSafeCall(hipFree(vecino5GPU));
  cutilSafeCall(hipFree(vecinopxpyGPU));
  cutilSafeCall(hipFree(vecinopxmyGPU));
  cutilSafeCall(hipFree(vecinopxpzGPU));
  cutilSafeCall(hipFree(vecinopxmzGPU));
  cutilSafeCall(hipFree(vecinomxpyGPU));
  cutilSafeCall(hipFree(vecinomxmyGPU));
  cutilSafeCall(hipFree(vecinomxpzGPU));
  cutilSafeCall(hipFree(vecinomxmzGPU));
  cutilSafeCall(hipFree(vecinopypzGPU));
  cutilSafeCall(hipFree(vecinopymzGPU));
  cutilSafeCall(hipFree(vecinomypzGPU));
  cutilSafeCall(hipFree(vecinomymzGPU));
  cutilSafeCall(hipFree(vecinopxpypzGPU));
  cutilSafeCall(hipFree(vecinopxpymzGPU));
  cutilSafeCall(hipFree(vecinopxmypzGPU));
  cutilSafeCall(hipFree(vecinopxmymzGPU));
  cutilSafeCall(hipFree(vecinomxpypzGPU));
  cutilSafeCall(hipFree(vecinomxpymzGPU));
  cutilSafeCall(hipFree(vecinomxmypzGPU));
  cutilSafeCall(hipFree(vecinomxmymzGPU));

  cutilSafeCall(hipFree(stepGPU));

  cutilSafeCall(hipFree(pF));

  cutilSafeCall(hipFree(gradKx));
  cutilSafeCall(hipFree(gradKy));
  cutilSafeCall(hipFree(gradKz));
  cutilSafeCall(hipFree(expKx));
  cutilSafeCall(hipFree(expKy));
  cutilSafeCall(hipFree(expKz));

  //cutilSafeCall(hipFree(WxGPU));
  //cutilSafeCall(hipFree(WyGPU));
  //cutilSafeCall(hipFree(WzGPU));
  cutilSafeCall(hipFree(WxZ));
  cutilSafeCall(hipFree(WyZ));
  cutilSafeCall(hipFree(WzZ));
  cutilSafeCall(hipFree(vxZ));
  cutilSafeCall(hipFree(vyZ));
  cutilSafeCall(hipFree(vzZ));
  cutilSafeCall(hipFree(cZ));

  cout << "FREE MEMORY GPU :               DONE" << endl; 


  return 1;
}

