#include "hip/hip_runtime.h"
// Filename: kernelFeedGhostCellsBinaryMixture.cu
//
// Copyright (c) 2010-2013, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


__global__ void kernelFeedGhostCellsBinaryMixture(int* ghostToPIGPU, 
						  int* ghostToGhostGPU,
						  double* densityGPU, 
						  double* densityPredictionGPU,
						  double* vxGPU, 
						  double* vyGPU, 
						  double* vzGPU,
						  double* vxPredictionGPU, 
						  double* vyPredictionGPU, 
						  double* vzPredictionGPU,
						  double* cGPU, 
						  double* cPredictionGPU, 
						  double* d_rand, 
						  int substep){

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i>=(ncellstGPU-ncellsGPU)) return;
    int ireal = ghostToPIGPU[i];
    int ighost = ghostToGhostGPU[i];
    densityGPU[ighost] = densityGPU[ireal];
    densityPredictionGPU[ighost] = densityPredictionGPU[ireal];
    vxGPU[ighost] = vxGPU[ireal];
    vyGPU[ighost] = vyGPU[ireal];
    vzGPU[ighost] = vzGPU[ireal];
    vxPredictionGPU[ighost] = vxPredictionGPU[ireal];
    vyPredictionGPU[ighost] = vyPredictionGPU[ireal];
    vzPredictionGPU[ighost] = vzPredictionGPU[ireal];

    cGPU[ighost] = cGPU[ireal];
    cPredictionGPU[ighost] = cPredictionGPU[ireal];

    
    int n0;
    n0 = substep * ncellstGPU * 18;
    d_rand[n0+ighost] = d_rand[n0+ireal];
    d_rand[n0+ighost+ncellstGPU] = d_rand[n0+ireal+ncellstGPU];
    d_rand[n0+ighost+2*ncellstGPU] = d_rand[n0+ireal+2*ncellstGPU];
    d_rand[n0+ighost+3*ncellstGPU] = d_rand[n0+ireal+3*ncellstGPU];
    d_rand[n0+ighost+4*ncellstGPU] = d_rand[n0+ireal+4*ncellstGPU];
    d_rand[n0+ighost+5*ncellstGPU] = d_rand[n0+ireal+5*ncellstGPU];
    d_rand[n0+ighost+6*ncellstGPU] = d_rand[n0+ireal+6*ncellstGPU];
    d_rand[n0+ighost+7*ncellstGPU] = d_rand[n0+ireal+7*ncellstGPU];
    d_rand[n0+ighost+8*ncellstGPU] = d_rand[n0+ireal+8*ncellstGPU];
    d_rand[n0+ighost+9*ncellstGPU] = d_rand[n0+ireal+9*ncellstGPU];
    d_rand[n0+ighost+10*ncellstGPU] = d_rand[n0+ireal+10*ncellstGPU];
    d_rand[n0+ighost+11*ncellstGPU] = d_rand[n0+ireal+11*ncellstGPU];
    d_rand[n0+ighost+12*ncellstGPU] = d_rand[n0+ireal+12*ncellstGPU];
    d_rand[n0+ighost+13*ncellstGPU] = d_rand[n0+ireal+13*ncellstGPU];
    d_rand[n0+ighost+14*ncellstGPU] = d_rand[n0+ireal+14*ncellstGPU];
    d_rand[n0+ighost+15*ncellstGPU] = d_rand[n0+ireal+15*ncellstGPU];
    d_rand[n0+ighost+16*ncellstGPU] = d_rand[n0+ireal+16*ncellstGPU];
    d_rand[n0+ighost+17*ncellstGPU] = d_rand[n0+ireal+17*ncellstGPU];    

}
