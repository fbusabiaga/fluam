#include "hip/hip_runtime.h"
// Filename: initForcesNonBonded.cu
//
// Copyright (c) 2010-2013, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


float functionForceNonBonded1(double r){
  float sigma, epsilon;
  sigma = 2 * lx / double(mx);
  epsilon = temperature ;
  //return -epsilon * ( r - sigma);
  return 48. * epsilon * (pow(sigma/r,12) - 0.5*pow(sigma/r,6))/r;


}

bool initForcesNonBonded(){
  texforceNonBonded1.normalized = true;
  texforceNonBonded1.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texforceNonBonded1.filterMode = hipFilterModeLinear;//hipFilterModeLinear and hipFilterModePoint

  float *h_data;
  int size = 4096;
  h_data = new float[size];
  float r, dr;
  float cutoff2 = cutoff * cutoff;
  dr = cutoff2/float(size);
  r = 0.5 * dr;
  for(int i=0;i<size;i++){
    h_data[i] = functionForceNonBonded1(sqrt(r))/sqrt(r);
    //cout << sqrt(r) << "   " << h_data[i] << endl;
    r += dr;
  }
  h_data[size-1] = 0.;
  h_data[0] = 0.;
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  cutilSafeCall( hipMallocArray( &forceNonBonded1, &channelDesc, size, 1 )); 
  cutilSafeCall( hipMemcpyToArray( forceNonBonded1, 0, 0, h_data, size*sizeof(float), hipMemcpyHostToDevice));
  cutilSafeCall( hipBindTextureToArray( texforceNonBonded1, forceNonBonded1, channelDesc));


  /*r = 0.5 * dr;
    for(int i=0;i<size;i++){
    cout << r << " " << h_data[i] << endl;
    r += dr;
    }*/
  cout << "INIT FORCE NON-BONDED 1 COMPLETED" << endl;
  delete[] h_data;


  return 1;
}
