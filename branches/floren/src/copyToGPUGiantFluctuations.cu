// Filename: copyToGPUGiantFluctuations.cu
//
// Copyright (c) 2010-2013, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


bool copyToGPUGiantFluctuations(){
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(cWall0GPU),&cWall0,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(cWall1GPU),&cWall1,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(densityWall0GPU),&densityWall0,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(densityWall1GPU),&densityWall1,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(vxWall0GPU),&vxWall0,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(vxWall1GPU),&vxWall1,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(vyWall0GPU),&vyWall0,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(vyWall1GPU),&vyWall1,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(vzWall0GPU),&vzWall0,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(vzWall1GPU),&vzWall1,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(soretCoefficientGPU),&soretCoefficient,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gradTemperatureGPU),&gradTemperature,sizeof(double)));

  cout << "COPY TO GPU :                   DONE" << endl;

  return 1;
}
