#include "hip/hip_runtime.h"

// Filename: kernelConstructW.cu
//
// Copyright (c) 2010-2013, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.



//In this kernel we construct the vector W
//In the first substep
//  W = u^n + 0.5*dt*nu*L*u^n + Advection(u^n) + (dt/rho)*f^n_{noise}
//
//In the second substep
//  W = u^n + 0.5*dt*nu*L*u^n + Advection(u^{n+1/2}) + (dt/rho)*f^n_{noise}
//with u^{n+1/2} = 0.5 * (u^n + u^{n+1}_{result from first substep})


__global__ void kernelConstructW_1(double *vxPredictionGPU, 
				   double *vyPredictionGPU, 
				   double *vzPredictionGPU, 
				   hipfftDoubleComplex *WxZ, 
				   hipfftDoubleComplex *WyZ, 
				   hipfftDoubleComplex *WzZ, 
				   double *d_rand){
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i>=ncellsGPU) return;   

  double wx, wy, wz;
  double vx, vy, vz;
  double vx0, vx1, vx2, vx3, vx4, vx5;
  double vy0, vy1, vy2, vy3, vy4, vy5;
  double vz0, vz1, vz2, vz3, vz4, vz5;
  int vecino0, vecino1, vecino2, vecino3, vecino4, vecino5; 
  int vecinopxmy, vecinopxmz;
  int vecinomxpy, vecinomxpz;
  int vecinopymz, vecinomypz;
  double vxmxpy,vxmxpz;
  double vypxmy,vymypz;
  double vzpxmz,vzpymz;

  vecino0 = tex1Dfetch(texvecino0GPU,i);
  vecino1 = tex1Dfetch(texvecino1GPU,i);
  vecino2 = tex1Dfetch(texvecino2GPU,i);
  vecino3 = tex1Dfetch(texvecino3GPU,i);
  //vecinopxpx = tex1Dfetch(texvecino3GPU, vecino3);
  vecino4 = tex1Dfetch(texvecino4GPU,i);
  //vecinopypy = tex1Dfetch(texvecino4GPU, vecino4);
  vecino5 = tex1Dfetch(texvecino5GPU,i);
  //vecinopzpz = tex1Dfetch(texvecino5GPU, vecino5);
  //vecinopxpy = tex1Dfetch(texvecinopxpyGPU,i);
  vecinopxmy = tex1Dfetch(texvecinopxmyGPU,i);
  //vecinopxpz = tex1Dfetch(texvecinopxpzGPU,i);
  vecinopxmz = tex1Dfetch(texvecinopxmzGPU,i);
  vecinomxpy = tex1Dfetch(texvecinomxpyGPU,i);
  vecinomxpz = tex1Dfetch(texvecinomxpzGPU,i);
  //vecinopypz = tex1Dfetch(texvecinopypzGPU,i);
  vecinopymz = tex1Dfetch(texvecinopymzGPU,i);
  vecinomypz = tex1Dfetch(texvecinomypzGPU,i);

  vx = fetch_double(texVxGPU,i);
  vy = fetch_double(texVyGPU,i);
  vz = fetch_double(texVzGPU,i);
  vx0 = fetch_double(texVxGPU,vecino0);
  vx1 = fetch_double(texVxGPU,vecino1);
  vx2 = fetch_double(texVxGPU,vecino2);
  vx3 = fetch_double(texVxGPU,vecino3);
  vx4 = fetch_double(texVxGPU,vecino4);
  vx5 = fetch_double(texVxGPU,vecino5);
  vy0 = fetch_double(texVyGPU,vecino0);
  vy1 = fetch_double(texVyGPU,vecino1);
  vy2 = fetch_double(texVyGPU,vecino2);
  vy3 = fetch_double(texVyGPU,vecino3);
  vy4 = fetch_double(texVyGPU,vecino4);
  vy5 = fetch_double(texVyGPU,vecino5);
  vz0 = fetch_double(texVzGPU,vecino0);
  vz1 = fetch_double(texVzGPU,vecino1);
  vz2 = fetch_double(texVzGPU,vecino2);
  vz3 = fetch_double(texVzGPU,vecino3);
  vz4 = fetch_double(texVzGPU,vecino4);
  vz5 = fetch_double(texVzGPU,vecino5);
  vxmxpy = fetch_double(texVxGPU,vecinomxpy);
  vxmxpz = fetch_double(texVxGPU,vecinomxpz);
  vypxmy = fetch_double(texVyGPU,vecinopxmy);
  vymypz = fetch_double(texVyGPU,vecinomypz);
  vzpxmz = fetch_double(texVzGPU,vecinopxmz);
  vzpymz = fetch_double(texVzGPU,vecinopymz);



  //Laplacian part
  wx  = invdxGPU * invdxGPU * (vx3 - 2*vx + vx2);
  wx += invdyGPU * invdyGPU * (vx4 - 2*vx + vx1);
  wx += invdzGPU * invdzGPU * (vx5 - 2*vx + vx0);
  wx  = 0.5 * dtGPU * (shearviscosityGPU/densfluidGPU) * wx;
  wy  = invdxGPU * invdxGPU * (vy3 - 2*vy + vy2);
  wy += invdyGPU * invdyGPU * (vy4 - 2*vy + vy1);
  wy += invdzGPU * invdzGPU * (vy5 - 2*vy + vy0);
  wy  = 0.5 * dtGPU * (shearviscosityGPU/densfluidGPU) * wy;
  wz  = invdxGPU * invdxGPU * (vz3 - 2*vz + vz2);
  wz += invdyGPU * invdyGPU * (vz4 - 2*vz + vz1);
  wz += invdzGPU * invdzGPU * (vz5 - 2*vz + vz0);
  wz  = 0.5 * dtGPU * (shearviscosityGPU/densfluidGPU) * wz;

  //Previous Velocity
  wx += vx;
  wy += vy;
  wz += vz;
  
  //Advection part
  double advX, advY, advZ; 
  advX  = invdxGPU * ((vx3+vx)*(vx3+vx) - (vx+vx2)*(vx+vx2));
  advX += invdyGPU * ((vx4+vx)*(vy3+vy) - (vx+vx1)*(vypxmy+vy1));
  advX += invdzGPU * ((vx5+vx)*(vz3+vz) - (vx+vx0)*(vzpxmz+vz0));
  advX  = 0.25 * dtGPU * advX;
  advY  = invdxGPU * ((vy3+vy)*(vx4+vx) - (vy+vy2)*(vxmxpy+vx2));
  advY += invdyGPU * ((vy4+vy)*(vy4+vy) - (vy+vy1)*(vy+vy1));
  advY += invdzGPU * ((vy5+vy)*(vz4+vz) - (vy+vy0)*(vzpymz+vz0));
  advY  = 0.25 * dtGPU * advY;
  advZ  = invdxGPU * ((vz3+vz)*(vx5+vx) - (vz+vz2)*(vxmxpz+vx2));
  advZ += invdyGPU * ((vz4+vz)*(vy5+vy) - (vz+vz1)*(vymypz+vy1));
  advZ += invdzGPU * ((vz5+vz)*(vz5+vz) - (vz+vz0)*(vz+vz0));
  advZ  = 0.25 * dtGPU * advZ;

  //advX=0; advY=0; advZ=0;
  wx -= advX;
  wy -= advY;
  wz -= advZ;

  //NOISE part
  double dnoise_sXX, dnoise_sXY, dnoise_sXZ;
  double dnoise_sYY, dnoise_sYZ;
  double dnoise_sZZ;
  double dnoise_tr;
  dnoise_tr = d_rand[vecino3] + d_rand[vecino3 + 3*ncellsGPU] + d_rand[vecino3 + 5*ncellsGPU];
  dnoise_sXX = d_rand[vecino3] - dnoise_tr/3.;
  wx += invdxGPU * fact1GPU * dnoise_sXX;

  dnoise_tr = d_rand[vecino4] + d_rand[vecino4 + 3*ncellsGPU] + d_rand[vecino4 + 5*ncellsGPU];
  dnoise_sYY = d_rand[vecino4 + 3*ncellsGPU] - dnoise_tr/3.;
  wy += invdyGPU * fact1GPU * dnoise_sYY;

  dnoise_tr = d_rand[vecino5] + d_rand[vecino5 + 3*ncellsGPU] + d_rand[vecino5 + 5*ncellsGPU];
  dnoise_sZZ = d_rand[vecino5 + 5*ncellsGPU] - dnoise_tr/3.;
  wz += invdzGPU * fact1GPU * dnoise_sZZ;

  dnoise_sXY = d_rand[i + ncellsGPU];
  wx += invdyGPU * fact4GPU * dnoise_sXY;
  wy += invdxGPU * fact4GPU * dnoise_sXY;

  dnoise_sXZ = d_rand[i + 2*ncellsGPU];
  wx += invdzGPU * fact4GPU * dnoise_sXZ;
  wz += invdxGPU * fact4GPU * dnoise_sXZ;

  dnoise_sYZ = d_rand[i + 4*ncellsGPU];
  wy += invdzGPU * fact4GPU * dnoise_sYZ;
  wz += invdyGPU * fact4GPU * dnoise_sYZ;

  dnoise_tr = d_rand[i] + d_rand[i + 3*ncellsGPU] + d_rand[i + 5*ncellsGPU];
  dnoise_sXX = d_rand[i] - dnoise_tr/3.;
  wx -= invdxGPU * fact1GPU * dnoise_sXX;

  dnoise_sYY = d_rand[i + 3*ncellsGPU] - dnoise_tr/3.;
  wy -= invdyGPU * fact1GPU * dnoise_sYY;

  dnoise_sZZ = d_rand[i + 5*ncellsGPU] - dnoise_tr/3.;
  wz -= invdzGPU * fact1GPU * dnoise_sZZ;

  dnoise_sXY = d_rand[vecino1 + ncellsGPU];
  wx -= invdyGPU * fact4GPU * dnoise_sXY;

  dnoise_sXZ = d_rand[vecino0 + 2*ncellsGPU];
  wx -= invdzGPU * fact4GPU * dnoise_sXZ;

  dnoise_sXY = d_rand[vecino2 + ncellsGPU];
  wy -= invdxGPU * fact4GPU * dnoise_sXY;

  dnoise_sYZ = d_rand[vecino0 + 4*ncellsGPU];
  wy -= invdzGPU * fact4GPU * dnoise_sYZ;

  dnoise_sXZ = d_rand[vecino2 + 2*ncellsGPU];
  wz -= invdxGPU * fact4GPU * dnoise_sXZ;

  dnoise_sYZ = d_rand[vecino1 + 4*ncellsGPU];
  wz -= invdyGPU * fact4GPU * dnoise_sYZ;
  
  /*dnoise_sXX = d_rand[vecino3];
  wx += invdxGPU * fact1GPU * dnoise_sXX;

  dnoise_sYY = d_rand[vecino4 + 4*ncellsGPU];
  wy += invdyGPU * fact1GPU * dnoise_sYY;

  dnoise_sZZ = d_rand[vecino5 + 8*ncellsGPU];
  wz += invdzGPU * fact1GPU * dnoise_sZZ;

  dnoise_sXY = d_rand[i + ncellsGPU];
  wx += invdyGPU * fact4GPU * dnoise_sXY;
  dnoise_sXY = d_rand[i + 3*ncellsGPU];
  wy += invdxGPU * fact4GPU * dnoise_sXY;

  dnoise_sXZ = d_rand[i + 2*ncellsGPU];
  wx += invdzGPU * fact4GPU * dnoise_sXZ;
  dnoise_sXZ = d_rand[i + 6*ncellsGPU];
  wz += invdxGPU * fact4GPU * dnoise_sXZ;

  dnoise_sYZ = d_rand[i + 5*ncellsGPU];
  wy += invdzGPU * fact4GPU * dnoise_sYZ;
  dnoise_sYZ = d_rand[i + 7*ncellsGPU];
  wz += invdyGPU * fact4GPU * dnoise_sYZ;

  dnoise_sXX = d_rand[i];
  wx -= invdxGPU * fact1GPU * dnoise_sXX;

  dnoise_sYY = d_rand[i + 4*ncellsGPU];
  wy -= invdyGPU * fact1GPU * dnoise_sYY;

  dnoise_sZZ = d_rand[i + 8*ncellsGPU];
  wz -= invdzGPU * fact1GPU * dnoise_sZZ;

  dnoise_sXY = d_rand[vecino1 + ncellsGPU];
  wx -= invdyGPU * fact4GPU * dnoise_sXY;

  dnoise_sXZ = d_rand[vecino0 + 2*ncellsGPU];
  wx -= invdzGPU * fact4GPU * dnoise_sXZ;

  dnoise_sXY = d_rand[vecino2 + 3*ncellsGPU];
  wy -= invdxGPU * fact4GPU * dnoise_sXY;

  dnoise_sYZ = d_rand[vecino0 + 5*ncellsGPU];
  wy -= invdzGPU * fact4GPU * dnoise_sYZ;

  dnoise_sXZ = d_rand[vecino2 + 6*ncellsGPU];
  wz -= invdxGPU * fact4GPU * dnoise_sXZ;

  dnoise_sYZ = d_rand[vecino1 + 7*ncellsGPU];
  wz -= invdyGPU * fact4GPU * dnoise_sYZ;*/

  WxZ[i].x = wx;
  WyZ[i].x = wy;
  WzZ[i].x = wz;

  WxZ[i].y = 0;
  WyZ[i].y = 0;
  WzZ[i].y = 0;

}






__global__ void kernelConstructW_2(double *vxPredictionGPU, 
				   double *vyPredictionGPU, 
				   double *vzPredictionGPU, 
				   hipfftDoubleComplex *WxZ, 
				   hipfftDoubleComplex *WyZ, 
				   hipfftDoubleComplex *WzZ, 
				   double *d_rand){
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i>=ncellsGPU) return;   

  double wx, wy, wz;
  double vx, vy, vz;
  double vx0, vx1, vx2, vx3, vx4, vx5;
  double vy0, vy1, vy2, vy3, vy4, vy5;
  double vz0, vz1, vz2, vz3, vz4, vz5;
  int vecino0, vecino1, vecino2, vecino3, vecino4, vecino5; 
  int vecinopxmy, vecinopxmz;
  int vecinomxpy, vecinomxpz;
  int vecinopymz, vecinomypz;
  double vxmxpy,vxmxpz;
  double vypxmy,vymypz;
  double vzpxmz,vzpymz;

  vecino0 = tex1Dfetch(texvecino0GPU,i);
  vecino1 = tex1Dfetch(texvecino1GPU,i);
  vecino2 = tex1Dfetch(texvecino2GPU,i);
  vecino3 = tex1Dfetch(texvecino3GPU,i);
  //vecinopxpx = tex1Dfetch(texvecino3GPU, vecino3);
  vecino4 = tex1Dfetch(texvecino4GPU,i);
  //vecinopypy = tex1Dfetch(texvecino4GPU, vecino4);
  vecino5 = tex1Dfetch(texvecino5GPU,i);
  //vecinopzpz = tex1Dfetch(texvecino5GPU, vecino5);
  //vecinopxpy = tex1Dfetch(texvecinopxpyGPU,i);
  vecinopxmy = tex1Dfetch(texvecinopxmyGPU,i);
  //vecinopxpz = tex1Dfetch(texvecinopxpzGPU,i);
  vecinopxmz = tex1Dfetch(texvecinopxmzGPU,i);
  vecinomxpy = tex1Dfetch(texvecinomxpyGPU,i);
  vecinomxpz = tex1Dfetch(texvecinomxpzGPU,i);
  //vecinopypz = tex1Dfetch(texvecinopypzGPU,i);
  vecinopymz = tex1Dfetch(texvecinopymzGPU,i);
  vecinomypz = tex1Dfetch(texvecinomypzGPU,i);

  vx = fetch_double(texVxGPU,i);
  vy = fetch_double(texVyGPU,i);
  vz = fetch_double(texVzGPU,i);
  vx0 = fetch_double(texVxGPU,vecino0);
  vx1 = fetch_double(texVxGPU,vecino1);
  vx2 = fetch_double(texVxGPU,vecino2);
  vx3 = fetch_double(texVxGPU,vecino3);
  vx4 = fetch_double(texVxGPU,vecino4);
  vx5 = fetch_double(texVxGPU,vecino5);
  vy0 = fetch_double(texVyGPU,vecino0);
  vy1 = fetch_double(texVyGPU,vecino1);
  vy2 = fetch_double(texVyGPU,vecino2);
  vy3 = fetch_double(texVyGPU,vecino3);
  vy4 = fetch_double(texVyGPU,vecino4);
  vy5 = fetch_double(texVyGPU,vecino5);
  vz0 = fetch_double(texVzGPU,vecino0);
  vz1 = fetch_double(texVzGPU,vecino1);
  vz2 = fetch_double(texVzGPU,vecino2);
  vz3 = fetch_double(texVzGPU,vecino3);
  vz4 = fetch_double(texVzGPU,vecino4);
  vz5 = fetch_double(texVzGPU,vecino5);
  vxmxpy = fetch_double(texVxGPU,vecinomxpy);
  vxmxpz = fetch_double(texVxGPU,vecinomxpz);
  vypxmy = fetch_double(texVyGPU,vecinopxmy);
  vymypz = fetch_double(texVyGPU,vecinomypz);
  vzpxmz = fetch_double(texVzGPU,vecinopxmz);
  vzpymz = fetch_double(texVzGPU,vecinopymz);



  //Laplacian part
  wx  = invdxGPU * invdxGPU * (vx3 - 2*vx + vx2);
  wx += invdyGPU * invdyGPU * (vx4 - 2*vx + vx1);
  wx += invdzGPU * invdzGPU * (vx5 - 2*vx + vx0);
  wx  = 0.5 * dtGPU * (shearviscosityGPU/densfluidGPU) * wx;
  wy  = invdxGPU * invdxGPU * (vy3 - 2*vy + vy2);
  wy += invdyGPU * invdyGPU * (vy4 - 2*vy + vy1);
  wy += invdzGPU * invdzGPU * (vy5 - 2*vy + vy0);
  wy  = 0.5 * dtGPU * (shearviscosityGPU/densfluidGPU) * wy;
  wz  = invdxGPU * invdxGPU * (vz3 - 2*vz + vz2);
  wz += invdyGPU * invdyGPU * (vz4 - 2*vz + vz1);
  wz += invdzGPU * invdzGPU * (vz5 - 2*vz + vz0);
  wz  = 0.5 * dtGPU * (shearviscosityGPU/densfluidGPU) * wz;

  //Previous Velocity
  wx += vx;
  wy += vy;
  wz += vz;
  
  //Advection part
  double advX, advY, advZ;
  vx = vxPredictionGPU[i];
  vy = vyPredictionGPU[i];
  vz = vzPredictionGPU[i];
  vx0 = vxPredictionGPU[vecino0];
  vx1 = vxPredictionGPU[vecino1];
  vx2 = vxPredictionGPU[vecino2];
  vx3 = vxPredictionGPU[vecino3];
  vx4 = vxPredictionGPU[vecino4];
  vx5 = vxPredictionGPU[vecino5];
  vy0 = vyPredictionGPU[vecino0];
  vy1 = vyPredictionGPU[vecino1];
  vy2 = vyPredictionGPU[vecino2];
  vy3 = vyPredictionGPU[vecino3];
  vy4 = vyPredictionGPU[vecino4];
  vy5 = vyPredictionGPU[vecino5];
  vz0 = vzPredictionGPU[vecino0];
  vz1 = vzPredictionGPU[vecino1];
  vz2 = vzPredictionGPU[vecino2];
  vz3 = vzPredictionGPU[vecino3];
  vz4 = vzPredictionGPU[vecino4];
  vz5 = vzPredictionGPU[vecino5];
  vxmxpy = vxPredictionGPU[vecinomxpy];
  vxmxpz = vxPredictionGPU[vecinomxpz];
  vypxmy = vyPredictionGPU[vecinopxmy];
  vymypz = vyPredictionGPU[vecinomypz];
  vzpxmz = vzPredictionGPU[vecinopxmz];
  vzpymz = vzPredictionGPU[vecinopymz];
  
  advX  = invdxGPU * ((vx3+vx)*(vx3+vx) - (vx+vx2)*(vx+vx2));
  advX += invdyGPU * ((vx4+vx)*(vy3+vy) - (vx+vx1)*(vypxmy+vy1));
  advX += invdzGPU * ((vx5+vx)*(vz3+vz) - (vx+vx0)*(vzpxmz+vz0));
  advX  = 0.25 * dtGPU * advX;
  advY  = invdxGPU * ((vy3+vy)*(vx4+vx) - (vy+vy2)*(vxmxpy+vx2));
  advY += invdyGPU * ((vy4+vy)*(vy4+vy) - (vy+vy1)*(vy+vy1));
  advY += invdzGPU * ((vy5+vy)*(vz4+vz) - (vy+vy0)*(vzpymz+vz0));
  advY  = 0.25 * dtGPU * advY;
  advZ  = invdxGPU * ((vz3+vz)*(vx5+vx) - (vz+vz2)*(vxmxpz+vx2));
  advZ += invdyGPU * ((vz4+vz)*(vy5+vy) - (vz+vz1)*(vymypz+vy1));
  advZ += invdzGPU * ((vz5+vz)*(vz5+vz) - (vz+vz0)*(vz+vz0));
  advZ  = 0.25 * dtGPU * advZ;

  //advX=0; advY=0; advZ=0;
  wx -= advX;
  wy -= advY;
  wz -= advZ;

  //NOISE part
  double dnoise_sXX, dnoise_sXY, dnoise_sXZ;
  double dnoise_sYY, dnoise_sYZ;
  double dnoise_sZZ;
  double dnoise_tr;
  dnoise_tr = d_rand[vecino3] + d_rand[vecino3 + 3*ncellsGPU] + d_rand[vecino3 + 5*ncellsGPU];
  dnoise_sXX = d_rand[vecino3] - dnoise_tr/3.;
  wx += invdxGPU * fact1GPU * dnoise_sXX;

  dnoise_tr = d_rand[vecino4] + d_rand[vecino4 + 3*ncellsGPU] + d_rand[vecino4 + 5*ncellsGPU];
  dnoise_sYY = d_rand[vecino4 + 3*ncellsGPU] - dnoise_tr/3.;
  wy += invdyGPU * fact1GPU * dnoise_sYY;

  dnoise_tr = d_rand[vecino5] + d_rand[vecino5 + 3*ncellsGPU] + d_rand[vecino5 + 5*ncellsGPU];
  dnoise_sZZ = d_rand[vecino5 + 5*ncellsGPU] - dnoise_tr/3.;
  wz += invdzGPU * fact1GPU * dnoise_sZZ;

  dnoise_sXY = d_rand[i + ncellsGPU];
  wx += invdyGPU * fact4GPU * dnoise_sXY;
  wy += invdxGPU * fact4GPU * dnoise_sXY;

  dnoise_sXZ = d_rand[i + 2*ncellsGPU];
  wx += invdzGPU * fact4GPU * dnoise_sXZ;
  wz += invdxGPU * fact4GPU * dnoise_sXZ;

  dnoise_sYZ = d_rand[i + 4*ncellsGPU];
  wy += invdzGPU * fact4GPU * dnoise_sYZ;
  wz += invdyGPU * fact4GPU * dnoise_sYZ;

  dnoise_tr = d_rand[i] + d_rand[i + 3*ncellsGPU] + d_rand[i + 5*ncellsGPU];
  dnoise_sXX = d_rand[i] - dnoise_tr/3.;
  wx -= invdxGPU * fact1GPU * dnoise_sXX;

  dnoise_sYY = d_rand[i + 3*ncellsGPU] - dnoise_tr/3.;
  wy -= invdyGPU * fact1GPU * dnoise_sYY;

  dnoise_sZZ = d_rand[i + 5*ncellsGPU] - dnoise_tr/3.;
  wz -= invdzGPU * fact1GPU * dnoise_sZZ;

  dnoise_sXY = d_rand[vecino1 + ncellsGPU];
  wx -= invdyGPU * fact4GPU * dnoise_sXY;

  dnoise_sXZ = d_rand[vecino0 + 2*ncellsGPU];
  wx -= invdzGPU * fact4GPU * dnoise_sXZ;

  dnoise_sXY = d_rand[vecino2 + ncellsGPU];
  wy -= invdxGPU * fact4GPU * dnoise_sXY;

  dnoise_sYZ = d_rand[vecino0 + 4*ncellsGPU];
  wy -= invdzGPU * fact4GPU * dnoise_sYZ;

  dnoise_sXZ = d_rand[vecino2 + 2*ncellsGPU];
  wz -= invdxGPU * fact4GPU * dnoise_sXZ;

  dnoise_sYZ = d_rand[vecino1 + 4*ncellsGPU];
  wz -= invdyGPU * fact4GPU * dnoise_sYZ;
  
  /*dnoise_sXX = d_rand[vecino3];
  wx += invdxGPU * fact1GPU * dnoise_sXX;

  dnoise_sYY = d_rand[vecino4 + 4*ncellsGPU];
  wy += invdyGPU * fact1GPU * dnoise_sYY;

  dnoise_sZZ = d_rand[vecino5 + 8*ncellsGPU];
  wz += invdzGPU * fact1GPU * dnoise_sZZ;

  dnoise_sXY = d_rand[i + ncellsGPU];
  wx += invdyGPU * fact4GPU * dnoise_sXY;
  dnoise_sXY = d_rand[i + 3*ncellsGPU];
  wy += invdxGPU * fact4GPU * dnoise_sXY;

  dnoise_sXZ = d_rand[i + 2*ncellsGPU];
  wx += invdzGPU * fact4GPU * dnoise_sXZ;
  dnoise_sXZ = d_rand[i + 6*ncellsGPU];
  wz += invdxGPU * fact4GPU * dnoise_sXZ;

  dnoise_sYZ = d_rand[i + 5*ncellsGPU];
  wy += invdzGPU * fact4GPU * dnoise_sYZ;
  dnoise_sYZ = d_rand[i + 7*ncellsGPU];
  wz += invdyGPU * fact4GPU * dnoise_sYZ;

  dnoise_sXX = d_rand[i];
  wx -= invdxGPU * fact1GPU * dnoise_sXX;

  dnoise_sYY = d_rand[i + 4*ncellsGPU];
  wy -= invdyGPU * fact1GPU * dnoise_sYY;

  dnoise_sZZ = d_rand[i + 8*ncellsGPU];
  wz -= invdzGPU * fact1GPU * dnoise_sZZ;

  dnoise_sXY = d_rand[vecino1 + ncellsGPU];
  wx -= invdyGPU * fact4GPU * dnoise_sXY;

  dnoise_sXZ = d_rand[vecino0 + 2*ncellsGPU];
  wx -= invdzGPU * fact4GPU * dnoise_sXZ;

  dnoise_sXY = d_rand[vecino2 + 3*ncellsGPU];
  wy -= invdxGPU * fact4GPU * dnoise_sXY;

  dnoise_sYZ = d_rand[vecino0 + 5*ncellsGPU];
  wy -= invdzGPU * fact4GPU * dnoise_sYZ;

  dnoise_sXZ = d_rand[vecino2 + 6*ncellsGPU];
  wz -= invdxGPU * fact4GPU * dnoise_sXZ;

  dnoise_sYZ = d_rand[vecino1 + 7*ncellsGPU];
  wz -= invdyGPU * fact4GPU * dnoise_sYZ;*/

  WxZ[i].x = wx;
  WyZ[i].x = wy;
  WzZ[i].x = wz;

  WxZ[i].y = 0;
  WyZ[i].y = 0;
  WzZ[i].y = 0;

}
